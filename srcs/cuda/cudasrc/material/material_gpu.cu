#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <float.h>

extern "C" {
  #include "../../../../includes/vectors.h"
  #include "../../../../includes/rt.h"
  #include "../../cudaheader/gpu_rt.h"
}

__host__ __device__ t_color 	handle_reflection_gpu(t_world world, t_ray *ray,
  t_intersection *intersection, int *flag)
{
  t_intersection  intersection_tmp;
  t_ray           ray_tmp;

  new_intersection(&intersection_tmp);
  intersection_tmp.id = intersection->id;
  intersection_tmp.depth = intersection->depth + 1;
  ray_tmp.origin = intersection->pos;
  ray_tmp.dir = vector_normalize(vector_substract(ray->dir,
    vector_scalar(intersection->normal_v,
    2 * vector_dot(ray->dir, intersection->normal_v))));
  get_closest_intersection(world, ray_tmp, &intersection_tmp);
	if (intersection_tmp.type == '0')
		return ((t_color){0, 0, 0});
	else
  {
    *intersection = intersection_tmp;
    *ray = ray_tmp;
    *flag = 0;
		return (intersection_tmp.color);
  }
}

__host__ __device__ t_color 	handle_transparence_gpu(t_world world,
  t_ray *ray, t_intersection *intersection, int *flag)
{
  t_intersection  intersection_tmp;
  t_ray           ray_tmp;

  new_intersection(&intersection_tmp);
  intersection_tmp.id = intersection->id;
  intersection_tmp.pos = intersection->pos;
  intersection_tmp.depth = intersection->depth + 1;
  ray_tmp.origin = intersection->pos;
  ray_tmp.dir = ray->dir;
  get_closest_intersection(world, ray_tmp, &intersection_tmp);
	if (intersection_tmp.type == '0')
		return ((t_color){0, 0, 0});
  else
  {
    *intersection = intersection_tmp;
    *ray = ray_tmp;
    *flag = 0;
		return (intersection_tmp.color);
  }
}

__host__ __device__ t_color 	handle_refraction_gpu(t_world world,
  t_ray *ray, t_intersection *intersection, int *flag)
{
  t_intersection  intersection_tmp;
  t_ray           ray_tmp;
  double          etai;
  double          cosi;
  double          eta;
  double          etat;
  double          k;
  t_vec3d         n;

  new_intersection(&intersection_tmp);
  intersection_tmp.id = intersection->id;
  intersection_tmp.pos = intersection->pos;
  intersection_tmp.depth = intersection->depth + 1;
  ray_tmp.origin = intersection->pos;
  cosi = clamp(-1, 1, vector_dot(ray->dir, intersection->normal_v));
  etai = 1.0;
  etat = intersection->refraction_coef + 1.;
  n = intersection->normal_v;
  if (cosi < 0)
    cosi = -cosi;
  else
  {
    swap_double(&etai, &etat);
    n = vector_scalar(n, -1);
  }
  eta = etai / etat;
  k = 1.0 - eta * eta * (1.0 - cosi * cosi);
  if (k < 0)
    return (intersection->color);
  ray_tmp.dir = vector_add(vector_scalar(ray->dir, eta),
      vector_scalar(n, (eta * cosi - sqrt(k))));
  if (get_closest_intersection(world, ray_tmp, &intersection_tmp) == 0)
		return ((t_color){0, 0, 0});
  else
  {
    *intersection = intersection_tmp;
    *ray = ray_tmp;
    *flag = 0;
    return (intersection_tmp.color);
  }
}
