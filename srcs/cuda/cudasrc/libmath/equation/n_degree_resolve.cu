#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   n_degree_resolve.cu                                :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: aanzieu <marvin@42.fr>                     +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/06/19 14:45:23 by aanzieu           #+#    #+#             */
/*   Updated: 2017/08/24 12:08:09 by aanzieu          ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

extern "C" {
#include "../../../cudaheader/gpu_rt.h"
}

__host__ __device__ static t_dichotomie		init_stru(double *a)
{
	t_dichotomie	ret;

	ret.q = a[2] * a[2] - 3 * a[1];
	ret.r = 2 * a[2] * a[2] * a[2] - 9 * a[2] * a[1] + 27 * a[0];
	ret.bq = ret.q / 9;
	ret.br = ret.r / 54;
	ret.bq3 = ret.bq * ret.bq * ret.bq;
	ret.br2 = ret.br * ret.br;
	ret.cr2 = 729 * ret.r * ret.r;
	ret.cq3 = 2916 * ret.q * ret.q * ret.q;
	ret.sgnbr = ret.br >= 0 ? 1 : -1;
	ret.i = 0;
	return (ret);
}

__host__ __device__ static int				solve_n(double *a, double *r)
{
	t_dichotomie	c;

	c = init_stru(a);
	if (c.br == 0 && c.bq == 0)
	{
		while (c.i++ < 3)
			r[c.i] = -a[2] / 3;
		return (3);
	}
	else if (c.cr2 == c.cq3)
		return (solve_second_case(c, a, r));
	else if (c.br2 < c.bq3)
		return (solve_third_case(c, a, r));
	else
		return (solve_fourth_case(c, a, r));
}

__host__ __device__ static int				find_inside(t_mobius m, t_vec3d h)
{
	double t;
	double s;

	t = atan2(h.y, h.x);
	if (dblsgn(sin(t / 2)) != 0)
		s = h.z / sin(t / 2);
	else
		s = dblsgn(cos(t)) ? (h.x / cos(t) - m.radius) / cos(t / 2) :
			(h.y / sin(t) - m.radius) / cos(t / 2);
	h.x -= (m.radius + s * cos(t / 2)) * cos(t);
	h.y -= (m.radius + s * cos(t / 2)) * sin(t);
	h.z -= s * sin(t / 2);
	if (dblsgn(h.x * h.x + h.y * h.y + h.z * h.z))
		return (0);
	return (s >= -0.5 && s <= 0.5);
}

__host__ __device__ double				solver_n_degree(double *coef,
		int degree, t_mobius m, t_ray ray)
{
	double	sols[3];
	int		i;
	int		ret;
	t_vec3d hit;

	if (degree < 1)
		return (-1.0);
	i = -1;
	ret = solve_n(coef, sols);
	while (++i < ret)
	{
		if (sols[i] > EPSILON)
		{
			hit = vector_add(vector_scalar(ray.dir, sols[i]), ray.origin);
			if (find_inside(m, hit))
				return (sols[i]);
		}
	}
	return (INFINITY);
}
