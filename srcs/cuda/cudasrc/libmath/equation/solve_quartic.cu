#include "hip/hip_runtime.h"
extern "C" {
#include "../../../../../includes/rt.h"
#include "../../../cudaheader/gpu_rt.h"
#include <equation.h>
#include <math.h>
}

__host__ __device__ void	set_pqr_quartic(double *pqr, double *coef)
{
  pqr[0] = -3.0 * (coef[1] * coef[1]) / (8.0 * coef[0] * coef[0])
    + (coef[2] / coef[0]);
  pqr[1] = (pow(coef[1] / 2.0, 3) / pow(coef[0], 3))
    - (0.5 * coef[1] * coef[2])
    / pow(coef[0], 2) + coef[3] / coef[0];
  pqr[2] = -3.0 * pow((coef[1] / 4.0) / coef[0], 4)
    + coef[2] * (pow(coef[1] / 4.0, 2) / pow(coef[0], 3))
    - ((0.25 * coef[1] * coef[3]) / (coef[0] * coef[0]))
    + (coef[4] / coef[0]);
}

__host__ __device__ void	init_to_send_cubic(double p, double q,
			   double r, double *coef)
{
  coef[0] = 8.0;
  coef[1] = -4.0 * p;
  coef[2] = -8.0 * r;
  coef[3] = 4.0 * r * p - q * q;
}

__host__ __device__ int		init_and_send_second(double *pqr, double y0, double *res)
{
  double	a0;
  double	b0;
  double	coef[3];
  int		nb_sol;

  nb_sol = 0;
  a0 = sqrt(-pqr[0] + 2.0 * y0);
  if ((2.0 * y0 - pqr[0]) != 0.0)
    b0 = -pqr[1] / (2.0 * a0);
  else
    b0 = sqrt(y0 * y0 - pqr[2]);
  coef[0] = 1.0;
  coef[1] = -a0;
  coef[2] = y0 - b0;
  nb_sol = solve_quadratic(coef, res);
  coef[0] = 1.0;
  coef[1] = a0;
  coef[2] = y0 + b0;
  nb_sol += solve_quadratic(coef, &(res[nb_sol]));
  return (nb_sol);
}

__host__ __device__ int		solve_quartic(double *coef, double *res)
{
  double	pqr[3];
  double	coef_tmp[4];
  double	res_tmp[3];
  int		nb_sol;
  int		count;

  count = 0;
  nb_sol = 0;
  if (fabs(coef[0]) <= SURFACE_TOLERANCE)
    return (solve_cubic(&(coef[1]), res));
  set_pqr_quartic(pqr, coef);
  init_to_send_cubic(pqr[0], pqr[1], pqr[2], coef_tmp);
  if (solve_cubic(coef_tmp, res_tmp) == 0)
    return (0);
  nb_sol = init_and_send_second(pqr, res_tmp[0], res);
  while (count < nb_sol)
    res[count++] -= coef[1] / (4.0 * coef[0]);
  return (nb_sol);
}
