#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   plane.c                                            :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: svilau <marvin@42.fr>                      +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/04/25 10:49:50 by svilau            #+#    #+#             */
/*   Updated: 2017/04/25 14:07:23 by svilau           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

extern "C" {
	#include <rt.h>
	#include "gpu_rt.h"
	#include <vectors.h>
}

/*
**	On envoie le rayon et la structure qui contient le plan et la fonction
**	ecrit sur 't_vec3d *intersection' les coordonees du point d'intersection
**	avec le plan
*/

__host__ __device__ int		get_plane(t_plane plane, t_ray ray,
	t_intersection *intersection_tmp)
{
	double	t;
	double	v, n;
	t_vec3d	x, normal_v, norm, pos;

	if (intersection_tmp->id == plane.id)
		return (0);
	normal_v = vector_normalize(vector_calculate(plane.pos, plane.up));
	v = vector_dot(ray.dir, normal_v);
	if (v != 0)
	{
		x = vector_scalar(vector_calculate(plane.pos, ray.origin), -1);
		n = vector_dot(x, normal_v);
		t = n / v;
		if (t > 0.0000001 && t < 100000)
		{
			intersection_tmp->t = t;
			if (v > 0)
				norm = vector_scalar(normal_v, -1);
			else
				norm = normal_v;
			pos = vector_add(ray.origin, vector_scalar(ray.dir, t));
			intersection_tmp->color = plane.color;
			intersection_tmp->normal_v = norm;
			intersection_tmp->pos = pos;
			return (1);
		}
	}
	return (0);
}

__host__ __device__ void	get_closest_plane(t_world world, t_ray ray,
			t_intersection *intersection, t_intersection *intersection_tmp)
{
	int i;

	i = 0;
	while (i < world.planes_len)
	{
		if (get_plane(world.planes[i], ray, intersection_tmp) == 1)
		{
			if (intersection_tmp->t < intersection->t && intersection_tmp->t != -1)
			{
				intersection->type = 'p';
				intersection->id = world.planes[i].id;
				intersection->id_save = world.planes[i].id;
				intersection->t = intersection_tmp->t;
				intersection->reflection_coef = world.planes[i].reflection_coef;
				intersection->refraction_coef = world.planes[i].refraction_coef;
				intersection->transparence_coef = world.planes[i].transparence_coef;
				intersection->color = world.planes[i].color;
				intersection->chess = world.planes[i].chess;
				intersection->pos = intersection_tmp->pos;
				intersection->normal_v = intersection_tmp->normal_v;
				if(world.on == 1){
				apply_noise_dist(world, intersection, world.planes[i].perlin);}
			}
		}
		i++;
	}
}
