#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   paraboloid.c                                       :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: aanzieu <marvin@42.fr>                     +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2016/10/20 10:49:50 by svilau            #+#    #+#             */
/*   Updated: 2017/03/17 14:07:23 by svilau           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

extern "C" {
	#include <rt.h>
	#include "gpu_rt.h"
	#include <vectors.h>
	#include <equation.h>
}

__host__ __device__ t_vec3d	get_normal_paraboloid(t_paraboloid para, t_ray ray, t_intersection intersection)
{
	t_vec3d	normal_v;
	t_vec3d	x;
	t_vec3d	v;
	double	m;
	
	x = vector_calculate(ray.origin, para.top);
	v = vector_normalize(para.normal);
	m = vector_dot(ray.dir, vector_scalar(v, intersection.t))
		+ vector_dot(x, v);
	normal_v = vector_normalize(vector_substract(vector_calculate(para.top, intersection.pos),
				vector_scalar(v, m + para.distance)));
	return (normal_v);
}

__host__ __device__ void	get_determinant_paraboloid(t_paraboloid para, t_camera camera, t_ray ray,
														t_2deg *equation)
{
}

__host__ __device__ int		get_paraboloid(t_paraboloid para, t_camera camera, t_ray ray,
									t_intersection *intersection_tmp)
{
	t_vec3d	x;
	t_vec3d	v;
	t_eq	eq;

	x = vector_calculate(para.top, ray.origin);
	v = vector_normalize(para.normal);
	eq.a = vector_dot(ray.dir, ray.dir)
		- (pow(vector_dot(ray.dir, v), 2));
	eq.b = 2 * (vector_dot(ray.dir, x)
			- (vector_dot(ray.dir, v)
				* ((vector_dot(x, v) + 2 * para.distance))));
	eq.c = vector_dot(x, x)
		- (vector_dot(x, v)
				* (vector_dot(x, v) + 4 * para.distance));
	eq.res = second_degres(eq.a, eq.b, eq.c);
	if(eq.res != NOT_A_SOLUTION)
	{
		intersection_tmp->t = eq.res;
		intersection_tmp->type = 'x';
		return(1);
	}
	intersection_tmp->t = -1.0;
	return(0);
}

__host__ __device__ void	get_closest_paraboloid(t_world world, t_ray ray,
			t_intersection *intersection, t_intersection *intersection_tmp)
{
	int i;

	i = 0;
	while (i < world.paraboloids_len)
	{
		if (get_paraboloid(world.paraboloids[i], world.camera, ray, intersection_tmp) == 1)
		{
			if (intersection_tmp->t < intersection->t)
			{
				intersection->t = intersection_tmp->t;
				intersection->type = intersection_tmp->type;
				intersection->reflexion_coef = 0;
				intersection->color = &world.paraboloids[i].color;
				intersection->pos = vector_add(ray.origin,
					vector_scalar(ray.dir, intersection_tmp->t));
				intersection->normal_v = get_normal_paraboloid(world.paraboloids[i], ray,
														*intersection);
			}
		}
		i++;
	}
}
