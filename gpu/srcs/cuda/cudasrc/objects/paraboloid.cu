#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   paraboloid.c                                       :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: aanzieu <marvin@42.fr>                     +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2016/10/20 10:49:50 by svilau            #+#    #+#             */
/*   Updated: 2017/03/17 14:07:23 by svilau           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

extern "C" {
	#include <rt.h>
	#include "gpu_rt.h"
	#include <vectors.h>
}

__host__ __device__ t_vec3d	get_normal_paraboloid(t_paraboloid para, t_ray ray, t_intersection intersection)
{
	t_vec3d	normal_v;
	t_vec3d	x;
	t_vec3d	v;
	double	m;
	
	x = vector_calculate(ray.origin, para.top);
	v = vector_normalize(para.normal);
	m = vector_dot(ray.dir, vector_scalar(v, intersection.t))
		+ vector_dot(x, v);
	normal_v = vector_normalize(vector_substract(vector_calculate(para.top, intersection.pos),
				vector_scalar(v, m + para.distance)));
	return (normal_v);
}

__host__ __device__ void	get_determinant_paraboloid(t_paraboloid para, t_camera camera, t_ray ray,
														t_2deg *equation)
{
	t_vec3d	x;
	t_vec3d	v;

	x = vector_calculate(para.top, ray.origin);
	v = vector_normalize(para.normal);
	equation->a = vector_dot(ray.dir, ray.dir)
		- (pow(vector_dot(ray.dir, v), 2));
	equation->b = 2 * (vector_dot(ray.dir, x)
			- (vector_dot(ray.dir, v)
				* ((vector_dot(x, v) + 2 * para.distance))));
	equation->c = vector_dot(x, x)
		- (vector_dot(x, v)
				* (vector_dot(x, v) + 4 * para.distance));
	equation->det = pow(equation->b, 2) - (4 * (equation->a) * (equation->c));
}

__host__ __device__ int		get_paraboloid(t_paraboloid para, t_camera camera, t_ray ray,
									t_intersection *intersection_tmp)
{
	t_2deg	equation;
	double	t1;
	double	t2;

	get_determinant_paraboloid(para, camera, ray, &equation);
	if (equation.det >= 0)
	{
		t1 = ((-1) * equation.b + sqrt(equation.det)) / (2 * equation.a);
		t2 = ((-1) * equation.b - sqrt(equation.det)) / (2 * equation.a);
		if (t1 <= t2 && t1 > 0.0000001)
		{
			intersection_tmp->t = t1;
			intersection_tmp->type = 'x';
			return (1);
		}
		else if (t2 > 0.0000001)
		{
			intersection_tmp->t = t2;
			intersection_tmp->type = 'x';
			return (1);
		}
	}
	return (0);
}

__host__ __device__ void	get_closest_paraboloid(t_world world, t_ray ray,
			t_intersection *intersection, t_intersection *intersection_tmp)
{
	int i;

	i = 0;
	while (i < world.paraboloids_len)
	{
		if (get_paraboloid(world.paraboloids[i], world.camera, ray, intersection_tmp) == 1)
		{
			if (intersection_tmp->t < intersection->t)
			{
				intersection->t = intersection_tmp->t;
				intersection->type = intersection_tmp->type;
				intersection->reflexion_coef = 0;
				intersection->color = &world.paraboloids[i].color;
				intersection->pos = vector_add(ray.origin,
					vector_scalar(ray.dir, intersection_tmp->t));
				intersection->normal_v = get_normal_paraboloid(world.paraboloids[i], ray,
														*intersection);
			}
		}
		i++;
	}
}
