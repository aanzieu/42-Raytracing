#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   paraboloid.c                                       :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: aanzieu <marvin@42.fr>                     +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2016/10/20 10:49:50 by svilau            #+#    #+#             */
/*   Updated: 2017/03/17 14:07:23 by svilau           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

extern "C" {
	#include <rt.h>
	#include "gpu_rt.h"
	#include <vectors.h>
	#include <equation.h>
}

__host__ __device__ t_vec3d	get_normal_paraboloid(t_paraboloid para, t_ray ray,
	t_intersection intersection)
{
	t_vec3d	normal_v;
	t_vec3d	x;
	t_vec3d	v;
	double	m;

	x = vector_calculate(para.top, ray.origin);
	v = vector_normalize(para.normal);
	m = vector_dot(ray.dir, vector_scalar(v, intersection.t))
		+ vector_dot(x, v);
	normal_v = vector_normalize(vector_substract(vector_calculate(para.top, intersection.pos),
				vector_scalar(v, m + para.distance)));
	return (normal_v);
}

__host__ __device__ int		get_paraboloid(t_paraboloid para, t_ray ray,
									t_intersection *intersection_tmp)
{
	t_vec3d	x;
	t_vec3d	v;
	t_eq	eq;

	if (intersection_tmp->id == para.id)
		return (0);
	x = vector_calculate(para.top, ray.origin);
	v = vector_normalize(para.normal);
	eq.a = vector_dot(ray.dir, ray.dir)
		- (pow(vector_dot(ray.dir, v), 2));
	eq.b = 2 * (vector_dot(ray.dir, x)
			- (vector_dot(ray.dir, v)
				* ((vector_dot(x, v) + 2 * para.distance))));
	eq.c = vector_dot(x, x)
		- (vector_dot(x, v)
				* (vector_dot(x, v) + 4 * para.distance));
	second_degres(&eq);
	if(eq.res[0] != NOT_A_SOLUTION)
	{
		intersection_tmp->t = eq.res[0];
		return (1);
	}
//	intersection_tmp->t = -1.0;
	return (0);
}

__host__ __device__ void	get_closest_paraboloid(t_world world, t_ray ray,
			t_intersection *intersection, t_intersection *intersection_tmp)
{
	int i;

	i = 0;
	while (i < world.paraboloids_len)
	{
		if (get_paraboloid(world.paraboloids[i], ray, intersection_tmp) == 1)
		{
			if (intersection_tmp->t < intersection->t && intersection_tmp->t != 1)
			{
				intersection->t = intersection_tmp->t;
				intersection->id = world.paraboloids[i].id;
				intersection->id_save = world.paraboloids[i].id;
				intersection->type = 'b';
				intersection->reflection_coef = world.paraboloids[i].reflection_coef;
				intersection->refraction_coef = world.paraboloids[i].refraction_coef;
				intersection->transparence_coef = world.paraboloids[i].transparence_coef;
				intersection->color = world.paraboloids[i].color;
				intersection->chess = world.paraboloids[i].chess;
				intersection->pos = vector_add(ray.origin,
					vector_scalar(ray.dir, intersection_tmp->t));
				intersection->normal_v = get_normal_paraboloid(world.paraboloids[i], ray,
														*intersection);
				apply_noise_dist(world, intersection, world.paraboloids[i].perlin);
			}
		}
		i++;
	}
}
