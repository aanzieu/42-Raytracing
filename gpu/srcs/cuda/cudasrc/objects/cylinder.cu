#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   cylinder.c                                         :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: svilau <marvin@42.fr>                      +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/04/25 10:49:50 by svilau            #+#    #+#             */
/*   Updated: 2017/04/25 14:07:23 by svilau           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

extern "C" {
	#include <rt.h>
	#include <gpu_rt.h>
}
#include <float.h>

__host__ __device__ t_vec3d	get_normal_cylinder(t_cylinder cylinder, t_camera camera, t_ray ray,
												t_intersection intersection)
{
	t_vec3d axis_v;
	t_vec3d	normal_v;
	t_vec3d	x;
	double	m;

	x = vector_substract(ray.origin, cylinder.pos);
	axis_v = vector_normalize(vector_calculate(cylinder.pos, cylinder.up));
	m = vector_dot(ray.dir, axis_v) * intersection.t + vector_dot(x, axis_v);
	normal_v = vector_normalize(
				vector_substract(
					vector_substract(
						intersection.pos,
						cylinder.pos),
					vector_scalar(
						axis_v,
						m)));
	return (normal_v);
}

__host__ __device__ void	get_determinant_cylinder(t_cylinder cylinder, t_camera camera,
												t_ray ray, t_2deg *equation)
{
	t_vec3d	normal_v;
	t_vec3d	x;

	normal_v = vector_normalize(vector_calculate(cylinder.pos, cylinder.up));
	x = vector_substract(ray.origin, cylinder.pos);
	equation->a = vector_dot(ray.dir, ray.dir) -
		pow(vector_dot(ray.dir, normal_v), 2);
	equation->b = 2 * (vector_dot(ray.dir, x) -
		(vector_dot(ray.dir, normal_v) * vector_dot(x, normal_v)));
	equation->c = vector_dot(x, x) -
		pow(vector_dot(x, normal_v), 2) - pow(cylinder.radius, 2);
	equation->det = pow(equation->b, 2) - (4 * (equation->a) * (equation->c));
}

__host__ __device__ int		get_cylinder(t_cylinder cylinder, t_camera camera, t_ray ray,
											t_intersection *intersection_tmp)
{
	t_2deg	equation;
	double	t1;
	double	t2;

	get_determinant_cylinder(cylinder, camera, ray, &equation);
	if (equation.det >= 0)
	{
		t1 = ((-1) * equation.b + sqrt(equation.det)) / (2 * equation.a);
		t2 = ((-1) * equation.b - sqrt(equation.det)) / (2 * equation.a);
		if (t1 <= t2 && t1 > 0.0000001)
		{
			intersection_tmp->t = t1;
			intersection_tmp->type = 'c';
			return (1);
		}
		else if (t2 > 0.0000001)
		{
			intersection_tmp->t = t2;
			intersection_tmp->type = 'c';
			return (1);
		}
	}
	return (0);
}

__host__ __device__ void	get_closest_cylinder(t_world world, t_ray ray,
				t_intersection *intersection, t_intersection *intersection_tmp)
{
	int i;

	i = 0;
	while (i < world.cylinders_len)
	{
		if (get_cylinder(world.cylinders[i], world.camera, ray, intersection_tmp) == 1)
		{
			if (intersection_tmp->t < intersection->t)
			{
				intersection->t = intersection_tmp->t;
				intersection->type = intersection_tmp->type;
				intersection->color = &world.cylinders[i].color;
				intersection->pos = vector_add(ray.origin,
					vector_scalar(ray.dir, intersection->t));
				intersection->normal_v = get_normal_cylinder(world.cylinders[i],
											world.camera, ray, *intersection);
			}
		}
		i++;
	}
}
