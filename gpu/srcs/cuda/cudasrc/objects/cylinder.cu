#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   cylinder.c                                         :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: svilau <marvin@42.fr>                      +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/04/25 10:49:50 by svilau            #+#    #+#             */
/*   Updated: 2017/04/25 14:07:23 by svilau           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

extern "C" {
	#include <rt.h>
	#include <gpu_rt.h>
	#include <equation.h>
}
#include <float.h>

//
// __host__ __device__ t_vec3d	get_normal_cylinder(t_cylinder cylinder, t_ray ray,
// 												t_intersection intersection)
// {
// 	t_vec3d axis_v;
// 	t_vec3d	normal_v;
// 	t_vec3d	x;
// 	double	m;
//
// 	x = vector_substract(ray.origin, cylinder.pos);
// 	axis_v = vector_normalize(vector_calculate(cylinder.pos, cylinder.up));
// 	m = vector_dot(ray.dir, axis_v) * intersection.t + vector_dot(x, axis_v);
// 	normal_v = vector_normalize(
// 		vector_substract(
// 		vector_substract(intersection.pos, cylinder.pos),
// 					vector_scalar(axis_v, m)));
// 	return (normal_v);
// }
//
//
// __host__ __device__ int 	happend_cap(t_cylinder cylinder, t_ray ray, t_eq eq,
// 	t_intersection *intersection_tmp, t_vec3d normal_v, t_vec3d x)
// {
// 	double type;
//
// 	if (eq.res[1] > eq.res[2] && eq.res[2] > 0)
// 		eq.res[1] = eq.res[2];
// 	if (eq.res[1] < 0)
// 	{
// 	 	intersection_tmp->t = -1.0;
// 		return (-1);
// 	}
// 	type = inter_caps(cylinder, ray, eq.res);
// 	intersection_tmp->color = cylinder.color;
// 	if (eq.res[1] < 0 || type < 0)
// 	{
// 	 	intersection_tmp->t = -1.0;
// 		return (-1);
// 	}
// 	intersection_tmp->pos = vector_add(ray.origin, vector_scalar(ray.dir, type == 1 ? eq.res[1] : eq.res[2]));
// 	intersection_tmp->t = type == 1 ? eq.res[2] : eq.res[1];
// 	if (type == 1)
// 		intersection_tmp->normal_v = get_normal_cylinder(cylinder, ray, *intersection_tmp);
// 	else
// 		intersection_tmp->normal_v = vector_dot(ray.dir, normal_v) > 0 ? vector_scalar(normal_v, -1) : normal_v;
// 	return (1);
// }
//
// __host__ __device__ int 	limit_cylinder2(t_cylinder cylinder, t_ray ray,
// 	t_eq eq, t_intersection *intersection_tmp, t_vec3d normal_v, t_vec3d x)
// {
// 	double m1, m2, m3, maxm;
// 	t_plane p1, p2;
// 	t_intersection test;
// 	t_vec3d axis_v;
//
// 	maxm = cylinder.height;
// 	axis_v = vector_normalize(vector_calculate(cylinder.pos, cylinder.up));
// 	eq.res[1] = eq.res[1] > eq.res[2] && eq.res[2] > 0 ? eq.res[2] : eq.res[1];
// 	m1 = vector_dot(ray.dir, axis_v) * eq.res[1] + vector_dot(x, axis_v);
// 	m2 = vector_dot(ray.dir, axis_v) * eq.res[2] + vector_dot(x, axis_v);
// 	// if (m1 > 0 && m1 < cylinder.height)// && )
// 	// 	m3 = m1;
// 	// if (m2 < m3 && m2 > 0 && m2 < cylinder.height)
// 	// 	m3 = m2;
//
// 	// if (m3 < cylinder.height)
// 	// {
// 	// 	intersection_tmp->t = eq.res[0];
// 	// 	intersection_tmp->pos = vector_add(ray.origin, vector_scalar(ray.dir, intersection_tmp->t));
// 	// 	intersection_tmp->normal_v = get_normal_cylinder(cylinder, ray, *intersection_tmp);
// 	// 	return (1);
// 	// }
// 	m1 = m1 < m2 ? m1 : m2;
// 	if (m1 < 0 || m1 > cylinder.height)// || m2 < 0 || m2 > cylinder.height)
// 	{
// 	//		(C,-V) and the other by (C+V*maxm,V)
// 		p1.pos = cylinder.pos;
// 		p2.pos = vector_add(cylinder.pos, vector_scalar(normal_v, cylinder.height));
// 		p1.up = cylinder.up;//vector_scalar(normal_v, -1);
// 		p2.up = cylinder.up;//normal_v;
// 		test.t = DBL_MAX;
// 		test.id = intersection.id;
// 		if ((get_plane(p2, ray, &test) == 1))// || (get_plane2(p2, ray, &test) == 1))
// 		{
// 			intersection_tmp->t = test.t;
// 			intersection_tmp->normal_v = test.normal_v;
// 			return (1);
// 		}
// 	}
// 	else
// 	{
// 		intersection_tmp->t = eq.res[0];
// 		intersection_tmp->pos = vector_add(ray.origin, vector_scalar(ray.dir, intersection_tmp->t));
// 		intersection_tmp->normal_v = get_normal_cylinder(cylinder, ray, *intersection_tmp);
// 		return (1);
// 	}
// 	return (-1);
// }

__host__ __device__ int		limit_cylinder(t_cylinder cylinder, t_ray ray,
		t_intersection *intersection_tmp, t_vec3d axis, t_vec3d x, t_eq eq)
{
	double b[2];

	b[0] = vector_dot(ray.dir, axis) * eq.res[0] + vector_dot(x, axis);
	b[1] = vector_dot(ray.dir, axis) * eq.res[1] + vector_dot(x, axis);
	if ((b[0] > 0 && b[0] < cylinder.height)
		|| (b[1] > 0 && b[1] < cylinder.height))
	{
		if (b[0] > 0 && b[0] < cylinder.height)
		intersection_tmp->normal_v = vector_normalize(
			vector_substract(vector_substract(
					intersection_tmp->pos, cylinder.pos), vector_scalar(axis, b[0])));
		else
			if (b[1] > 0 && b[1] < cylinder.height)
				intersection_tmp->normal_v = (t_vec3d){0, 0, 0};
		if (cylinder.refraction_coef != 0 || cylinder.reflection_coef != 0)
			intersection_tmp->id = cylinder.id;
		return (1);
	}
	return (0);
}

__host__ __device__ int		get_cylinder(t_world world, t_cylinder cylinder,
							t_ray ray, t_intersection *intersection_tmp)
{
	t_eq		eq;
	t_vec3d axis;
	t_vec3d x;

 	if (intersection_tmp->id == cylinder.id)
		return (0);
	axis = vector_normalize(vector_substract(cylinder.up, cylinder.pos));
	x = vector_substract(ray.origin, cylinder.pos);
	eq.a = vector_dot(ray.dir, ray.dir) - pow(vector_dot(ray.dir, axis), 2);
	eq.b = 2 * (vector_dot(ray.dir, x) - (vector_dot(ray.dir, axis)
		* vector_dot(x, axis)));
	eq.c = vector_dot(x, x) - pow(vector_dot(x, axis), 2)
		- pow(cylinder.radius, 2);
	second_degres(&eq);
	if (eq.res[0] != NOT_A_SOLUTION)
	{
		intersection_tmp->t = eq.res[0];
		intersection_tmp->pos = vector_add(ray.origin,
			vector_scalar(ray.dir, intersection_tmp->t));
		x = vector_substract(ray.origin, cylinder.pos);
		if (limit_cylinder(cylinder, ray, intersection_tmp, axis, x, eq))
			return (1);
	}
 	intersection_tmp->t = -1;
	return (-1);
}

__host__ __device__ void	get_closest_cylinder(t_world world, t_ray ray,
				t_intersection *intersection, t_intersection *intersection_tmp)
{
	int i;

	i = 0;
	while (i < world.cylinders_len)
	{
		if (get_cylinder(world, world.cylinders[i], ray, intersection_tmp) == 1)
		{
			if (intersection_tmp->t < intersection->t && intersection_tmp->t > 0)
			{
				intersection->id = world.cylinders[i].id;
				intersection->t = intersection_tmp->t;
				intersection->type = 'c';
				intersection->reflection_coef = world.cylinders[i].reflection_coef;
				intersection->refraction_coef = world.cylinders[i].refraction_coef;
				intersection->transparence_coef = world.cylinders[i].transparence_coef;
				intersection->color = world.cylinders[i].color;
				intersection->chess = world.cylinders[i].chess;
				intersection->pos = intersection_tmp->pos;
				intersection->normal_v = intersection_tmp->normal_v;
			}
		}
		i++;
	}
}
