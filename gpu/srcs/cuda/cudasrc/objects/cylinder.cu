#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   cylinder.c                                         :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: svilau <marvin@42.fr>                      +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/04/25 10:49:50 by svilau            #+#    #+#             */
/*   Updated: 2017/04/25 14:07:23 by svilau           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

extern "C" {
	#include <rt.h>
	#include <gpu_rt.h>
	#include <equation.h>
}
#include <float.h>

__host__ __device__ static void		get_normal_cylinder(t_cylinder cylinder,\
		t_intersection *intersection_tmp)
{
	t_vec3d		pos;
	double		tmp;

	pos = vector_substract(intersection_tmp->pos,\
		vector_add(cylinder.pos, cylinder.up));
	tmp = vector_dot(cylinder.up, pos) / vector_dot(cylinder.up, cylinder.up);
	pos = vector_add(vector_add(cylinder.pos, cylinder.up),\
		vector_scalar(cylinder.up, tmp));
	intersection_tmp->normal_v = vector_normalize(\
		vector_substract(intersection_tmp->pos, pos));
}

__host__ __device__ static double	limit_cylinder_next(t_eq eq,\
		t_cylinder cyl, t_ray ray)
{
	t_vec3d pos[2];
	t_vec3d caps[2];

	pos[0] = vector_add(ray.origin, vector_scalar(ray.dir, eq.res[0]));
	pos[1] = vector_add(cyl.pos, vector_scalar(cyl.up, cyl.height));
	caps[0] = vector_substract(pos[0], cyl.pos);
	caps[1] = vector_substract(pos[0], pos[1]);
	if (vector_dot(cyl.up, caps[0]) > SURFACE_TOLERANCE\
		&& vector_dot(cyl.up, caps[1]) < SURFACE_TOLERANCE)
		return (eq.res[0]);
	eq.res[0] = ((vector_dot(cyl.up, cyl.pos) -\
				vector_dot(cyl.up, ray.origin)) / vector_dot(cyl.up, ray.dir));
	eq.res[1] = ((vector_dot(cyl.up, pos[1]) -\
				vector_dot(cyl.up, ray.origin)) / vector_dot(cyl.up, ray.dir));
	if (eq.res[0] < eq.res[1] && eq.res[0] > SURFACE_TOLERANCE)
	{
		pos[0] = vector_add(ray.origin, vector_scalar(ray.dir, eq.res[0]));
		caps[0] = vector_substract(pos[0], cyl.pos);
		if (sqrt(vector_dot(caps[0], caps[0])) < cyl.radius)
			return (eq.res[0]);
	}
	else if (eq.res[1] < eq.res[0] && eq.res[1] > SURFACE_TOLERANCE)
	{
		pos[0] = vector_add(ray.origin, vector_scalar(ray.dir, eq.res[1]));
		caps[1] = vector_substract(pos[0], pos[1]);
		if (sqrt(vector_dot(caps[1], caps[1])) < cyl.radius)
			return (eq.res[1]);
	}
	return (-1);
}
__host__ __device__ static int limit_cylinder(t_eq eq, t_cylinder cylinder,\
		t_ray ray, t_intersection *intersection_tmp)
{
	double	t_save;

	if (eq.res[1] > eq.res[0])
		eq.res[1] = eq.res[0];
	t_save = eq.res[1];
	if (cylinder.height > 0)
		if ((eq.res[1] = limit_cylinder_next(eq, cylinder, ray)) == -1)
			return (-1);
	intersection_tmp->t = eq.res[0] == t_save ? eq.res[1] : eq.res[0];
	intersection_tmp->pos = vector_add(ray.origin,
		vector_scalar(ray.dir, intersection_tmp->t));
	if (intersection_tmp->t == t_save)
		get_normal_cylinder(cylinder, intersection_tmp);
	else
		intersection_tmp->normal_v = (t_vec3d){0, 1, 0};
	return (1);
}
__host__ __device__ int		get_cylinder(t_world world, t_cylinder cylinder,\
		t_ray ray, t_intersection *intersection_tmp)
{
	t_eq		eq;
	t_vec3d x;
	t_vec3d axis_v[2];

 	if (intersection_tmp->id == cylinder.id)
		return (0);
	x = vector_substract(ray.origin, cylinder.pos);
	cylinder.up = vector_normalize(cylinder.up);
	axis_v[0] = vector_scalar(cylinder.up, vector_dot(ray.dir, cylinder.up));
	axis_v[0] = vector_substract(ray.dir, axis_v[0]);
	axis_v[1] = vector_scalar(cylinder.up, vector_dot(x, cylinder.up));
	axis_v[1] = vector_substract(x, axis_v[1]);
	eq.a = vector_dot(axis_v[0], axis_v[0]);
	eq.b = 2 * vector_dot(axis_v[0], axis_v[1]);
	eq.c = vector_dot(axis_v[1], axis_v[1]) - cylinder.radius * cylinder.radius;
	second_degres(&eq);
	if (eq.res[0] != NOT_A_SOLUTION)
		return (limit_cylinder(eq, cylinder, ray, intersection_tmp));
	intersection_tmp->t = -1;
	return (-1);
}

__host__ __device__ void	get_closest_cylinder(t_world world, t_ray ray,
				t_intersection *intersection, t_intersection *intersection_tmp)
{
	int i;

	i = 0;
	while (i < world.cylinders_len)
	{
		if (get_cylinder(world, world.cylinders[i], ray, intersection_tmp) == 1)
		{
			if (intersection_tmp->t < intersection->t && intersection_tmp->t > 0)
			{
				intersection->id = world.cylinders[i].id;
				intersection->id_save = world.cylinders[i].id;
				intersection->t = intersection_tmp->t;
				intersection->type = 'y';
				intersection->reflection_coef = world.cylinders[i].reflection_coef;
				intersection->refraction_coef = world.cylinders[i].refraction_coef;
				intersection->transparence_coef = world.cylinders[i].transparence_coef;
				intersection->color = world.cylinders[i].color;
				intersection->chess = world.cylinders[i].chess;
				intersection->pos = intersection_tmp->pos;
				intersection->normal_v = intersection_tmp->normal_v;
				apply_noise_dist(world, intersection, world.cylinders[i].perlin);
			}
		}
		i++;
	}
}
