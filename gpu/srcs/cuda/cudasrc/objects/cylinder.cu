#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   cylinder.c                                         :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: svilau <marvin@42.fr>                      +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/04/25 10:49:50 by svilau            #+#    #+#             */
/*   Updated: 2017/04/25 14:07:23 by svilau           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

extern "C" {
	#include <rt.h>
	#include <gpu_rt.h>
	#include <equation.h>
}
#include <float.h>

__host__ __device__ int		limit_cylinder(t_cylinder cylinder, t_ray ray,
		t_intersection *intersection_tmp, t_vec3d axis, t_vec3d x, t_eq eq)
{
	double b[2];

	b[0] = vector_dot(ray.dir, axis) * eq.res[0] + vector_dot(x, axis);
	b[1] = vector_dot(ray.dir, axis) * eq.res[1] + vector_dot(x, axis);
	intersection_tmp->normal_v = vector_normalize(
		vector_substract(vector_substract(
				intersection_tmp->pos, cylinder.pos), vector_scalar(axis, b[0])));
	if (cylinder.height <= 0)
		return (1);
	if ((b[0] > 0 && b[0] < cylinder.height)
		|| (b[1] > 0 && b[1] < cylinder.height))
		return (1);
	return (0);
}

__host__ __device__ int		get_cylinder(t_world world, t_cylinder cylinder,
							t_ray ray, t_intersection *intersection_tmp)
{
	t_eq		eq;
	t_vec3d axis;
	t_vec3d x;

 	if (intersection_tmp->id == cylinder.id)
		return (0);
	axis = vector_normalize(vector_substract(cylinder.up, cylinder.pos));
	x = vector_substract(ray.origin, cylinder.pos);
	eq.a = vector_dot(ray.dir, ray.dir) - pow(vector_dot(ray.dir, axis), 2);
	eq.b = 2 * (vector_dot(ray.dir, x) - (vector_dot(ray.dir, axis)
		* vector_dot(x, axis)));
	eq.c = vector_dot(x, x) - pow(vector_dot(x, axis), 2)
		- pow(cylinder.radius, 2);
	second_degres(&eq);
	if (eq.res[0] != NOT_A_SOLUTION)
	{
		intersection_tmp->t = eq.res[0];
		intersection_tmp->pos = vector_add(ray.origin,
			vector_scalar(ray.dir, intersection_tmp->t));
		x = vector_substract(ray.origin, cylinder.pos);
		// intersection_tmp->normal_v = vector_normalize(
		// 	vector_substract(vector_substract(
		// 			intersection_tmp->pos, cylinder.pos),
		// 			vector_scalar(axis, b[0])));
		if (limit_cylinder(cylinder, ray, intersection_tmp, axis, x, eq))
			return (1);
	}
 	intersection_tmp->t = -1;
	return (-1);
}

__host__ __device__ void	get_closest_cylinder(t_world world, t_ray ray,
				t_intersection *intersection, t_intersection *intersection_tmp)
{
	int i;

	i = 0;
	while (i < world.cylinders_len)
	{
		if (get_cylinder(world, world.cylinders[i], ray, intersection_tmp) == 1)
		{
			if (intersection_tmp->t < intersection->t && intersection_tmp->t > 0)
			{
				intersection->id = world.cylinders[i].id;
				intersection->id_save = world.cylinders[i].id;
				intersection->t = intersection_tmp->t;
				intersection->type = 'y';
				intersection->reflection_coef = world.cylinders[i].reflection_coef;
				intersection->refraction_coef = world.cylinders[i].refraction_coef;
				intersection->transparence_coef = world.cylinders[i].transparence_coef;
				intersection->color = world.cylinders[i].color;
				intersection->chess = world.cylinders[i].chess;
				intersection->pos = intersection_tmp->pos;
				intersection->normal_v = intersection_tmp->normal_v;
			}
		}
		i++;
	}
}
