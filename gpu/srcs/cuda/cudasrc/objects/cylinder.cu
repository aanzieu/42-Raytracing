#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   cylinder.c                                         :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: svilau <marvin@42.fr>                      +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/04/25 10:49:50 by svilau            #+#    #+#             */
/*   Updated: 2017/04/25 14:07:23 by svilau           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

extern "C" {
	#include <rt.h>
	#include <gpu_rt.h>
	#include <equation.h>
}
#include <float.h>

__host__ __device__ t_vec3d	get_normal_cylinder(t_cylinder cylinder, t_camera camera, t_ray ray,
												t_intersection intersection)
{
	t_vec3d axis_v;
	t_vec3d	normal_v;
	t_vec3d	x;
	double	m;

	x = vector_substract(ray.origin, cylinder.pos);
	axis_v = vector_normalize(vector_calculate(cylinder.pos, cylinder.up));

	m = vector_dot(ray.dir, axis_v) * intersection.t + vector_dot(x, axis_v);

	normal_v = vector_normalize(vector_substract(vector_substract(intersection.pos, cylinder.pos),
					vector_scalar(axis_v, m)));
	return (normal_v);
}

__host__ __device__ int		get_cylinder(t_cylinder cylinder, t_camera camera, t_ray ray,
											t_intersection *intersection_tmp)
{
	t_vec3d	normal_v;
	t_vec3d	x;
	t_eq	eq;
	normal_v = vector_normalize(vector_calculate(cylinder.pos, cylinder.up));
	x = vector_substract(ray.origin, cylinder.pos);
	eq.a = vector_dot(ray.dir, ray.dir) -
		pow(vector_dot(ray.dir, normal_v), 2);
	eq.b = 2 * (vector_dot(ray.dir, x) -
		(vector_dot(ray.dir, normal_v) * vector_dot(x, normal_v)));
	eq.c = vector_dot(x, x) -
		pow(vector_dot(x, normal_v), 2) - pow(cylinder.radius, 2);
	eq.res = second_degres(eq.a, eq.b, eq.c);
	if(eq.res != NOT_A_SOLUTION)
	{
		intersection_tmp->t = eq.res;
		intersection_tmp->type = 'c';
		return(1);
	}
	intersection_tmp->t = -1.0;
	return(0);
}

__host__ __device__ void	get_closest_cylinder(t_world world, t_ray ray,
				t_intersection *intersection, t_intersection *intersection_tmp)
{
	int i;

	i = 0;
	while (i < world.cylinders_len)
	{
		if (get_cylinder(world.cylinders[i], world.camera, ray, intersection_tmp) == 1)
		{
			if (intersection_tmp->t < intersection->t && intersection_tmp->t != -1 && intersection_tmp->id != intersection->id)
			{
				intersection->id = world.cylinders[i].id;
				intersection->t = intersection_tmp->t;
				intersection->type = intersection_tmp->type;
				intersection->reflexion_coef = world.cylinders[i].reflexion_coef;
				intersection->refraxion_coef = world.cylinders[i].refraxion_coef;
				intersection->color = &world.cylinders[i].color;
				intersection->chess = &world.cylinders[i].chess;
				intersection->pos = vector_add(ray.origin,
					vector_scalar(ray.dir, intersection->t));
				intersection->normal_v = get_normal_cylinder(world.cylinders[i],
											world.camera, ray, *intersection);
			}
		}
		i++;
	}
}
