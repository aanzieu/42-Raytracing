#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   h_cube.c                                           :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: huweber <marvin@42.fr>                     +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/08/23 19:29:36 by huweber           #+#    #+#             */
/*   Updated: 2017/08/23 19:29:39 by huweber          ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

extern "C" {
	#include <rt.h>
	#include "gpu_rt.h"
	#include <vectors.h>
}

/*
**	On envoie le rayon et la structure qui contient le h_cube et la fonction
**	ecrit sur 't_vec3d *intersection' les coordonees du point d'intersection
**	avec le plan
*/

__host__ __device__ t_vec3d get_normal_hollow_cube(t_intersection *intersection,
		t_h_cube cu)
{
  t_vec3d norm;
  t_vec3d pos;

  pos = vector_substract(intersection->pos, cu.pos);
	norm.x = 4. * pow(pos.x, 3) - 10. * pos.x;
  norm.y = 4. * pow(pos.y, 3) - 10. * pos.y;
  norm.z = 4. * pow(pos.z, 3) - 10. * pos.z;
	return (vector_normalize(norm));
}

__host__ __device__ int	get_hollow_cube(t_h_cube cu, t_ray ray,
		t_intersection *intersection_tmp)
{
	double	a[5];
	double	roots[4];
	double	res = DBL_MAX;
	double	nb_roots = 0;
	int			i = 0;

	if (intersection_tmp->id == cu.id)
		return (0);
	ray.origin = vector_calculate(cu.pos, ray.origin);
  a[0] = (pow(ray.dir.x, 4) + pow(ray.dir.y, 4) + pow(ray.dir.z, 4));
  a[1] = (4 * (pow(ray.dir.x, 3) * ray.origin.x + pow(ray.dir.y, 3)
          * ray.origin.y + pow(ray.dir.z, 3) * ray.origin.z));
  a[2] = (6 * (pow(ray.dir.x, 2) * pow(ray.origin.x, 2) + pow(ray.dir.y, 2)
          * pow(ray.origin.y, 2) + pow(ray.dir.z, 2) * pow(ray.origin.z, 2))
          - 5 * vector_magnitude(ray.dir));
  a[3] = (4 * (pow(ray.origin.x, 3) * ray.dir.x + pow(ray.origin.y, 3)
          * ray.dir.y + pow(ray.origin.z, 3) * ray.dir.z)
          - 10 * (vector_dot(ray.dir, ray.origin)));
  a[4] = (pow(ray.origin.x, 4) + pow(ray.origin.y, 4)
          + pow(ray.origin.z, 4) - 5 * (vector_magnitude(ray.origin))  + 11.8);
	if ((nb_roots = solve_quartic(a, roots)) > 0)
	{
		while (i < nb_roots)
		{
			if (roots[i] < res && roots[i] > 0.1)//SURFACE_TOLERANCE)
				res = roots[i];
			i++;
		}
		if (res != DBL_MAX)
		{
			intersection_tmp->t = res;
			return (1);
		}
	}
	return (0);
}

__host__ __device__ void	get_closest_hollow_cube(t_world world, t_ray ray,
			t_intersection *intersection, t_intersection *intersection_tmp)
{
	int i;

	i = 0;
	while (i < world.h_cubes_len)
	{
		if (get_hollow_cube(world.h_cubes[i], ray, intersection_tmp) == 1)
		{
			if (intersection_tmp->t < intersection->t && intersection_tmp->t != -1)
			{
				intersection->id = world.h_cubes[i].id;
				intersection->id_save = world.h_cubes[i].id;
				intersection->t = intersection_tmp->t;
				intersection->type = 'u';
				intersection->reflection_coef = world.h_cubes[i].reflection_coef;
				intersection->reflection_coef = world.h_cubes[i].reflection_coef;
				intersection->transparence_coef = world.h_cubes[i].transparence_coef;
				intersection->color = world.h_cubes[i].color;
				intersection->chess = world.h_cubes[i].chess;
				intersection->pos =
					vector_add(ray.origin, vector_scalar(ray.dir, intersection->t));
				intersection->normal_v =
					get_normal_hollow_cube(intersection, world.h_cubes[i]);
			}
		}
		i++;
	}
}
