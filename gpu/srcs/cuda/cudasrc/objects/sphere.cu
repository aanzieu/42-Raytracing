#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   sphere.c                                           :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: svilau <marvin@42.fr>                      +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/04/25 10:49:50 by svilau            #+#    #+#             */
/*   Updated: 2017/04/26 14:07:23 by svilau           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

extern "C" {
	#include <rt.h>
	#include <gpu_rt.h>
	#include <vectors.h>
	#include <equation.h>
}

__host__ __device__ static t_vec3d			get_normal_sphere(t_sphere sphere,
		t_intersection intersection)
{
	t_vec3d normal;

	normal = vector_normalize(vector_calculate(sphere.pos, intersection.pos));
	return (normal);
}

/*
**	On envoie le rayon et la structure qui contient la sphere et la fonction
**	ecrit sur 't_vec3d *intersection' les coordonees du point d'intersection
**	avec la sphere
*/

__host__ __device__ static double			get_sphere(t_sphere sphere,
		t_ray ray, t_intersection *intersection_tmp)
{
	t_vec3d	x;
	t_eq	eq;

	x = vector_calculate(sphere.pos, ray.origin);
	eq.a = vector_dot(ray.dir, ray.dir);
	eq.b = 2 * vector_dot(ray.dir, x);
	eq.c = vector_dot(x, x) - pow(sphere.radius, 2);
	eq.res = second_degres(eq.a, eq.b, eq.c);
	if(eq.res != NOT_A_SOLUTION && intersection_tmp->id != sphere.id)
	{
		intersection_tmp->t = eq.res;
		intersection_tmp->type = 's';
		if (sphere.refraxion_coef != 0)
			intersection_tmp->id = sphere.id;
		return (1);
	}
	return (0);
}

__host__ __device__ void	get_closest_sphere(t_world world, t_ray ray,
				t_intersection *intersection, t_intersection *intersection_tmp)
{
	int i;

	i = 0;
	while (i < world.spheres_len)
	{
		if (get_sphere(world.spheres[i], ray, intersection_tmp) == 1)// && intersection_tmp->id != intersection->id)
		{
			if (intersection_tmp->t < intersection->t && intersection_tmp->t != -1)
			{
				intersection->id = intersection_tmp->id;
				intersection->t = intersection_tmp->t;
				intersection->type = intersection_tmp->type;
				intersection->reflexion_coef = world.spheres[i].reflexion_coef;
				intersection->refraxion_coef = world.spheres[i].refraxion_coef;
				intersection->color = &world.spheres[i].color;
				intersection->pos = vector_add(ray.origin, vector_scalar(ray.dir,
							intersection_tmp->t));
				intersection->normal_v = get_normal_sphere(world.spheres[i],
														*intersection);
			}
		}
		i++;
	}
}
