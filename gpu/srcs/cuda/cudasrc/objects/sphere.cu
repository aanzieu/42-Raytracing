#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   sphere.c                                           :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: svilau <marvin@42.fr>                      +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/04/25 10:49:50 by svilau            #+#    #+#             */
/*   Updated: 2017/04/26 14:07:23 by svilau           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

extern "C" {
	#include <rt.h>
	#include "gpu_rt.h"
	#include <vectors.h>
}

__host__ __device__ t_vec3d	get_normal_sphere(t_sphere sphere, t_intersection intersection)
{
	t_vec3d normal;

	normal = vector_normalize(vector_calculate(sphere.pos,
															intersection.pos));
	return (normal);
}

__host__ __device__ void	get_determinant_sphere(t_sphere s, t_ray r, t_2deg *equation)
{
	t_vec3d x;

	x = vector_calculate(s.pos, r.origin);
	equation->a = vector_dot(r.dir, r.dir);
	equation->b = 2 * vector_dot(r.dir, x);
	equation->c = vector_dot(x, x) - pow(s.radius, 2);
	equation->det = pow(equation->b, 2) - (4 * (equation->a) * (equation->c));
}

/*
**	On envoie le rayon et la structure qui contient la sphere et la fonction
**	ecrit sur 't_vec3d *intersection' les coordonees du point d'intersection
**	avec la sphere
*/

__host__ __device__ static int	get_sphere(t_sphere sphere, t_ray ray, t_intersection *intersection_tmp)
{
	t_2deg	equation;
	double	t1;
	double	t2;

	get_determinant_sphere(sphere, ray, &equation);
	if (equation.det >= 0)
	{
		t1 = ((-1) * equation.b + sqrt(equation.det)) / (2 * equation.a);
		t2 = ((-1) * equation.b - sqrt(equation.det)) / (2 * equation.a);
		if (t1 <= t2 && t1 > 0)
		{
			intersection_tmp->t = t1;
			intersection_tmp->type = 's';
			return (1);
		}
		else if (t2 > 0)
		{
			intersection_tmp->t = t2;
			intersection_tmp->type = 's';
			return (1);
		}
	}
	return (0);
}

__host__ __device__ void	get_closest_sphere(t_world world, t_ray ray,
				t_intersection *intersection, t_intersection *intersection_tmp)
{
	int i;

	i = 0;
	while (i < world.spheres_len)
	{
		if(get_sphere(world.spheres[i], ray, intersection_tmp) == 1)
		{
			if (intersection_tmp->t < intersection->t)
			{
				intersection->t = intersection_tmp->t;
				intersection->type = intersection_tmp->type;
				intersection->color = &world.spheres[i].color;
				intersection->pos = vector_add(vector_scalar(ray.dir, intersection_tmp->t), ray.origin);
				intersection->normal_v = get_normal_sphere(world.spheres[i],
														*intersection);
			}
		}
		i++;		
	}
}
