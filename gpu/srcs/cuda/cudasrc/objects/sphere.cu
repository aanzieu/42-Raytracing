#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   sphere.c                                           :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: svilau <marvin@42.fr>                      +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/04/25 10:49:50 by svilau            #+#    #+#             */
/*   Updated: 2017/04/26 14:07:23 by svilau           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

extern "C" {
	#include <rt.h>
	#include "gpu_rt.h"
	#include <vectors.h>
	#include <equation.h>
}

__host__ __device__ t_vec3d	get_normal_sphere(t_sphere sphere,
		t_intersection intersection)
{
	t_vec3d normal;

	normal = vector_normalize(vector_calculate(sphere.pos, intersection.pos));
	return (normal);
}

/*
**	On envoie le rayon et la structure qui contient la sphere et la fonction
**	ecrit sur 't_vec3d *intersection' les coordonees du point d'intersection
**	avec la sphere
*/

__host__ __device__ static double get_sphere(t_sphere sphere,
		t_ray ray, t_intersection *intersection_tmp)
{
	t_vec3d x;
	t_eq	eq;

	x = vector_calculate(sphere.pos, ray.origin);
	eq.a = vector_dot(ray.dir, ray.dir);
	eq.b = 2 * vector_dot(ray.dir, x);
	eq.c = vector_dot(x, x) - pow(sphere.radius, 2);
	return ((intersection_tmp->t = second_degres(eq.a, eq.b, eq.c)));
}

__host__ __device__ void	get_closest_sphere(t_world world, t_ray ray,
				t_intersection *intersection, t_intersection *intersection_tmp)
{
	int i;

	i = 0;
	while (i < world.spheres_len)
	{
		if(get_sphere(world.spheres[i], ray, intersection_tmp) > ZERO_DP)
		{
			intersection_tmp->type = 's';
			if (intersection_tmp->t < intersection->t)
			{
				intersection->t = intersection_tmp->t;
				intersection->type = intersection_tmp->type;
				intersection->reflexion_coef = world.spheres[i].reflexion_coef;				
				intersection->color = &world.spheres[i].color;
				intersection->pos = vector_add(vector_scalar(ray.dir, intersection_tmp->t), ray.origin);
				intersection->normal_v = get_normal_sphere(world.spheres[i],
														*intersection);
			}
		}
		i++;		
	}
}
