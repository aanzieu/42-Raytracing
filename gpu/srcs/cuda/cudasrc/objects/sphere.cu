#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   sphere.c                                           :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: svilau <marvin@42.fr>                      +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/04/25 10:49:50 by svilau            #+#    #+#             */
/*   Updated: 2017/04/26 14:07:23 by svilau           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

extern "C" {
	#include <rt.h>
	#include <gpu_rt.h>
	#include <vectors.h>
	#include <equation.h>
}

/*
**	On envoie le rayon et la structure qui contient la sphere et la fonction
**	ecrit sur 't_vec3d *intersection' les coordonees du point d'intersection
**	avec la sphere
*/

__host__ __device__ double			get_sphere(t_sphere sphere,
		t_ray ray, t_intersection *intersection_tmp)
{
	t_vec3d	x;
	t_eq	eq;

	if (sphere.id == intersection_tmp->id)
		return (0);
	x = vector_calculate(sphere.pos, ray.origin);
	eq.a = vector_dot(ray.dir, ray.dir);
	eq.b = 2 * vector_dot(ray.dir, x);
	eq.c = vector_dot(x, x) - pow(sphere.radius, 2);
	second_degres(&eq);
	if(eq.res[0] != NOT_A_SOLUTION)
	{
		// tfar = eq.res[1] < eq.res[2] ? eq.res[2] : eq.res[1];
		// printf("%f\n", sphere.transparence_coef);
		intersection_tmp->id = sphere.id;
		intersection_tmp->t = eq.res[0];
		// intersection_tmp->t1 = eq.res[0];//tnear; //intersection_tmp->t;
		// intersection_tmp->t2 = eq.res[1];//tfar;
		return (1);
	}
	return (0);
}

__host__ __device__ void	get_closest_sphere(t_world world, t_ray ray,
				t_intersection *intersection, t_intersection *intersection_tmp)
{
	int i;

	i = 0;
	while (i < world.spheres_len)
	{
		if (get_sphere(world.spheres[i], ray, intersection_tmp) == 1)
		{
			if (intersection_tmp->t < intersection->t && intersection_tmp->t != -1)
			{
				intersection->type = 's';
				intersection->t = intersection_tmp->t;
				intersection->id = world.spheres[i].id;
				intersection->id_save = world.spheres[i].id;
				intersection->reflection_coef = world.spheres[i].reflection_coef;
				intersection->refraction_coef = world.spheres[i].refraction_coef;
				intersection->transparence_coef = world.spheres[i].transparence_coef;
				intersection->color = world.spheres[i].color;
				intersection->chess = world.spheres[i].chess;
				intersection->pos = vector_add(ray.origin, vector_scalar(ray.dir,
					intersection_tmp->t));
				intersection->normal_v = vector_normalize(
					vector_calculate(world.spheres[i].pos, intersection->pos));
			}
		}
		i++;
	}
}
