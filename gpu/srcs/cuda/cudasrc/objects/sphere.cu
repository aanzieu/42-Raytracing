#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   sphere.c                                           :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: svilau <marvin@42.fr>                      +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/04/25 10:49:50 by svilau            #+#    #+#             */
/*   Updated: 2017/04/26 14:07:23 by svilau           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

extern "C" {
	#include "../../../../includes/rtv1.h"
	#include "../../../../includes/vectors.h"
}

__host__ __device__ t_vec3d	get_normal_sphere(t_sphere sphere, t_intersection intersection)
{
	t_vec3d normal;

	normal = vector_normalize(vector_calculate(sphere.pos,
															intersection.pos));
	return (normal);
}

__host__ __device__ void	get_determinant_sphere(t_sphere sphere, t_ray ray, t_2deg *equation)
{
	equation->a = pow(ray.dir.x, 2) + pow(ray.dir.y, 2) + pow(ray.dir.z, 2);
	equation->b = 2 * (ray.dir.x * (ray.origin.x - sphere.pos.x) +
				ray.dir.y * (ray.origin.y - sphere.pos.y) +
				ray.dir.z * (ray.origin.z - sphere.pos.z));
	equation->c = (pow((ray.origin.x - sphere.pos.x), 2) +
			pow((ray.origin.y - sphere.pos.y), 2) +
			pow((ray.origin.z - sphere.pos.z), 2)) -
			pow(sphere.radius, 2);
	equation->det = pow(equation->b, 2) - (4 * (equation->a) * (equation->c));
}

/*
**	On envoie le rayon et la structure qui contient la sphere et la fonction
**	ecrit sur 't_vec3d *intersection' les coordonees du point d'intersection
**	avec la sphere
*/

__host__ __device__ double		get_sphere(t_sphere sphere, t_ray ray, t_intersection *intersection_tmp)
{
	t_2deg	equation;
	float	t1;
	float	t2;

	get_determinant_sphere(sphere, ray, &equation);
	if (equation.det >= 0)
	{
		t1 = ((-1) * equation.b + sqrt(equation.det)) / (2 * equation.a);
		t2 = ((-1) * equation.b - sqrt(equation.det)) / (2 * equation.a);
		if (t1 <= t2 && t1 > 0.0000001)
		{
			intersection_tmp->t = t1;
			intersection_tmp->type = 's';
			return (1);
		}
		else if (t2 > 0.0000001)
		{
			intersection_tmp->t = t2;
			intersection_tmp->type = 's';
			return (1);
		}
	}
	return (0);
}

__host__ __device__ double	get_closest_sphere(t_world world, t_ray ray,
				t_intersection *intersection, t_intersection *intersection_tmp)
{
	int i;

	i = 0;
	while (i < world.spheres_len)
	{
		if(get_sphere(world.spheres[i], ray, intersection_tmp) == 1)
		{
			if (intersection_tmp->t < intersection->t)
			{
				intersection->t = intersection_tmp->t;
				intersection->type = intersection_tmp->type;
				intersection->color = &world.spheres[i].color;
				intersection->pos = vector_add(ray.origin,
					vector_scalar(ray.dir, intersection_tmp->t));
				intersection->normal_v = get_normal_sphere(world.spheres[i],
														*intersection);
			}
		}
		i++;		
	}
	return (0);
}
