#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   sphere.c                                           :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: svilau <marvin@42.fr>                      +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/04/25 10:49:50 by svilau            #+#    #+#             */
/*   Updated: 2017/04/26 14:07:23 by svilau           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

extern "C" {
	#include <rt.h>
	#include <gpu_rt.h>
	#include <vectors.h>
	#include <equation.h>
}

/*
**	On envoie le rayon et la structure qui contient la sphere et la fonction
**	ecrit sur 't_vec3d *intersection' les coordonees du point d'intersection
**	avec la sphere
*/

__host__ __device__ double			get_sphere(t_world world, t_sphere sphere,
		t_ray ray, t_intersection *intersection_tmp)
{
	t_vec3d	x;
	t_eq	eq;

	if (sphere.id == intersection_tmp->id)
		return (0);
	x = vector_calculate(sphere.pos, ray.origin);
	eq.a = vector_dot(ray.dir, ray.dir);
	eq.b = 2 * vector_dot(ray.dir, x);
	eq.c = vector_dot(x, x) - pow(sphere.radius, 2);
	second_degres(&eq);
	if(eq.res[0] != NOT_A_SOLUTION)
	{
		intersection_tmp->id = sphere.id;
		intersection_tmp->t = eq.res[0];
		return (1);
	}
	return (0);
}

__host__ __device__ void	get_closest_sphere(t_world world, t_ray ray,\
				t_intersection *intersection, t_intersection *intersection_tmp)
{
	int i;

	i = 0;
	while (i < world.spheres_len)
	{
		if (get_sphere(world, world.spheres[i], ray, intersection_tmp) == 1)
		{
			if (intersection_tmp->t < intersection->t && intersection_tmp->t != -1)
			{
				intersection->type = 's';
				intersection->t = intersection_tmp->t;
				intersection->id = world.spheres[i].id;
				intersection->id_save = world.spheres[i].id;
				intersection->reflection_coef = world.spheres[i].reflection_coef;
				intersection->refraction_coef = world.spheres[i].refraction_coef;
				intersection->transparence_coef = world.spheres[i].transparence_coef;
				intersection->chess = world.spheres[i].chess;
				intersection->pos = vector_add(ray.origin, vector_scalar(ray.dir,\
					intersection_tmp->t));
				intersection->color = world.spheres[i].color;
				intersection->normal_v = vector_normalize(\
					vector_calculate(world.spheres[i].pos, intersection->pos));
				apply_noise_dist(world, intersection, world.spheres[i].perlin);
			}
		}
		i++;
	}
}
