#include "hip/hip_runtime.h"

extern "C" {
#include "../../../../includes/rt.h"
#include "../../cudaheader/gpu_rt.h"
#include <vectors.h>
#include <equation.h>
#include <float.h>
}

__host__ __device__ static void			get_normal_mobius(t_intersection *inter, t_mobius m, t_ray ray)
{

//	t_vec3d       	dt;
//	t_vec3d       	dv;
//	double			v;
//	double			t;

//	v = atan(inter->pos.y / inter->pos.x);
//	t = (2.0 * inter->pos.z) / sin((v / 2.0));
//	if (!(v > 0 && v < 2 * M_PI && t >= -1.0 && t <= 1.0))
//		inter->t = -1;//INFINITY;
//	dt.x = (1.0 / 2.0) * cos(v) * cos(v / 2.0);
//	dt.y = (1.0 / 2.0) * sin(v) * cos(v / 2.0);
//	dt.z = (1.0 / 2.0) * sin(v / 2.0);
//	dv.y = -sin(v) - (t / 2.0) * sin(v) * cos(v / 2.0) - (t / 4.0) * cos(v) *
//		sin(v / 2);
//	dv.y = cos(v) + (t / 2.0) * cos(v) * cos(v / 2.0) - (t / 4.0) * sin(v) *
//		sin(v / 2);
//	dv.z = (t / 4.0) * cos(v / 2.0);
//	inter->normal_v.x = dt.y * dv.z - dt.z * dv.y;
//	inter->normal_v.y = dt.z * dv.x - dt.x * dv.z;
//	inter->normal_v.z = dt.x * dv.y - dt.y * dv.x;
		t_vec3d real;
		t_vec3d ret;

		real = vector_substract(inter->pos, m.pos);
		ret.x = -2 * m.radius * real.z + 2 * real.x * real.y - 4 * real.x * real.z;
		ret.y = -(m.radius * m.radius) + real.x * real.x + 3 * real.y * real.y - 4 * real.y * real.z + real.z * real.z;
		ret.z = -2 * m.radius * real.x - 2 * real.x * real.x - 2 * real.y * real.y + 2 * real.y * real.z;
		inter->normal_v = vector_normalize(ret);//inter->normal_v);
}

/*
 **	On envoie le rayon et la structure qui contient la sphere et la fonction
 **	ecrit sur 't_vec3d *intersection' les coordonees du point d'intersection
 **	avec la sphere
 */

__host__ __device__ static int			get_mobius(t_mobius m, t_ray ray, t_intersection *intersection_tmp)
{
	double	s[4];
	double  res;
	double	 a = m.radius;
	double	 b = ray.origin.x;
	double c = ray.dir.x;
	double d = ray.origin.y;
	double e = ray.dir.y;
	double f = ray.origin.z;
	double g = ray.dir.z;

	s[3] = c * c * e + e * e * e - 2 * c * c * g - 2 * e * e * g + e * g * g;
	s[0] = (b * b * d + d * d * d - 2 * b * b * f - 2 * d * d * f + d * f * f - 2 * b * f * a - d * a * a) / s[3];
	s[1] = (e * b * b - 2 * g * b * b + 2 * c * b * d + 3 * e * d * d - 2 * g * d * d - 4 * c * b * f - 4 * e * d * f + 2 * g * d * f + e * f * f - 2 * g * b * a - 2 * c * f * a - e * a * a) / s[3];
	s[2] = (2 * c * e * b - 4 * c * g * b + c * c * d + 3 * e * e * d - 4 * e * g * d + g * g * d - 2 * c * c * f - 2 * e * e * f + 2 * e * g * f - 2 * c * g * a) / s[3];

	res = solver_n_degree(s, 4, m, ray);
	if(res != NOT_A_SOLUTION)
	{
		intersection_tmp->t = res;
		intersection_tmp->type = 'm';
		return(1);
	}
	intersection_tmp->t = -1.0;
	return(0);
}



__host__ __device__ void	get_closest_mobius(t_world world, t_ray ray,
		t_intersection *intersection, t_intersection *intersection_tmp)
{
	int i;

	i = 0;
	while (i < world.mobius_len)
	{
//		printf("%lf posx \n", world.mobius[i].reflexion_coef);
		if(get_mobius(world.mobius[i], ray, intersection_tmp) == 1)
		{
			if (intersection_tmp->t < intersection->t && intersection_tmp->t != -1)
			{
				intersection->pos = vector_add(ray.origin,
						vector_scalar(ray.dir, intersection_tmp->t));
				intersection->t = intersection_tmp->t;
				intersection->type = intersection_tmp->type;
				intersection->reflexion_coef = world.mobius[i].reflexion_coef;
				intersection->color = &world.mobius[i].color;
			//	if(intersection->t != INFINITY)
				get_normal_mobius(intersection, world.mobius[i], ray);
			}
		}
		//	else
		//			intersection->t = INFINITY;
		i++;
	}
}
