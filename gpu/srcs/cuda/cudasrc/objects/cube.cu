#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   cube.cu                                            :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: huweber <marvin@42.fr>                     +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/08/23 19:29:36 by huweber           #+#    #+#             */
/*   Updated: 2017/08/24 18:02:22 by huweber          ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

extern "C" {
#include "../../../../includes/rt.h"
#include "../../cudaheader/gpu_rt.h"
#include <vectors.h>
#include <equation.h>
#include <float.h>
#include <math.h>
}

__host__ __device__ t_vec3d	get_normal_cube(t_cube cu, t_ray ray,
	t_intersection intersection)
{
	t_vec3d		normal;
	t_vec3d		c;
	t_vec3d		p;
	t_vec3d		d;
	double		bias;
	t_vec3d		vmin = cu.min;//(t_vec3d){-1, -1, -1};//cu.vec1;
	t_vec3d		vmax = cu.max;//(t_vec3d){1, 1, 1};//cu.vec2;

  c = vector_scalar(vector_add(vmin, vmax), 0.5f);
  p = vector_substract(intersection.pos, c);
  d = vector_scalar(vector_substract(vmin, vmax), 0.5f);
  bias = 1.000001f;

	normal.x = (p.x / fabs(d.x) * bias);
	normal.y = (p.y / fabs(d.y) * bias);
	normal.z = (p.z / fabs(d.z) * bias);
	normal = vector_normalize(normal);
//	normal = (t_vec3d){0, 0, 1};
	return (normal);
}

__host__ __device__ int			get_cube(t_cube cu, t_ray ray,
	t_intersection *intersection_tmp)
{
	t_vec3d		bounds[2];
	int				sign[3];
	t_vec3d		invdir;
	double		tmin, tmax, tymin, tymax, tzmin, tzmax, t;

	if (intersection_tmp->id == cu.id)
		return (-1);
	invdir.x = 1.0 / ray.dir.x;
	invdir.y = 1.0 / ray.dir.y;
	invdir.z = 1.0 / ray.dir.z;
	bounds[0] = cu.min;//(t_vec3d){-1, -1, -1};//cu.vec1;
	bounds[1] = cu.max;//(t_vec3d){1, 1, 1};//cu.vec2;
	sign[0] = (invdir.x < 0) ? 1 : 0;
	sign[1] = (invdir.y < 0) ? 1 : 0;
	sign[2] = (invdir.z < 0) ? 1 : 0;
	tmin = (bounds[sign[0]].x - ray.origin.x) * invdir.x;
	tmax = (bounds[1 - sign[0]].x - ray.origin.x) * invdir.x;
	tymin = (bounds[sign[1]].y - ray.origin.y) * invdir.y;
	tymax = (bounds[1 - sign[1]].y - ray.origin.y) * invdir.y;
	if ((tmin > tymax) || (tymin > tmax))
			return (-1);
	if (tymin > tmin)
		tmin = tymin;
	if (tymax < tmax)
		tmax = tymax;
	tzmin = (bounds[sign[2]].z - ray.origin.z) * invdir.z;
	tzmax = (bounds[1 - sign[2]].z - ray.origin.z) * invdir.z;
	if ((tmin > tzmax) || (tzmin > tmax))
		return (-1);
	if (tzmin > tmin)
	tmin = tzmin;
	if (tzmax < tmax)
	tmax = tzmax;
	t = tmin;
	if (t <= 1.00000001)
	{
			t = tmax;
			if (t <= 1.00000001)
				return (-1);
	}
	intersection_tmp->t = t;
	return (1);
}

__host__ __device__ void	get_closest_cube(t_world world, t_ray ray,
		t_intersection *intersection, t_intersection *intersection_tmp)
{
	int i;

	i = 0;
//printf("%f %f %f\n", world.cube[i].min.x, world.cube[i].min.y, world.cube[i].min.z);
	while (i < world.cubes_len)
	{
		if (get_cube(world.cubes[i], ray, intersection_tmp) == 1)
		{
			if (intersection_tmp->t < intersection->t && intersection_tmp->t != -1)
			{
				intersection->t = intersection_tmp->t;
				intersection->type = 'b';
				intersection->reflection_coef = world.cubes[i].reflection_coef;
				intersection->reflection_coef = world.cubes[i].reflection_coef;
				intersection->transparence_coef = world.cubes[i].transparence_coef;
				intersection->id = intersection_tmp->id;
				intersection->color = world.cubes[i].color;
				intersection->chess = world.cubes[i].chess;
				intersection->pos =
					vector_add(ray.origin, vector_scalar(ray.dir, intersection->t));
				intersection->normal_v =
					get_normal_cube(world.cubes[i], ray, *intersection);
			}
		}
		i++;
	}
}
