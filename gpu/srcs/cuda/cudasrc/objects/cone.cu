#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   cone.c                                             :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: svilau <marvin@42.fr>                      +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2016/10/20 10:49:50 by svilau            #+#    #+#             */
/*   Updated: 2017/03/17 14:07:23 by svilau           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

extern "C" {
#include "../../cudaheader/gpu_rt.h"
}

__host__ __device__ t_vec3d		get_normal_cone(t_cone cone, t_ray ray,
		t_intersection intersection)
{
	t_vec3d axis_v;
	t_vec3d	normal_v;
	t_vec3d	x;
	double	m;
	double	k;

	k = cone.half_angle;
	x = vector_substract(ray.origin, cone.pos);
	axis_v = vector_normalize(vector_calculate(cone.pos, cone.up));
	m = vector_dot(ray.dir, vector_scalar(axis_v, intersection.t))
		+ vector_dot(x, axis_v);
	normal_v = vector_normalize(
			vector_substract(
				vector_substract(
					intersection.pos, cone.pos),
				vector_scalar(
					vector_scalar(
						axis_v, m), 1 + (k * k))));
	return (normal_v);
}

__host__ __device__ static int		limit_cone(t_cone cone, t_ray ray,
		t_intersection *intersection_tmp, t_vec3d axis_v, t_vec3d x, t_eq eq)
{
	double b[2];
	double k;

	k = cone.half_angle;
	b[0] = vector_dot(ray.dir, vector_scalar(axis_v, eq.res[0]))
	 	+ vector_dot(x, axis_v);
	b[1] = vector_dot(ray.dir, vector_scalar(axis_v, eq.res[1]))
	 	+ vector_dot(x, axis_v);
	intersection_tmp->pos = vector_add(ray.origin,
		vector_scalar(ray.dir, eq.res[0]));
	intersection_tmp->normal_v = vector_normalize(vector_substract(
		vector_substract(intersection_tmp->pos, cone.pos),
		vector_scalar(vector_scalar(axis_v, b[0]), 1 + (k * k))));
	if (cone.height <= 0)
		return (1);
	if ((b[0] > 0 && b[0] < cone.height) || (b[1] > 0 && b[1] < cone.height))
		if (b[0] > 0 && b[0] < cone.height)
			return (1);
	return (0);
}

__host__ __device__ static int		get_cone(t_cone cone, t_ray ray,
	t_intersection *intersection_tmp)
{
	t_vec3d axis_v;
	t_vec3d	normal;
	t_vec3d x;
	t_eq		eq;
	double	k;

	if (intersection_tmp->id == cone.id)
		return (0);
	normal = vector_normalize(vector_calculate(cone.pos, cone.up));
	x = vector_calculate(cone.pos, ray.origin);
	k = cone.half_angle;
	eq.a = vector_dot(ray.dir, ray.dir)
		- (1 + k * k) * pow(vector_dot(ray.dir, normal), 2);
	eq.b = 2 * (vector_dot(ray.dir, x)
			- (1 + k * k) * (vector_dot(ray.dir, normal) * vector_dot(x, normal)));
	eq.c = vector_dot(x, x)
		- (1 + k * k) * pow(vector_dot(x, normal), 2);
	second_degres(&eq);
	if(eq.res[0] != NOT_A_SOLUTION)
	{
		intersection_tmp->t = eq.res[0];
		x = vector_substract(ray.origin, cone.pos);
		axis_v = vector_normalize(vector_calculate(cone.pos, cone.up));
		if (limit_cone(cone, ray, intersection_tmp, axis_v, x, eq) == 1)
			return (1);
	}
	intersection_tmp->t = -1.0;
	return (0);
}

__host__ __device__ void			get_closest_cone(t_world world, t_ray ray,
		t_intersection *intersection, t_intersection *intersection_tmp)
{
	int i;

	i = 0;
	while (i < world.cones_len)
	{
		if (get_cone(world.cones[i], ray, intersection_tmp) == 1)
		{
			if (intersection_tmp->t < intersection->t && intersection_tmp->t != -1)
			{
				intersection->type = 'x';
				intersection->id = world.cones[i].id;
				intersection->t = intersection_tmp->t;
				intersection->reflection_coef = world.cones[i].reflection_coef;
				intersection->refraction_coef = world.cones[i].refraction_coef;
				intersection->transparence_coef = world.cones[i].transparence_coef;
				intersection->color = world.cones[i].color;
				intersection->chess = world.cones[i].chess;
				intersection->pos = intersection_tmp->pos;
				intersection->normal_v = intersection_tmp->normal_v;
			}
		}
		i++;
	}
}
