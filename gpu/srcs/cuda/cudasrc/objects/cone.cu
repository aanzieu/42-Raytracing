#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   cone.c                                             :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: svilau <marvin@42.fr>                      +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2016/10/20 10:49:50 by svilau            #+#    #+#             */
/*   Updated: 2017/03/17 14:07:23 by svilau           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

extern "C" {
#include "../../cudaheader/gpu_rt.h"
}

/*
__host__ __device__ void		get_dist_cone_face(t_cone cone,
	t_ray ray, t_intersection intersection)
{
	double	t;
	double	denominator;
	t_vec3d	x;
	double	n;
	double tmp;

	if(cone.height > 0.0f)
	{
		tmp = vector_dot(intersection.normal_v, ray.dir);
		if (tmp < 0.0f || tmp > cone.height)
		{
			intersection.normal_v = vector_normalize(
				vector_calculate(cone.pos, cone.up));
			denominator = vector_dot(ray.dir, intersection.normal_v);
			if (denominator != 0)
			{
				x = vector_scalar(vector_calculate(cone.pos, ray.origin), -1);
				n = vector_dot(x, intersection.normal_v);
				t = n / denominator;
				if (t > 0.0000001)
				{
				intersection.t = t;
				if (denominator > 0)
					intersection.normal_v = vector_scalar(intersection.normal_v, -1);
				}
			}
		}
	}
	intersection.t = -1;
}
*/
/*
__host__ __device__ int		limit_cone(t_cone cone, t_ray ray,
		t_intersection *intersection_tmp, t_vec3d axis_v, t_vec3d x, t_eq eq)
{
	double b[2];
	double k;

	k = cone.half_angle;
	b[0] = vector_dot(ray.dir, vector_scalar(axis_v, eq.res[0]))
	 	+ vector_dot(x, axis_v);
	b[1] = vector_dot(ray.dir, vector_scalar(axis_v, eq.res[1]))
	 	+ vector_dot(x, axis_v);
	intersection_tmp->pos = vector_add(ray.origin,
		vector_scalar(ray.dir, eq.res[0]));
	intersection_tmp->t = eq.res[0];
	if ((b[0] > 0 && b[0] < cone.height) || (b[1] > 0 && b[1] < cone.height))
	{
	if (b[0] > 0 && b[0] < cone.height)
		intersection_tmp->normal_v = vector_normalize(vector_substract(
			vector_substract(intersection_tmp->pos, cone.pos),
				vector_scalar(vector_scalar(axis_v, b[0]), 1 + (k * k))));
	else
		if (b[1] > 0 && b[1] < cone.height)
			intersection_tmp->normal_v = (t_vec3d){0, 0, 0};
		if (cone.refraction_coef != 0 || cone.reflection_coef != 0)
			intersection_tmp->id = cone.id;
		return (1);
	}
	return (0);
}
*/

__host__ __device__ t_vec3d		get_normal_cone(t_cone cone, t_ray ray,
		t_intersection intersection)
{
	t_vec3d axis_v;
	t_vec3d	normal_v;
	t_vec3d	x;
	double	m;
	double	k;

	k = cone.half_angle;
	x = vector_substract(ray.origin, cone.pos);
	axis_v = vector_normalize(vector_calculate(cone.pos, cone.up));
	m = vector_dot(ray.dir, vector_scalar(axis_v, intersection.t))
		+ vector_dot(x, axis_v);
	normal_v = vector_normalize(
			vector_substract(
				vector_substract(
					intersection.pos, cone.pos),
				vector_scalar(
					vector_scalar(
						axis_v, m), 1 + (k * k))));
	return (normal_v);
}

__host__ __device__ int		limit_cone(t_cone cone, t_ray ray,
		t_intersection *intersection_tmp, t_vec3d axis_v, t_vec3d x, t_eq eq)
{
	double b[2];
	double k;

	k = cone.half_angle;
	b[0] = vector_dot(ray.dir, vector_scalar(axis_v, eq.res[0]))
	 	+ vector_dot(x, axis_v);
	b[1] = vector_dot(ray.dir, vector_scalar(axis_v, eq.res[1]))
	 	+ vector_dot(x, axis_v);
	intersection_tmp->pos = vector_add(ray.origin,
		vector_scalar(ray.dir, eq.res[0]));
	intersection_tmp->t = eq.res[0];
	if ((b[0] > 0 && b[0] < cone.height) || (b[1] > 0 && b[1] < cone.height))
	{
		if (b[0] > 0 && b[0] < cone.height)
{
			intersection_tmp->normal_v = vector_normalize(vector_substract(
				vector_substract(intersection_tmp->pos, cone.pos),
				vector_scalar(vector_scalar(axis_v, b[0]), 1 + (k * k))));
				return (1);
}
		else
		{
			if (b[1] > 0 && b[1] < cone.height) // inside
				intersection_tmp->normal_v = (t_vec3d){0, 0, 0};
			return (1);
		}
	}
	return (0);
}

__host__ __device__ static int		get_cone(t_cone cone, t_ray ray,
	t_intersection *intersection_tmp)
{
	t_vec3d x;
	t_vec3d axis_v;
	t_vec3d	normal;
	t_eq		eq;
	double	k;

	if (intersection_tmp->id == cone.id)
		return (0);
	normal = vector_normalize(vector_calculate(cone.pos, cone.up));
	x = vector_calculate(cone.pos, ray.origin);
	k = cone.half_angle;
	eq.a = vector_dot(ray.dir, ray.dir)
		- (1 + k * k) * pow(vector_dot(ray.dir, normal), 2);
	eq.b = 2 * (vector_dot(ray.dir, x)
			- (1 + k * k) * (vector_dot(ray.dir, normal) * vector_dot(x, normal)));
	eq.c = vector_dot(x, x)
		- (1 + k * k) * pow(vector_dot(x, normal), 2);
	second_degres(&eq);
	if(eq.res[0] != NOT_A_SOLUTION)
	{
		intersection_tmp->t = eq.res[0];
		x = vector_substract(ray.origin, cone.pos);
		axis_v = vector_normalize(vector_calculate(cone.pos, cone.up));
		if (limit_cone(cone, ray, intersection_tmp, axis_v, x, eq) == 1)
			return (1);
	}
	intersection_tmp->t = -1.0;
	return (0);
}

__host__ __device__ void			get_closest_cone(t_world world, t_ray ray,
		t_intersection *intersection, t_intersection *intersection_tmp)
{
	int i;

	i = 0;
	while (i < world.cones_len)
	{
		if (get_cone(world.cones[i], ray, intersection_tmp) == 1)
		{
			if (intersection_tmp->t < intersection->t && intersection_tmp->t != -1)
			{
				intersection->type = 'x';
				intersection->id = world.cones[i].id;
				intersection->t = intersection_tmp->t;
				intersection->reflection_coef = world.cones[i].reflection_coef;
				intersection->refraction_coef = world.cones[i].refraction_coef;
				intersection->transparence_coef = world.cones[i].transparence_coef;
				intersection->color = world.cones[i].color;
				intersection->chess = world.cones[i].chess;
				intersection->pos = vector_add(ray.origin,
						vector_scalar(ray.dir, intersection_tmp->t));//intersection_tmp->pos;
				intersection->normal_v = get_normal_cone(world.cones[i],
						ray, *intersection);//intersection_tmp->normal_v;
			}
		}
		i++;
	}
}
