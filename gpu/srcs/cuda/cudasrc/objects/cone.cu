#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   cone.c                                             :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: svilau <marvin@42.fr>                      +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2016/10/20 10:49:50 by svilau            #+#    #+#             */
/*   Updated: 2017/03/17 14:07:23 by svilau           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

extern "C" {
#include "../../cudaheader/gpu_rt.h"
}

__host__ __device__ static t_vec3d	get_normal_cone(t_cone cone, t_ray ray,
		t_intersection intersection)
{
	t_vec3d axis_v;
	t_vec3d	normal_v;
	t_vec3d	x;
	double	m;
	double	k;

	k = cone.half_angle;
	x = vector_substract(ray.origin, cone.pos);
	axis_v = vector_normalize(vector_calculate(cone.pos, cone.up));
	m = vector_dot(ray.dir, vector_scalar(axis_v, intersection.t))
		+ vector_dot(x, axis_v);
	normal_v = vector_normalize(
			vector_substract(
				vector_substract(
					intersection.pos, cone.pos),
				vector_scalar(
					vector_scalar(
						axis_v, m), 1 + (k * k))));
	return (normal_v);
}

__host__ __device__ static int		get_cone(t_cone cone, t_camera camera,
		t_ray ray, t_intersection *intersection_tmp)
{
	t_vec3d	normal;
	t_vec3d	x;
	t_eq	eq;
	double	k;

	normal = vector_normalize(vector_calculate(cone.pos, cone.up));
	x = vector_calculate(cone.pos, ray.origin);
	k = cone.half_angle;
	eq.a = vector_dot(ray.dir, ray.dir)
		- (1 + k * k) * pow(vector_dot(ray.dir, normal), 2);
	eq.b = 2 * (vector_dot(ray.dir, x)
			- (1 + k * k) * (vector_dot(ray.dir, normal) * vector_dot(x, normal)));
	eq.c = vector_dot(x, x)
		- (1 + k * k) * pow(vector_dot(x, normal), 2);
	eq.res = second_degres(eq.a, eq.b, eq.c);
	if(eq.res != NOT_A_SOLUTION && intersection_tmp->id != cone.id)
	{
		intersection_tmp->t = eq.res;
		intersection_tmp->type = 'x';
		if (cone.refraction_coef != 0 || cone.reflection_coef != 0)
			intersection_tmp->id = cone.id;
		return (1);
	}
	intersection_tmp->t = -1.0;
	return (0);
}

__host__ __device__ void		get_dist_cone_face(t_cone cone, t_ray ray,  t_intersection intersection)
{
	double	t;
	double	denominator;
	t_vec3d	x;
	double	n;
	double tmp;
	
	if(cone.height > 0.0f)
	{
		tmp = vector_dot(intersection.normal_v, ray.dir);
		if (tmp < 0.0f || tmp > cone.height)
		{
			intersection.normal_v = vector_normalize(vector_calculate(cone.pos, cone.up));
			denominator = vector_dot(ray.dir, intersection.normal_v);
			if (denominator != 0)
			{
				x = vector_scalar(vector_calculate(cone.pos, ray.origin), -1);
				n = vector_dot(x, intersection.normal_v);
				t = n / denominator;
				if (t > 0.0000001)
				{
				intersection.t = t;
				if (denominator > 0)
					intersection.normal_v = vector_scalar(intersection.normal_v, -1);
				}
			}
		}
	}
	intersection.t = -1;
}

__host__ __device__ void			get_closest_cone(t_world world, t_ray ray,
		t_intersection *intersection, t_intersection *intersection_tmp)
{
	int i;

	i = 0;
	while (i < world.cones_len)
	{
		if (get_cone(world.cones[i], world.camera, ray, intersection_tmp) == 1)
		{
			if (intersection_tmp->t < intersection->t && intersection_tmp->t != -1)
			{
				intersection->id = world.cones[i].id;
				intersection->t = intersection_tmp->t;
				intersection->type = intersection_tmp->type;
				intersection->reflection_coef = world.cones[i].reflection_coef;
				intersection->refraction_coef = world.cones[i].refraction_coef;
				intersection->color = &world.cones[i].color;
				intersection->chess = &world.cones[i].chess;
				intersection->pos = vector_add(ray.origin,
						vector_scalar(ray.dir, intersection_tmp->t));
				intersection->normal_v = get_normal_cone(world.cones[i],
						ray, *intersection);
			}
		}
		i++;
	}
}
