#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   cone.c                                             :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: svilau <marvin@42.fr>                      +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2016/10/20 10:49:50 by svilau            #+#    #+#             */
/*   Updated: 2017/03/17 14:07:23 by svilau           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

extern "C" {
	#include <rt.h>
	#include <vectors.h>
	#include <gpu_rt.h>
}

__host__ __device__ t_vec3d	get_normal_cone(t_cone cone, t_ray ray, t_intersection intersection)
{
	t_vec3d axis_v;
	t_vec3d	normal_v;
	t_vec3d	x;
	double	m;
	double	k;

	k = tan(deg_to_radians(cone.half_angle));
	x = vector_substract(ray.origin, cone.pos);
	axis_v = vector_normalize(vector_calculate(cone.pos, cone.up));
	m = vector_dot(ray.dir, axis_v) * intersection.t + vector_dot(x, axis_v);
	normal_v = vector_normalize(
				vector_substract(
					vector_substract(
						intersection.pos,
						cone.pos),
					vector_scalar(
						vector_scalar(
							axis_v,
							m),
						1 + (k * k))));
	return (normal_v);
}

__host__ __device__ void	get_determinant_cone(t_cone cone, t_camera camera, t_ray ray,
														t_2deg *equation)
{
	t_vec3d	normal;
	t_vec3d	x;
	double	k;

	normal = vector_normalize(vector_calculate(cone.pos,
														cone.up));
	x = vector_substract(ray.origin, cone.pos);
	k = tan(deg_to_radians(cone.half_angle));
	equation->a = vector_dot(ray.dir, ray.dir) - (1 + k * k) *
										pow(vector_dot(ray.dir, normal), 2);
	equation->b = 2 * (vector_dot(ray.dir, x) - (1 + k * k) *
						(vector_dot(ray.dir, normal) * vector_dot(x, normal)));
	equation->c = vector_dot(x, x) - (1 + k * k) *
							pow(vector_dot(x, normal), 2);
	equation->det = pow(equation->b, 2) - (4 * (equation->a) * (equation->c));
}

__host__ __device__ int		get_cone(t_cone cone, t_camera camera, t_ray ray,
									t_intersection *intersection_tmp)
{
	t_2deg	equation;
	double	t1;
	double	t2;

	get_determinant_cone(cone, camera, ray, &equation);
	if (equation.det >= 0)
	{
		t1 = ((-1) * equation.b + sqrt(equation.det)) / (2 * equation.a);
		t2 = ((-1) * equation.b - sqrt(equation.det)) / (2 * equation.a);
		if (t1 <= t2 && t1 > 0.0000001)
		{
			intersection_tmp->t = t1;
			intersection_tmp->type = 'x';
			return (1);
		}
		else if (t2 > 0.0000001)
		{
			intersection_tmp->t = t2;
			intersection_tmp->type = 'x';
			return (1);
		}
	}
	return (0);
}

__host__ __device__ void	get_closest_cone(t_world world, t_ray ray,
			t_intersection *intersection, t_intersection *intersection_tmp)
{
	int i;

	i = 0;
	while (i < world.cones_len)
	{
		if (get_cone(world.cones[i], world.camera, ray, intersection_tmp) == 1)
		{
			if (intersection_tmp->t < intersection->t)
			{
				intersection->t = intersection_tmp->t;
				intersection->type = intersection_tmp->type;
				intersection->reflexion_coef = world.cones[i].reflexion_coef;
				intersection->color = &world.cones[i].color;
				intersection->pos = vector_add(ray.origin,
					vector_scalar(ray.dir, intersection_tmp->t));
				intersection->normal_v = get_normal_cone(world.cones[i], ray,
														*intersection);
			}
		}
		i++;
	}
}
