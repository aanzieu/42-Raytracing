#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   cone.c                                             :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: svilau <marvin@42.fr>                      +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2016/10/20 10:49:50 by svilau            #+#    #+#             */
/*   Updated: 2017/03/17 14:07:23 by svilau           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

extern "C" {
#include "../../cudaheader/gpu_rt.h"
}

__host__ __device__ t_vec3d		get_normal_cone(t_cone cone,
		t_intersection *intersection_tmp)
{
	t_vec3d axis_v;
	t_vec3d	normal_v;
	t_vec3d	x;
	double	k;

	k = cone.half_angle;
	axis_v = vector_substract(intersection_tmp->pos, \
		vector_add(cone.pos, cone.up));
	axis_v = vector_scalar(cone.up, \
		vector_dot(cone.up, axis_v) / vector_dot(cone.up, cone.up));
	x = vector_add(axis_v, cone.up);
	normal_v = vector_substract(intersection_tmp->pos, cone.pos);
	x = vector_scalar(vector_normalize(x), vector_length(normal_v) / cos(k));
	normal_v = vector_normalize(vector_substract(normal_v, x));
	return (normal_v);
}

__host__ __device__ static double	limit_cone_next(t_eq eq,\
		t_cone cone, t_ray ray, t_intersection *intersection_tmp)
{
	t_vec3d pos[2];
	t_vec3d lim[2];

	pos[0] = vector_add(ray.origin, vector_scalar(ray.dir, eq.res[0]));
	pos[1] = vector_add(cone.pos, vector_scalar(cone.up, cone.height));
	lim[0] = vector_substract(pos[0], cone.pos);
	lim[1] = vector_substract(pos[0], pos[1]);
	if (vector_dot(cone.up, lim[0]) > SURFACE_TOLERANCE \
		&& vector_dot(cone.up, lim[1]) < SURFACE_TOLERANCE)
		return (eq.res[0]);
	return (-1);
}

__host__ __device__ static int limit_cone(t_eq eq, t_cone cone,\
		t_ray ray, t_intersection *intersection_tmp)
{
	double	t_save;

	if (eq.res[1] > eq.res[0])
		eq.res[1] = eq.res[0];
	t_save = eq.res[1];
	if (cone.height > 0)
		if ((eq.res[1] = limit_cone_next(eq, cone, ray, intersection_tmp)) == -1)
			return (-1);
	intersection_tmp->t = eq.res[1] == t_save ? eq.res[0] : eq.res[1];
	intersection_tmp->pos = vector_add(ray.origin,
		vector_scalar(ray.dir, intersection_tmp->t));
	intersection_tmp->normal_v = get_normal_cone(cone, intersection_tmp);
	return (1);
}

__host__ __device__ static int		get_cone(t_cone cone, t_ray ray,\
	t_intersection *intersection_tmp)
{
	t_eq		eq;
	t_vec3d x;
	t_vec3d axis_v[2];
	double 	k;

	if (intersection_tmp->id == cone.id)
		return (0);

	axis_v[0] = vector_substract(ray.origin, vector_add(cone.pos, cone.up));
	axis_v[1] = vector_cross(axis_v[0], cone.up);
	axis_v[0] = vector_cross(ray.dir, cone.up);
	x = vector_substract(ray.origin, cone.pos);
	k = cone.half_angle;
	eq.a = pow(cos(k), 2) * vector_dot(axis_v[0], axis_v[0]) - \
		pow(sin(k), 2) * pow(vector_dot(ray.dir, cone.up), 2);
	eq.b = pow(cos(k), 2) * vector_dot(axis_v[0], axis_v[1]) * 2 - \
		2 * pow(sin(k), 2) * vector_dot(ray.dir, cone.up) * vector_dot(x, cone.up);
	eq.c = pow(cos(k), 2) * vector_dot(axis_v[1], axis_v[1]) - \
		pow(sin(k), 2) * pow(vector_dot(x, cone.up), 2);
	second_degres(&eq);
	if(eq.res[0] != NOT_A_SOLUTION)
		return (limit_cone(eq, cone, ray, intersection_tmp));
	intersection_tmp->t = -1.0;
	return (0);
}

__host__ __device__ void			get_closest_cone(t_world world, t_ray ray,\
		t_intersection *intersection, t_intersection *intersection_tmp)
{
	int i;

	i = 0;
	while (i < world.cones_len)
	{
		if (get_cone(world.cones[i], ray, intersection_tmp) == 1)
		{
			if (intersection_tmp->t < intersection->t && intersection_tmp->t != -1)
			{
				intersection->type = 'x';
				intersection->id = world.cones[i].id;
				intersection->id_save = world.cones[i].id;
				intersection->t = intersection_tmp->t;
				intersection->reflection_coef = world.cones[i].reflection_coef;
				intersection->refraction_coef = world.cones[i].refraction_coef;
				intersection->transparence_coef = world.cones[i].transparence_coef;
				intersection->color = world.cones[i].color;
				intersection->chess = world.cones[i].chess;
				intersection->pos = intersection_tmp->pos;
				intersection->normal_v = intersection_tmp->normal_v;
				if(world.on == 1){
				apply_noise_dist(world, intersection, world.cones[i].perlin);}
			}
		}
		i++;
	}
}