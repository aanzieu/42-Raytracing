#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   torus.c                                            :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: huweber <marvin@42.fr>                     +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/08/23 19:29:36 by huweber           #+#    #+#             */
/*   Updated: 2017/08/23 19:29:39 by huweber          ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

extern "C" {
	#include <rt.h>
	#include "gpu_rt.h"
	#include <vectors.h>
}

/*
**	On envoie le rayon et la structure qui contient le torus et la fonction
**	ecrit sur 't_vec3d *intersection' les coordonees du point d'intersection
**	avec le plan
*/

__host__ __device__ t_vec3d get_normal_torus(t_intersection *intersection,
		t_torus to, t_ray ray)
{
	t_vec3d			normal;

	t_vec3d pos = vector_substract(intersection->pos, to.pos);
	double a = 1.0 - (to.radius_big / sqrt(pos.x * pos.x + pos.y * pos.y));
	normal.x = a * pos.x;
	normal.y = a * pos.y;
	normal.z = pos.z;
	normal = vector_normalize(normal);
	return (normal);
}

__host__ __device__ int	get_torus(t_torus to, t_ray ray,
		t_intersection *intersection_tmp)
{
	double	a[5];
	double	equ[8];

	if (intersection_tmp->id == to.id)
		return (0);
	ray.origin = vector_calculate(to.pos, ray.origin);
	equ[0] = 4.0 * to.radius_big * to.radius_big;
	equ[1] = equ[0] * (ray.dir.x * ray.dir.x + ray.dir.y * ray.dir.y);
	equ[2] = 2.0 * equ[0] * (ray.origin.x * ray.dir.x + ray.origin.y * ray.dir.y);
	equ[3] = equ[0] * (ray.origin.x * ray.origin.x + ray.origin.y * ray.origin.y);
	equ[4] = vector_magnitude(ray.dir);
	equ[5] = 2.0 * vector_dot(ray.origin, ray.dir);
	equ[6] = vector_magnitude(ray.origin) + to.radius_big * to.radius_big -
						to.radius_small * to.radius_small;
	a[0] = equ[4] * equ[4];
	a[1] = 2.0 * equ[4] * equ[5];
	a[2] = 2.0 * equ[4] * equ[6] + equ[5] * equ[5] - equ[1];
	a[3] = 2.0 * equ[5] * equ[6] - equ[2];
	a[4] = equ[6] * equ[6] - equ[3];
	if ((equ[7] = solver_quadra(a, 4)) > 0)
	{
		intersection_tmp->t = equ[7];
		return (1);
	}
//	intersection_tmp->t = -1.0;
	return (0);
}

__host__ __device__ void	get_closest_torus(t_world world, t_ray ray,
			t_intersection *intersection, t_intersection *intersection_tmp)
{
	int i;

	i = 0;
	while (i < world.torus_len)
	{
		if (get_torus(world.torus[i], ray, intersection_tmp) == 1)
		{
			if (intersection_tmp->t < intersection->t && intersection_tmp->t != -1)
			{
				intersection->id = world.torus[i].id;
				intersection->t = intersection_tmp->t;
				intersection->type = 'o';
				intersection->reflection_coef = world.torus[i].reflection_coef;
				intersection->reflection_coef = world.torus[i].reflection_coef;
				intersection->transparence_coef = world.torus[i].transparence_coef;
				intersection->id = intersection_tmp->id;
				intersection->color = world.torus[i].color;
				intersection->chess = world.torus[i].chess;
				intersection->pos =
					vector_add(ray.origin, vector_scalar(ray.dir, intersection->t));
				intersection->normal_v =
					get_normal_torus(intersection, world.torus[i], ray);
			}
		}
		i++;
	}
}
