#include "hip/hip_runtime.h"
extern "C" {
#include "../../../../includes/rt.h"
#include "../../cudaheader/gpu_rt.h"
#include <vectors.h>
#include <equation.h>
#include <float.h>
#include <math.h>
}

// __host__ __device__ int hitbox_torus(t_torus to, t_ray ray)
// {
// 	t_intersection inter;
//
// 	if (get_sphere(to.outer, ray, &inter) == 0)
// 		if (get_sphere(to.inner, ray, &inter) == 1)
// 			return (-1);
// 	return (1);
// }

__host__ __device__ int	get_torus(t_torus to, t_ray ray, t_intersection *inter)
{
	double	a[5], equ[7];//, roots[5];
	double t;

	ray.origin = vector_calculate(to.pos, ray.origin);
	equ[0] = 4.0 * to.radius_big * to.radius_big;
	equ[1] = equ[0] * (ray.dir.x * ray.dir.x + ray.dir.y * ray.dir.y);
	equ[2] = 2.0 * equ[0] * (ray.origin.x * ray.dir.x + ray.origin.y * ray.dir.y);
	equ[3] = equ[0] * (ray.origin.x * ray.origin.x + ray.origin.y * ray.origin.y);
	equ[4] = vector_magnitude(ray.dir);
	equ[5] = 2.0 * vector_dot(ray.origin, ray.dir);
	equ[6] = vector_magnitude(ray.origin) + to.radius_big * to.radius_big -
						to.radius_small * to.radius_small;
	a[0] = equ[4] * equ[4];
	a[1] = 2.0 * equ[4] * equ[5];
	a[2] = 2.0 * equ[4] * equ[6] + equ[5] * equ[5] - equ[1];
	a[3] = 2.0 * equ[5] * equ[6] - equ[2];
	a[4] = equ[6] * equ[6] - equ[3];
	if ((t = solver_n_degree3(a, 4)) > 0)
	{
		inter->t = t;
		inter->id = to.id;
		inter->type = 't';
		return (1);
	}
	inter->t = -1.0;
	return (0);
}

__host__ __device__ t_vec3d get_normal_torus(t_intersection *intersection,
		t_torus to, t_ray ray)
{
	t_vec3d			normal;

	t_vec3d pos = vector_substract(intersection->pos, to.pos);
	double a = 1.0 - (to.radius_big / sqrt(pos.x * pos.x + pos.y * pos.y));
	normal.x = a * pos.x;
	normal.y = a * pos.y;
	normal.z = pos.z;
	normal = vector_normalize(normal);
	return (normal);
}

__host__ __device__ void	get_closest_torus(t_world world, t_ray ray,
		t_intersection *intersection, t_intersection *intersection_tmp)
{
	int i;

	i = 0;
	while (i < world.torus_len)
	{
		if (get_torus(world.torus[i], ray, intersection_tmp) == 1)
		{
			if (intersection_tmp->t < intersection->t && intersection_tmp->t != -1)
			{
				intersection->t = intersection_tmp->t;
				intersection->type = intersection_tmp->type;
				intersection->reflection_coef = world.torus[i].reflection_coef;
				intersection->reflection_coef = world.torus[i].reflection_coef;
				intersection->id = intersection_tmp->id;
				intersection->color = &world.torus[i].color;
				intersection->chess = &world.torus[i].chess;
				intersection->pos =
					vector_add(ray.origin, vector_scalar(ray.dir, intersection->t));
				intersection->normal_v =
					get_normal_torus(intersection, world.torus[i], ray);
			}
		}
		i++;
	}
}
