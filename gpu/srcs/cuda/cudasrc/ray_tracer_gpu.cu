extern "C" {
	#include <rt.h>
	#include "gpu_rt.h"
	#include "object.h"
	#include "vectors.h"
}
#include "../../../frameworks/SDL2.framework/Headers/SDL.h"
#include "../../../includes/display.h"
#include <math.h>
#include <float.h>
#include <hip/hip_runtime.h>

__host__ __device__ double get_indice(t_intersection inter)
{
	if (inter.reflection_coef > 0)
		return (inter.reflection_coef);
	if (inter.transparence_coef > 0)
		return (inter.transparence_coef);
	return (0);
}

__host__ __device__ t_color apply_materials_gpu(t_world world, t_ray ray,
	t_intersection intersection, t_color color)
{
	t_color tmp = color;
	int			flag = 0, i = 0;
	double 	indice;

	indice = get_indice(intersection);
	while (flag == 0 && intersection.depth < MAX_DEPTH)
	{
		flag = 1;
		if (intersection.transparence_coef > 0)
			tmp = handle_transparence_gpu(world, &ray, &intersection, &flag);
		if (intersection.reflection_coef > 0)
			tmp = handle_reflection_gpu(world, &ray, &intersection, &flag);
		if (intersection.refraction_coef > 0)
			tmp = handle_refraction_gpu(world, &ray, &intersection, &flag);
		if (intersection.chess.r >= 0)
				tmp = handle_chess(ray, intersection);
		if (flag == 0)
		{
			i = 0;
			tmp = color_multiply(tmp, world.ambient.color);
			tmp = color_scalar(tmp, world.ambient.intensity);
			while (i < world.lights_len)
			{
				tmp = get_light_at(world, tmp, world.lights[i], intersection, ray);
				i++;
			}
		}
	}
	if (indice > 0 && indice < 1)
		return (color_scalar(tmp, indice));
	else
		return (tmp);
}

__host__ __device__ t_color		ray_tracer_depth_gpu(t_world world, t_ray ray,
				t_intersection intersection)
{
	t_color					color;
	int							i;

	i = 0;
	color = new_color(0, 0, 0);
	get_closest_intersection(world, ray, &intersection);
	if (intersection.type == '0')
		return ((t_color){0, 0, 0});
	// color = apply_materials_gpu(world, ray, intersection, color);
	color = intersection.color;
	color = color_multiply(color, world.ambient.color);
	color = color_scalar(color, world.ambient.intensity);
	if(world.keys.light_none == 0)
		color = intersection.color;
	if(world.keys.select == 1)
		cartoon_effect(world, &color, intersection, ray);
	while (i < world.lights_len && world.keys.light_none == 1)
	{
		color = get_light_at(world, color, world.lights[i], intersection, ray);
		if (world.keys.pad_0 == 6)
			cartoon_effect(world, &color, intersection, ray);
		i++;
	}
	return (color);
}

__host__ __device__ int		ray_tracer_gpu(t_world world, int x, int y)
{
	t_ray      			ray;
	t_color					color = (t_color){0, 0, 0};
	t_intersection	intersection;
	int i = 0, j = 0, xx = 0, yy = 0;

	while (i < world.aa)
	{
		j = 0;
		yy = y + i;
		while (j < world.aa)
		{
			xx = x + j;
			new_intersection(&intersection);
			ray.dir = (t_vec3d){0, 0, 0};
			ray.origin = (t_vec3d){0, 0, 0};
			get_up_left(&world);
			get_ray_direction(world, &ray, xx, yy);
			color = color_add(color, ray_tracer_depth_gpu(world, ray, intersection));
			j++;
		}
		i++;
	}
	color = color_divide(color, world.aa * world.aa);
	return (get_color(color));
}
