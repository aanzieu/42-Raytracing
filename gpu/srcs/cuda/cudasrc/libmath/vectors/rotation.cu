#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   rotation.c                                         :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: svilau <marvin@42.fr>                      +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2016/10/20 10:49:50 by svilau            #+#    #+#             */
/*   Updated: 2017/03/17 14:07:23 by svilau           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

extern "C" {
	#include <rt.h>
	#include <gpu_rt.h>
	#include <vectors.h>
}
#include <math.h>
#include <vectors.h>

__host__ __device__ void	rotate(t_vec3d *point, double degrees, char axis)
{
	t_vec3d tmp;

	tmp.x = point->x;
	tmp.y = point->y;
	tmp.z = point->z;
	degrees = (-1) * deg_to_radians(degrees);
	if (axis == 'z')
	{
		point->x = tmp.x * cos(degrees) - tmp.y * sin(degrees);
		point->y = tmp.x * sin(degrees) + tmp.y * cos(degrees);
	}
	else if (axis == 'x')
	{
		point->y = tmp.y * cos(degrees) - tmp.z * sin(degrees);
		point->z = tmp.y * sin(degrees) + tmp.z * cos(degrees);
	}
	else if (axis == 'y')
	{
		point->z = tmp.z * cos(degrees) - tmp.x * sin(degrees);
		point->x = tmp.z * sin(degrees) + tmp.x * cos(degrees);
	}
}

__host__ __device__ void	cam_rot(t_camera *camera, double degrees, char axis)
{
	degrees = (-1) * deg_to_radians(degrees);
	if (axis == 'z')
	{
		camera->look_at.z -= camera->pos.z;
		camera->look_at.y -= camera->pos.y;
		rotate(&camera->look_at, degrees, 'z');
		camera->look_at.z += camera->pos.z;
		camera->look_at.y += camera->pos.y;
		get_camera_axes(camera);
	}
	else if (axis == 'x')
	{
		camera->look_at.z -= camera->pos.z;
		camera->look_at.y -= camera->pos.y;
		rotate(&camera->look_at, degrees, 'x');
		camera->look_at.z += camera->pos.z;
		camera->look_at.y += camera->pos.y;
		get_camera_axes(camera);
	}
	else if (axis == 'y')
	{
		camera->look_at.z -= camera->pos.z;
		camera->look_at.x -= camera->pos.x;
		rotate(&camera->look_at, degrees, 'y');
		camera->look_at.z += camera->pos.z;
		camera->look_at.x += camera->pos.x;
		get_camera_axes(camera);
	}
}
