#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   vectors2.c                                         :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: svilau <marvin@42.fr>                      +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2016/10/20 10:49:50 by svilau            #+#    #+#             */
/*   Updated: 2017/03/17 14:07:23 by svilau           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

extern "C" {
	#include <vectors.h>
	#include <gpu_rt.h>
}
#include <math.h>

__host__ __device__ t_vec3d	vector_calculate(t_vec3d vect1, t_vec3d vect2)
{
	return (vector_substract(vect2, vect1));
}

__host__ __device__ t_vec3d	vector_cross(t_vec3d vect1, t_vec3d vect2)
{
	t_vec3d tmp;

	tmp.x = vect1.y * vect2.z - vect1.z * vect2.y;
	tmp.y = vect1.z * vect2.x - vect1.x * vect2.z;
	tmp.z = vect1.x * vect2.y - vect1.y * vect2.x;
	return (tmp);
}

__host__ __device__ double	vector_dot(t_vec3d vect1, t_vec3d vect2)
{
	double result;

	result = vect1.x * vect2.x + vect1.y * vect2.y + vect1.z * vect2.z;
	return (result);
}

__host__ __device__ double	vector_length(t_vec3d vect1)
{
	double length;

	length = sqrt(pow(vect1.x, 2) + pow(vect1.y, 2) + pow(vect1.z, 2));
	return (length);
}
