
#include <hip/hip_runtime.h>
// extern "C" {
// #include <../../../../../includes/rt.h>
// #include <../../../cudaheader/gpu_rt.h>
// #include <equation.h>
// #include <math.h>
// }
//
// #define	MAXPOS 100000000000000.0
//
// __host__ __device__ static void	third_degree_calc_trigo(t_solve s)
// {
//   double	k;
//
// 	s.trigo[0] = sqrt(((s.val[1] * s.val[1]) / 4.0) - s.val[2]);
// 	k = (s.trigo[0] < 0) ? -1 : 1;
// 	s.trigo[1] = pow(s.trigo[0] * k, (1.0 / 3.0)) * k;
// 	s.trigo[2] = (acos((-s.val[1]) / (2.0 * (s.trigo[0])))) / 3.0;
// }
//
// __host__ __device__ static void	third_degree_calc_tmp(double *tmp, double rc, double theta, double val)
// {
// 	tmp[0] = rc;
// 	tmp[1] = cos(theta);
// 	tmp[2] = sqrt(3.0) * sin(theta);
// 	tmp[3] = val * -1.0;
// }
//
// __host__ __device__ static void third_degree_round(double *x, double *tmp, double val)
// {
// 	x[1] = (tmp[0] * (tmp[1] + tmp[2])) - val;
// 	x[2] = (tmp[0] * (tmp[1] - tmp[2])) - val;
// 	x[0] = check_solution(round(x[0] * MAXPOS) / MAXPOS);
// 	x[1] = check_solution(round(x[1] * MAXPOS) / MAXPOS);
// 	x[2] = check_solution(round(x[2] * MAXPOS) / MAXPOS);
// 	x[1] = check_solution_equation(x[1], x[2]);
// }
//
// __host__ __device__ static double	third_degree_neg(double g, double h, double val)
// {
// 	double	m[2];
// 	double	n[2];
// 	double	k;
//
// 	m[0] = (-g / 2.0) + sqrt(h);
// 	k = (m[0] < 0) ? -1 : 1;
// 	m[1] = pow(m[0] * k, val) * k;
// 	n[0] = (-g / 2.0) - sqrt(h);
// 	k = (n[0] < 0) ? -1 : 1;
// 	n[1] = pow(n[0] * k, val) * k;
// 	return (n[1] + m[1]);
// }
//
// __host__ __device__ double		third_degree(double a, double b, double c, double d)
// {
// 	t_solve s;
//
// 	s.val[0] = (((3.0 * c) / a) - (((b * b) / (a * a)))) / 3.0;
// 	s.val[1] = ((2.0 * ((b * b * b) / (a * a * a))) - ((9.0 * b * c) / (a * a)) + ((27.0 * (d / a)))) / 27.0;
// 	s.val[2] = ((s.val[1] * s.val[1]) / 4.0) + ((s.val[0] * s.val[0] * s.val[0]) / 27.0);
// 	if (s.val[2] > 0)
// 	s.x[0] = s.x[1] = s.x[2] = (third_degree_neg(s.val[1], s.val[2], (1.0 / 3.0))) - (b / (3.0 * a));
// 	else
// 	{
// 		third_degree_calc_trigo(s);
// 		s.x[0] = (2.0 * (s.trigo[1] * cos(s.trigo[2]))) - (b / (3.0 * a));
// 		third_degree_calc_tmp(s.tmp, -1 * s.trigo[1], s.trigo[2], (b / (3.0 * a)));
// 		third_degree_round(s.x, s.tmp, (b / (3.0 * a)));
// 	}
// 	if (is_zero(s.val[0] + s.val[1] + s.val[2]))
// 		s.x[0] = s.x[1] = s.x[2] = (d < 0) ? pow(d / a, (1.0 / 3.0)) * -1 :
// 	pow((-d) / a, (1.0 / 3.0));
// 	return (check_solution_equation(check_solution(s.x[0]), check_solution(s.x[1])));
// }
