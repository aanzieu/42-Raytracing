#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   equation_resolve.c                              :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: aanzieu <marvin@42.fr>                     +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/05/11 16:32:15 by aanzieu           #+#    #+#             */
/*   Updated: 2017/06/20 11:44:40 by aanzieu          ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

extern "C" {
#include "../../../cudaheader/gpu_rt.h"
}

__host__ __device__ int				solve_second_case(t_dichotomie c, double *a,
									double *r)
{
	c.sqrtbq = sqrt(c.bq);
	if (c.br > 0)
	{
		r[0] = -2 * c.sqrtbq - a[2] / 3;
		r[1] = c.sqrtbq - a[2] / 3;
		r[2] = c.sqrtbq - a[2] / 3;
	}
	else
	{
		r[0] = -c.sqrtbq - a[2] / 3;
		r[1] = -c.sqrtbq - a[2] / 3;
		r[2] = 2 * c.sqrtbq - a[2] / 3;
	}
	return (3);
}

__host__ __device__ int				solve_third_case(t_dichotomie c, double *a,
									double *r)
{
	c.ratio = c.sgnbr * sqrt(c.br2 / c.bq3);
	c.theta = acos(c.ratio);
	c.norm = -2 * sqrt(c.bq);
	c.r0 = c.norm * cos(c.theta / 3) - a[2] / 3;
	c.r1 = c.norm * cos((c.theta + 2.0 * M_PI) / 3) - a[2] / 3;
	c.r2 = c.norm * cos((c.theta - 2.0 * M_PI) / 3) - a[2] / 3;
	if (c.r0 > c.r1)
		swap_double(&c.r0, &c.r1);
	if (c.r1 > c.r2)
	{
		swap_double(&c.r1, &c.r2);
		if (c.r0 > c.r1)
			swap_double(&c.r0, &c.r1);
	}
	r[0] = c.r0;
	r[1] = c.r1;
	r[2] = c.r2;
	return (3);
}

__host__ __device__ int				solve_fourth_case(t_dichotomie c,
									double *a, double *r)
{
	c.ba = -c.sgnbr * pow(fabs(c.br) + sqrt(c.br2 - c.bq3), 1.0 / 3.0);
	c.bb = c.bq / c.ba;
	r[0] = c.ba + c.bb - a[2] / 3;
	return (1);
}

__host__ __device__ static double	check_solution_equation(double a, double b)
{
	if (a > 0.0000000001 && b > 0.0000000001)
		return (ft_smaller(a, b));
	else if (a > 0.0000000001)
		return (a);
	else if (b > 0.0000000001)
		return (b);
	return (-1.0);
}

__host__ __device__ void			second_degres(t_eq *eq)
{
	t_solve s;

	s.det = eq->b * eq->b - 4.0 * eq->a * eq->c;
	if (is_zero(s.det))
	{
		eq->res[0] = -1.0 * eq->b / (2.0 * eq->a);
		eq->res[0] = check_solution(s.res);
		eq->res[1] = eq->res[0];
		eq->res[2] = eq->res[0];
 	}
	else if (s.det >= 0.0)
	{
		s.tmp[0] = ((-1) * eq->b + sqrt(s.det)) / (2 * eq->a);
		s.tmp[1] = ((-1) * eq->b - sqrt(s.det)) / (2 * eq->a);
		s.tmp[0] = check_solution(s.tmp[0]);
		s.tmp[1] = check_solution(s.tmp[1]);
		eq->res[0] = check_solution_equation(s.tmp[0], s.tmp[1]);
		eq->res[1] = eq->res[0] == s.tmp[0] ? s.tmp[1] : s.tmp[0];
	}
	else
		eq->res[0] = NOT_A_SOLUTION;
}
