#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   equation_resolve.c                              :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: aanzieu <marvin@42.fr>                     +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/05/11 16:32:15 by aanzieu           #+#    #+#             */
/*   Updated: 2017/06/20 11:44:40 by aanzieu          ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

extern "C" {
#include <rt.h>
#include <gpu_rt.h>
#include <equation.h>
}

__host__ __device__ inline double		ft_smaller(double a, double b)
{
	return(a < b ? a : b);
}

__host__ __device__ inline double		check_solution(double res)
{
	return((res > 0.0) ? res : -1);
}

__host__ __device__ double		check_solution_equation(double a, double b)
{
	if (a > 0.0 && b > 0.0)
		return (ft_smaller(a, b));
	else if (a > 0.0)
		return (a);
	else if (b > 0.0)
		return (b);
	return (-1.0);
}
__host__ __device__ int		is_zero(double x)
{
	return(fabs(x) < ZERO_DP ? 1 : 0);
}
__host__ __device__ double		second_degres(double a, double b, double c)
{
//	double	res;
//	double	det;
//	double	tmp[4];
	t_solve s;

	s.det = b * b - 4.0 * a * c;
//	else if (det > 0.0)
	if (is_zero(s.det))
	{
		s.res = -1.0 * b / (2.0 * a);
		s.res = check_solution(s.res);
	}
	else if (s.det > 0.0)
	{
		s.tmp[0] = ((-1) * b + sqrt(s.det)) / (2 * a);
		s.tmp[1] = ((-1) * b - sqrt(s.det)) / (2 * a);
		s.tmp[0] = check_solution(s.tmp[0]);
		s.tmp[1] = check_solution(s.tmp[1]);
		s.res = check_solution_equation(s.tmp[0], s.tmp[1]);
	}
	else
		s.res = NOT_A_SOLUTION;//-1;
	return (s.res);
/*
	t_solve s;
	s.det = b * b - 4.0 * a * c;
	if (ZERO(s.det))
	{
		s.res = -1.0 * b / (2.0 * a);
		s.res = check_solution(s.res);
	}
	else if (s.det > 0.0)
	{
		s.tmp[2] = sqrt(s.det);
		s.tmp[3] = 2.0 * a;
		s.tmp[0] = -(b - s.tmp[2]) / s.tmp[3];
		s.tmp[1] = -(b + s.tmp[2]) / s.tmp[3];
		s.tmp[0] = check_solution(s.tmp[0]);
		s.tmp[1] = check_solution(s.tmp[1]);
		s.res = check_solution_equation(s.tmp[0], s.tmp[1]);
	}
	else
		s.res = NOT_A_SOLUTION;
	return (s.res);
	*/
}
