#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   equation_resolve.c                              :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: aanzieu <marvin@42.fr>                     +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/05/11 16:32:15 by aanzieu           #+#    #+#             */
/*   Updated: 2017/06/20 11:44:40 by aanzieu          ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

extern "C" {
#include <rt.h>
#include <gpu_rt.h>
#include <equation.h>
#include <math.h>
}

__host__ __device__ inline double		ft_smaller(double a, double b)
{
	return(a <= b ? a : b);
}

__host__ __device__ inline double		check_solution(double res)
{
	return((res > 0.0) ? res : -1);
}

__host__ __device__ double		check_solution_equation(double a, double b)
{
	if (a > 0.0000000001 && b > 0.0000000001)
		return (ft_smaller(a, b));
	else if (a > 0.0000000001)
		return (a);
	else if (b > 0.0000000001)
		return (b);
	return (-1.0);
}
__host__ __device__ int		is_zero(double x)
{
	return(fabs(x) < ZERO_DP ? 1 : 0);
}
__host__ __device__ double		second_degres(double a, double b, double c)
{
	t_solve s;

	s.det = b * b - 4.0 * a * c;
	if (is_zero(s.det))
	{
		s.res = -1.0 * b / (2.0 * a);
		s.res = check_solution(s.res);
	}
	else if (s.det >= 0.0)
	{
		s.tmp[0] = ((-1) * b + sqrt(s.det)) / (2 * a);
		s.tmp[1] = ((-1) * b - sqrt(s.det)) / (2 * a);
		s.tmp[0] = check_solution(s.tmp[0]);
		s.tmp[1] = check_solution(s.tmp[1]);
		s.res = check_solution_equation(s.tmp[0], s.tmp[1]);
	}
	else
		s.res = NOT_A_SOLUTION;
	return (s.res);
}
