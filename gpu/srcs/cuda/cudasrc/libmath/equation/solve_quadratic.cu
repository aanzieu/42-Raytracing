#include "hip/hip_runtime.h"
extern "C" {
#include "../../../../../includes/rt.h"
#include "../../../cudaheader/gpu_rt.h"
#include <equation.h>
#include <math.h>
}

__host__ __device__ void	delta_neg(double p, double q, double *res, double trans)
{
  int	nb_sol;

  nb_sol = 0;
  while (nb_sol <= 2)
    {
      res[nb_sol] = (2.0 * sqrt(- p / 3.0)
		     * cos((1.0 / 3.0)
			   * acos((-q / 2.0)
				  * sqrt(27.0 / (- p * p * p)))
			   + (2.0 * nb_sol * M_PI) / 3.0)) - trans;
      nb_sol++;
    }
}

__host__ __device__ int		cardan_method(double p, double q, double *res, double trans)
{
  double	delta;

  delta = q * q + (4.0 / 27.0) * p * p * p;
  if (delta > 0)
    {
      res[0] = (cbrt((-q + sqrt(delta)) / 2.0)
		+ cbrt((-q - sqrt(delta)) / 2.0)) - trans;
      return (1);
    }
  if (delta == 0.0)
    {
      res[0] = ((3.0 * q) / p) - trans;
      res[1] = ((-3.0 * q) / (2.0 * p)) - trans;
      return (2);
    }
  if (delta < 0.0)
    {
      delta_neg(p, q, res, trans);
      return (3);
    }
  return (0);
}

__host__ __device__ int		solve_quadratic(double *coef, double *res)
{
  double	delta;
  double	k;
  int		nb_sol;

  nb_sol = 0;
  delta = coef[1] * coef[1] - 4 * coef[0] * coef[2];
  if (delta < 0.0)
    return (0);
  if (delta >= 0.0)
    {
      k = (-coef[1] + sqrt(delta)) / (2 * coef[0]);
      res[nb_sol++] = k;
    }
  if (delta > 0.0)
    {
      k = (-coef[1] - sqrt(delta)) / (2 * coef[0]);
      res[nb_sol++] = k;
    }
  return (nb_sol);
}

__host__ __device__ int		solve_cubic(double *coef, double *res)
{
  double	p;
  double	q;

  if (fabs(coef[0]) <= SURFACE_TOLERANCE)
    return (solve_quadratic(&(coef[1]), res));
  p = -((coef[1] * coef[1]) / (3.0 * coef[0] * coef[0]))
    + (coef[2] / coef[0]);
  q = (coef[1] / (27.0 * coef[0])) *
    ((2.0 * coef[1] * coef[1]) / (coef[0] * coef[0]) -
     (9.0 * coef[2]) / coef[0]) + coef[3] / coef[0];
  return (cardan_method(p, q, res, coef[1] / (3.0 * coef[0])));
}
