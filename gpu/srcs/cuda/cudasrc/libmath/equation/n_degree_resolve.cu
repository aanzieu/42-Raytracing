#include "hip/hip_runtime.h"
extern "C" {
#include "../../../../../includes/rt.h"
#include "../../../cudaheader/gpu_rt.h"
#include <equation.h>
#include <math.h>
}

#define EPSILON (1e-9)

__host__ __device__ void	swap_double(double *a, double *b)
{
	double tmp;
	tmp = *a;
	*a = *b;
	*b = tmp;
}

__host__ __device__ static t_dichotomie	init_stru(double *a)
{
	t_dichotomie	ret;

	ret.q = a[2] * a[2] - 3 * a[1];
	ret.r = 2 * a[2] * a[2] * a[2] - 9 * a[2] * a[1] + 27 * a[0];
	ret.bq = ret.q / 9;
	ret.br = ret.r / 54;
	ret.bq3 = ret.bq * ret.bq * ret.bq;
	ret.br2 = ret.br * ret.br;
	ret.cr2 = 729 * ret.r * ret.r;
	ret.cq3 = 2916 * ret.q * ret.q * ret.q;
	ret.sgnbr = ret.br >= 0 ? 1 : -1;
	ret.i = 0;
	return (ret);
}

__host__ __device__ static int		solve_second_case(t_dichotomie c, double *a, double *r)
{
	c.sqrtbq = sqrt(c.bq);
	if (c.br > 0)
	{
		r[0] = -2 * c.sqrtbq - a[2] / 3;
		r[1] = c.sqrtbq - a[2] / 3;
		r[2] = c.sqrtbq - a[2] / 3;
	}
	else
	{
		r[0] = -c.sqrtbq - a[2] / 3;
		r[1] = -c.sqrtbq - a[2] / 3;
		r[2] = 2 * c.sqrtbq - a[2] / 3;
	}
	return (3);
}

__host__ __device__ static int		solve_third_case(t_dichotomie c, double *a, double *r)
{
	c.ratio = c.sgnbr * sqrt(c.br2 / c.bq3);
	c.theta = acos(c.ratio);
	c.norm = -2 * sqrt(c.bq);
	c.r0 = c.norm * cos(c.theta / 3) - a[2] / 3;
	c.r1 = c.norm * cos((c.theta + 2.0 * M_PI) / 3) - a[2] / 3;
	c.r2 = c.norm * cos((c.theta - 2.0 * M_PI) / 3) - a[2] / 3;
	if (c.r0 > c.r1)
		swap_double(&c.r0, &c.r1);
	if (c.r1 > c.r2)
	{
		swap_double(&c.r1, &c.r2);
		if (c.r0 > c.r1)
			swap_double(&c.r0, &c.r1);
	}
	r[0] = c.r0;
	r[1] = c.r1;
	r[2] = c.r2;
	return (3);
}

__host__ __device__ static int		solve_fourth_case(t_dichotomie c, double *a, double *r)
{
	c.ba = -c.sgnbr * pow(fabs(c.br) + sqrt(c.br2 - c.bq3), 1.0 / 3.0);
	c.bb = c.bq / c.ba;
	r[0] = c.ba + c.bb - a[2] / 3;
	return (1);
}

__host__ __device__ int				solve_n(double *a, double *r)
{
	t_dichotomie		c;
	
	c = init_stru(a);
	if (c.br == 0 && c.bq == 0)
	{
		while (c.i++ < 3)
			r[c.i] = -a[2] / 3;
		return (3);
	}
	else if (c.cr2 == c.cq3)
		return (solve_second_case(c, a, r));
	else if (c.br2 < c.bq3)
		return (solve_third_case(c, a, r));
	else
		return (solve_fourth_case(c, a, r));
}

__host__ __device__ int		dblsgn(double x)
{
	if (x < -EPSILON)
		return (-1);
	return (x > EPSILON);
}

__host__ __device__ int	 find_inside(t_mobius m, t_vec3d h)
{
	double t;
	double s;

	t = atan2(h.y, h.x);
	if (dblsgn(sin(t / 2)) != 0)
		s = h.z / sin(t / 2);
	else
		s = dblsgn(cos(t)) ? (h.x / cos(t) - m.radius) / cos(t / 2) :
			(h.y / sin(t) - m.radius) / cos(t / 2);
	h.x -= (m.radius + s * cos(t / 2)) * cos(t);
	h.y -= (m.radius + s * cos(t / 2)) * sin(t);
	h.z -= s * sin(t / 2);
	if (dblsgn(h.x * h.x + h.y * h.y + h.z * h.z))
		return (0);
	return (s >= -0.5 && s <= 0.5);
	//define un max pour le mobius
}

__host__ __device__ double		solver_n_degree(double *coef, int degree, t_mobius m, t_ray ray)
{
	double	sols[3];
	int		i;
	int		ret;
	t_vec3d hit;

	if (degree < 1)
		return (-1.0);
	i = -1;
	ret = solve_n(coef, sols);
	while (++i < ret)
	{
		if (sols[i] > EPSILON)
		{
			
			hit = vector_add(vector_scalar(ray.dir, sols[i]), ray.origin);
			if (find_inside(m, hit))
				return (sols[i]);
//			return ((sols[i] < MAX_ROOT) ? sols[i] : -1.0);
		}
	}
	return (INFINITY);
}

/*
__host__ __device__ double	calc_poly(double *coef, int degree, double a)
{
	double	res;
	double	x;
	int		i;

	res = 0;
	x = 1.0;
	i = -1;
	while (++i < degree + 1)
	{
		res += coef[degree - i] * x;
		x *= a;
	}
	return (res);
}

__host__ __device__ static double	*get_dcoef(double *dcoef, double *coef, int deg)
{
	int		i;

	i = -1;
	while (++i < deg)
		dcoef[i] = coef[i] * (deg - i);
	return (dcoef);
}

__host__ __device__ void	reduce_coef(double *coef, int *deg)
{
	int		i;

	while (is_zero(coef[0]) && *deg > 1)
	{
		i = -1;
		while (++i < *deg)
			coef[i] = coef[i + 1];
		--(*deg);
	}
}

__host__ __device__ void		solve_n(double *coef, double *sol, int deg)
{
	double	dcoef[10];
	double	dsol[10];
	int		i;
	int		j;
	double	tmp;

	reduce_coef(coef, &deg);
	if (deg == 1)
		solve_first(coef, sol);
	else if (deg == 2)
		solve_second(coef, sol);
	if (deg < 3)
		return ;
	get_dcoef(dcoef, coef, deg);
	solve_n(dcoef, dsol, deg - 1);
	sol[0] = -MAX_ROOT;
	i = -(j = 1);
	while (++i < deg && dsol[i] != MAX_ROOT)
		if (fabs(tmp = calc_root(coef, deg, dsol[i], dsol[i + 1])) < MAX_ROOT)
			sol[j++] = tmp;
	sol[j] = MAX_ROOT;
}*/
