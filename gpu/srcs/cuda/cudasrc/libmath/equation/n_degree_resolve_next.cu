
#include <hip/hip_runtime.h>
// extern "C" {
// #include "../../../../../includes/rt.h"
// #include "../../../cudaheader/gpu_rt.h"
// #include <equation.h>
// #include <math.h>
// }
// /*
// __host__ __device__ void	solve_first(double *coef, double *sol)
// {
// 	sol[0] = -MAX_ROOT;
// 	sol[1] = -coef[1] / coef[0];
// 	sol[2] = MAX_ROOT;
// }
// __host__ __device__ void	swap_double(double *a, double *b)
// {
// 	double tmp;
// 	tmp = *a;
// 	*a = *b;
// 	*b = tmp;
// }
//
// __host__ __device__ void	solve_second(double *coef, double *sol)
// {
// 	double	det;
// 	double	opti;
//
// 	sol[0] = -MAX_ROOT;
// 	opti = -2.0 * coef[0];
// 	det = coef[1] * coef[1] + 2.0 * opti * coef[2];
// 	if (det == 0.0)
// 	{
// 		sol[1] = coef[1] / opti;
// 		sol[2] = MAX_ROOT;
// 	}
// 	else if (det >= 0.0)
// 	{
// 		det = sqrt(det);
// 		sol[1] = (coef[1] - det) / opti;
// 		sol[2] = (coef[1] + det) / opti;
// 		if (sol[1] > sol[2])
// 			swap_double(sol + 1, sol + 2);
// 		sol[3] = MAX_ROOT;
// 	}
// 	else
// 		sol[1] = MAX_ROOT;
// }
//
// __host__ __device__ static double	loop_dicho(double *coeff, int deg, t_dichotomie *d)
// {
// 	int		i;
// 	double	tmp;
//
// 	i = -1;
// 	while (++i < 1000000)
// 	{
// 		if (d->sl == 0 || d->sr == 0)
// 			return ((d->sl == 0) ? d->l : d->r);
// 		d->mid = d->l / 2.0 + d->r / 2.0;
// 		tmp = calc_poly(coeff, deg, d->mid);
// 		d->smid = SIGNOF(tmp);
// 		if (fabs(d->l - d->r) < 0.0001 || !d->smid)
// 			return (d->mid);
// 		else if (d->sl * d->smid > 0)
// 		{
// 			d->l = d->mid;
// 			d->sl = d->smid;
// 		}
// 		else
// 		{
// 			d->r = d->mid;
// 			d->sr = d->smid;
// 		}
// 	}
// 	return ((d->l + d->r) / 2.0);
// }
//
// __host__ __device__ double		calc_root(double *coeff, int deg, double l, double r)
// {
// 	t_dichotomie d;
// 	double	tmp;
//
// 	d.l = l;
// 	d.r = r;
// 	tmp = calc_poly(coeff, deg, l);
// 	d.sl = SIGNOF(tmp);
// 	tmp = calc_poly(coeff, deg, r);
// 	d.sr = SIGNOF(tmp);
// 	if (d.sl * d.sr > 0)
// 		return (MAX_ROOT + 1);
// 	return (loop_dicho(coeff, deg, &d));
// }*/
