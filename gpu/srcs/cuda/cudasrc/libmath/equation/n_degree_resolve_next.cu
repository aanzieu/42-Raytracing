#include "hip/hip_runtime.h"
extern "C" {
#include "../../../../../includes/rt.h"
#include "../../../cudaheader/gpu_rt.h"
#include <equation.h>
#include <math.h>
}

#pragma hd_warning_disable

__host__ __device__ static void 	swap_double_cuda(double *a, double *b)
{
	double tmp;
	tmp = *a;
	*a = *b;
	*b = tmp;
}

__host__ __device__ static double	calc_poly(double *coef, int degree, double a)
{
  double	res;
  double	x;
  int		i;

  res = 0;
  x = 1.0;
  i = -1;
  while (++i < degree + 1)
    {
      res += coef[degree - i] * x;
      x *= a;
    }
  return (res);
}

__host__ __device__ static double	*get_dcoef(double *dcoef, double *coef,
  int deg)
{
  int		i;

  i = -1;
  while (++i < deg)
    dcoef[i] = coef[i] * (deg - i);
  return (dcoef);
}

__host__ __device__ static void	reduce_coef(double *coef, int *deg)
{
  int		i;

  while (IS_ZERO(coef[0]) && *deg > 1)
    {
      i = -1;
      while (++i < *deg)
	coef[i] = coef[i + 1];
      --(*deg);
    }
}

__host__ __device__ static void	solve_first(double *coef, double *sol)
{
  sol[0] = -MAX_ROOT;
  sol[1] = -coef[1] / coef[0];
  sol[2] = MAX_ROOT;
}

__host__ __device__ static void	solve_second(double *coef, double *sol)
{
  double	det;
  double	opti;

  sol[0] = -MAX_ROOT;
  opti = -2.0 * coef[0];
  det = coef[1] * coef[1] + 2.0 * opti * coef[2];
  if (det == 0.0)
    {
      sol[1] = coef[1] / opti;
      sol[2] = MAX_ROOT;
    }
  else if (det >= 0.0)
    {
      det = sqrt(det);
      sol[1] = (coef[1] - det) / opti;
      sol[2] = (coef[1] + det) / opti;
      if (sol[1] > sol[2])
				swap_double_cuda(sol + 1, sol + 2);
      sol[3] = MAX_ROOT;
    }
  else
    sol[1] = MAX_ROOT;
}

__host__ __device__ static double	loop_dicho(double *coeff, int deg,
    t_dichotomie *d)
{
  int		i;
  double	tmp;

  i = -1;
  while (++i < 1000000)
    {
      if (d->sl == 0 || d->sr == 0)
	return ((d->sl == 0) ? d->l : d->r);
      d->mid = d->l / 2.0 + d->r / 2.0;
      tmp = calc_poly(coeff, deg, d->mid);
      d->smid = SIGNOF(tmp);
      if (fabs(d->l - d->r) < 0.0001 || !d->smid)
	return (d->mid);
      else if (d->sl * d->smid > 0)
	{
	  d->l = d->mid;
	  d->sl = d->smid;
	}
      else
	{
	  d->r = d->mid;
	  d->sr = d->smid;
	}
    }
  return ((d->l + d->r) / 2.0);
}

__host__ __device__ static double		calc_root(double *coeff, int deg, double l,
    double r)
{
  t_dichotomie	d;
  double	tmp;

  d.l = l;
  d.r = r;
  tmp = calc_poly(coeff, deg, l);
  d.sl = SIGNOF(tmp);
  tmp = calc_poly(coeff, deg, r);
  d.sr = SIGNOF(tmp);
  if (d.sl * d.sr > 0)
    return (MAX_ROOT + 1);
  return (loop_dicho(coeff, deg, &d));
}

__host__ __device__ void		  solve_quadra_next(double *coef, double *sol,
    int deg)
{
  double	dcoef[10];
  double	dsol[10];
  int		i;
  int		j;
  double	tmp;

  reduce_coef(coef, &deg);
  if (deg == 1)
    solve_first(coef, sol);
  else if (deg == 2)
    solve_second(coef, sol);
  if (deg < 3)
    return ;
  get_dcoef(dcoef, coef, deg);
  solve_quadra_next(dcoef, dsol, deg - 1);
  sol[0] = -MAX_ROOT;
  i = -(j = 1);
  while (++i < deg && dsol[i] != MAX_ROOT)
    if (fabs(tmp = calc_root(coef, deg, dsol[i], dsol[i + 1])) < MAX_ROOT)
      sol[j++] = tmp;
  sol[j] = MAX_ROOT;
}

__host__ __device__ double		solver_quadra(double *coef, int degree)
{
  double	sols[10];
  int		i;

  if (degree < 1)
    return (-1.0);
  i = -1;
  solve_quadra_next(coef, sols, degree);
  while (++i < 10)
    if (sols[i] > 0.0001)
      return ((sols[i] < MAX_ROOT) ? sols[i] : -1.0);
  return (-1.0);
}
