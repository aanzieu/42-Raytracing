#include "hip/hip_runtime.h"
extern "C" {
#include "../../../../../includes/rt.h"
#include "../../../cudaheader/gpu_rt.h"
#include <equation.h>
#include <math.h>
}

__host__ __device__ void	delta_neg(double p, double q, double *res, double trans)
{
  int	nb_sol;

  nb_sol = 0;
  while (nb_sol <= 2)
    {
      res[nb_sol] = (2.0 * sqrt(- p / 3.0)
		     * cos((1.0 / 3.0)
			   * acos((-q / 2.0)
				  * sqrt(27.0 / (- p * p * p)))
			   + (2.0 * nb_sol * M_PI) / 3.0)) - trans;
      nb_sol++;
    }
}

__host__ __device__ int		cardan_method(double p, double q, double *res, double trans)
{
  double	delta;

  delta = q * q + (4.0 / 27.0) * p * p * p;
  if (delta > 0)
    {
      res[0] = (cbrt((-q + sqrt(delta)) / 2.0)
		+ cbrt((-q - sqrt(delta)) / 2.0)) - trans;
      return (1);
    }
  if (delta == 0.0)
    {
      res[0] = ((3.0 * q) / p) - trans;
      res[1] = ((-3.0 * q) / (2.0 * p)) - trans;
      return (2);
    }
  if (delta < 0.0)
    {
      delta_neg(p, q, res, trans);
      return (3);
    }
  return (0);
}

__host__ __device__ int		solve_quadratic(double *coef, double *res)
{
  double	delta;
  double	k;
  int		nb_sol;

  nb_sol = 0;
  delta = coef[1] * coef[1] - 4 * coef[0] * coef[2];
  if (delta < 0.0)
    return (0);
  if (delta >= 0.0)
    {
      k = (-coef[1] + sqrt(delta)) / (2 * coef[0]);
      res[nb_sol++] = k;
    }
  if (delta > 0.0)
    {
      k = (-coef[1] - sqrt(delta)) / (2 * coef[0]);
      res[nb_sol++] = k;
    }
  return (nb_sol);
}

__host__ __device__ int		solve_cubic(double *coef, double *res)
{
  double	p;
  double	q;

  if (fabs(coef[0]) <= SURFACE_TOLERANCE)
    return (solve_quadratic(&(coef[1]), res));
  p = -((coef[1] * coef[1]) / (3.0 * coef[0] * coef[0]))
    + (coef[2] / coef[0]);
  q = (coef[1] / (27.0 * coef[0])) *
    ((2.0 * coef[1] * coef[1]) / (coef[0] * coef[0]) -
     (9.0 * coef[2]) / coef[0]) + coef[3] / coef[0];
  return (cardan_method(p, q, res, coef[1] / (3.0 * coef[0])));
}

__host__ __device__ void	set_pqr_quartic(double *pqr, double *coef)
{
  pqr[0] = -3.0 * (coef[1] * coef[1]) / (8.0 * coef[0] * coef[0])
    + (coef[2] / coef[0]);
  pqr[1] = (pow(coef[1] / 2.0, 3) / pow(coef[0], 3))
    - (0.5 * coef[1] * coef[2])
    / pow(coef[0], 2) + coef[3] / coef[0];
  pqr[2] = -3.0 * pow((coef[1] / 4.0) / coef[0], 4)
    + coef[2] * (pow(coef[1] / 4.0, 2) / pow(coef[0], 3))
    - ((0.25 * coef[1] * coef[3]) / (coef[0] * coef[0]))
    + (coef[4] / coef[0]);
}

__host__ __device__ void	init_to_send_cubic(double p, double q,
			   double r, double *coef)
{
  coef[0] = 8.0;
  coef[1] = -4.0 * p;
  coef[2] = -8.0 * r;
  coef[3] = 4.0 * r * p - q * q;
}

__host__ __device__ int		init_and_send_second(double *pqr, double y0, double *res)
{
  double	a0;
  double	b0;
  double	coef[3];
  int		nb_sol;

  nb_sol = 0;
  a0 = sqrt(-pqr[0] + 2.0 * y0);
  if ((2.0 * y0 - pqr[0]) != 0.0)
    b0 = -pqr[1] / (2.0 * a0);
  else
    b0 = sqrt(y0 * y0 - pqr[2]);
  coef[0] = 1.0;
  coef[1] = -a0;
  coef[2] = y0 - b0;
  nb_sol = solve_quadratic(coef, res);
  coef[0] = 1.0;
  coef[1] = a0;
  coef[2] = y0 + b0;
  nb_sol += solve_quadratic(coef, &(res[nb_sol]));
  return (nb_sol);
}

__host__ __device__ int		solve_quartic(double *coef, double *res)
{
  double	pqr[3];
  double	coef_tmp[4];
  double	res_tmp[3];
  int		nb_sol;
  int		count;

  count = 0;
  nb_sol = 0;
  if (fabs(coef[0]) <= SURFACE_TOLERANCE)
    return (solve_cubic(&(coef[1]), res));
  set_pqr_quartic(pqr, coef);
  init_to_send_cubic(pqr[0], pqr[1], pqr[2], coef_tmp);
  if (solve_cubic(coef_tmp, res_tmp) == 0)
    return (0);
  nb_sol = init_and_send_second(pqr, res_tmp[0], res);
  while (count < nb_sol)
    res[count++] -= coef[1] / (4.0 * coef[0]);
  return (nb_sol);
}
