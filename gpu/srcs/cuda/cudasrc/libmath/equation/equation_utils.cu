#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   equation_utils.cu                                  :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: aanzieu <marvin@42.fr>                     +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2017/06/19 14:45:23 by aanzieu           #+#    #+#             */
/*   Updated: 2017/08/24 12:08:09 by aanzieu          ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

extern "C" {
#include "../../../cudaheader/gpu_rt.h"
}

__host__ __device__ int			is_zero(double x)
{
	return(fabs(x) < ZERO_DP ? 1 : 0);
}

__host__ __device__ double		ft_smaller(double a, double b)
{
	return(a <= b ? a : b);
}

__host__ __device__ double		check_solution(double res)
{
	return((res > 0.0) ? res : -1);
}

__host__ __device__ int			dblsgn(double x)
{
	if (x < -EPSILON)
		return (-1);
	return (x > EPSILON);
}

__host__ __device__ void		swap_double(double *a, double *b)
{
	double tmp;
	tmp = *a;
	*a = *b;
	*b = tmp;
}

__host__ __device__ double    clamp(double min, double max, double value)
{
  if (value >= min && value <= max)
    return (value);
  if (value < min)
    return (min);
  return (max);
}
