#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <float.h>

extern "C" {
  #include "../../../../includes/vectors.h"
  #include "../../../../includes/rt.h"
  #include "../../cudaheader/gpu_rt.h"
}

__host__ __device__ t_color 	handle_reflection_cpu(t_world world, t_ray ray,
  t_intersection *intersection)
{
  t_intersection  intersection_tmp;

  intersection_tmp.t = DBL_MAX;
  intersection_tmp.id = intersection->id;
  intersection_tmp.type = '0';
  intersection_tmp.depth = intersection->depth + 1;
  ray.origin = intersection->pos;
  ray.dir = vector_normalize(vector_substract(ray.dir,
    vector_scalar(intersection->normal_v,
    2 * vector_dot(ray.dir, intersection->normal_v))));
  return (ray_tracer_depth_cpu(world, ray, intersection_tmp));
}

__host__ __device__ t_color 	handle_transparence_cpu(t_world world,
  t_ray ray, t_intersection *intersection)
{
  t_intersection  intersection_tmp;

  intersection_tmp.t = DBL_MAX;
  intersection_tmp.id = intersection->id;
  intersection_tmp.pos = intersection->pos;
  intersection_tmp.type = '0';
  intersection_tmp.depth = intersection->depth + 1;
  intersection_tmp.reflection_coef = 0;
  intersection_tmp.refraction_coef = 0;
  intersection_tmp.transparence_coef = 0;
  ray.origin = intersection->pos;
  return (ray_tracer_depth_cpu(world, ray, intersection_tmp));
}
__host__ __device__ void swap_double_cuda(double *a, double *b)
{
  double *tmp = a;
  a = b;
  b = tmp;
}

__host__ __device__ t_color 	handle_refraction_cpu(t_world world,
  t_ray ray, t_intersection *intersection)
{
  t_intersection  intersection_tmp;
  double          etai;
  double          cosi;
  double          eta;
  double          etat;
  double          k;
  t_vec3d         n;

  intersection_tmp.t = DBL_MAX;
  intersection_tmp.id = intersection->id;
  intersection_tmp.pos = intersection->pos;
  intersection_tmp.type = '0';
  intersection_tmp.depth = intersection->depth + 1;
  ray.origin = intersection->pos;
  cosi = clamp(-1, 1, vector_dot(ray.dir, intersection->normal_v));
  etai = 1.0;
  etat = intersection->refraction_coef;
  n = intersection->normal_v;
  if (cosi < 0)
    cosi = -cosi;
  else
  {
    swap_double_cuda(&etai, &etat);
    n = vector_scalar(n, -1);
  }
  eta = etai / etat;
  k = 1.0 - eta * eta * (1.0 - cosi * cosi);
  if (k < 0)
    ray.dir = vector_add(vector_scalar(ray.dir, eta),
        vector_scalar(n, (eta * cosi - sqrt(k))));
  return (ray_tracer_depth_cpu(world, ray, intersection_tmp));
}

__host__ __device__ t_color handle_chess(t_ray ray,
  t_intersection intersection)
{
	float			x;
	float			y;
	t_color		color;
  t_vec3d   pos;

  pos = vector_calculate(intersection.pos, intersection.normal_v);
  pos = vector_calculate(ray.origin, pos);
  if (intersection.type == 'p')
  {
		if (((int)((pos.x + 4500) * CHESS_PATTERN) ^
					(int)((pos.y + 4500) * CHESS_PATTERN) ^
					(int)((pos.z + 4500) * CHESS_PATTERN)) % 2 == 0)
			color = intersection.chess;
	else
		color = intersection.color;
  }
	else
	{
		x = atan2(intersection.normal_v.z, intersection.normal_v.x) / M_PI + 1;
	  y = acos(intersection.normal_v.y) / M_PI;
		if ((fmodf(x * CHESS_PATTERN, 1) > 0.5) ^
				(fmodf(y * CHESS_PATTERN, 1) > 0.5) == 0)
			color = intersection.chess;
		else
			color = intersection.color;
	}
	return (color);
}
