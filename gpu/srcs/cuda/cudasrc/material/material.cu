#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <float.h>

extern "C" {
  #include "../../../../includes/vectors.h"
  #include "../../../../includes/rt.h"
  #include "../../cudaheader/gpu_rt.h"
}

__host__ __device__ inline t_color color_cpy(t_color res)
{
  return ((t_color){res.r, res.g, res.b});
}

__host__ __device__ t_color 	handle_reflection(t_world world, t_ray ray,
  t_intersection *intersection)
{
	int				    	i;
	int			     		depth;
  t_color         color;
	t_intersection  intersection_tmp;

  i = 0;
  depth = intersection->reflection_coef < MAX_DEPTH ?
		intersection->reflection_coef : MAX_DEPTH;
  while (i < depth)
  {
    intersection_tmp.t = DBL_MAX;
    intersection_tmp.type = '0';
		intersection_tmp.id = intersection->id;
		intersection_tmp.pos = intersection->pos;
    ray.origin = intersection_tmp.pos;
    ray.dir = vector_normalize(vector_substract(ray.dir,
            vector_scalar(intersection->normal_v,
                    2 * vector_dot(ray.dir, intersection->normal_v))));
    get_closest_intersection(world, ray, &intersection_tmp);
    if (intersection_tmp.type != '0')
		{
			if (intersection_tmp.chess.r != -1)
				color = handle_chess(ray, &intersection_tmp);
			color = intersection_tmp.color;
			color = color_scalar(color, 0.6);
		}
    else
    {
      color = new_color(0, 0, 0);
      intersection->color = color;
      return (color);
    }
    i++;
  }
	intersection->color = color;
  return (color);
}

__host__ __device__ t_color 	handle_transparence(t_world world,
  t_ray ray, t_intersection *intersection)
{
  t_intersection  intersection_tmp;

  intersection_tmp.t = DBL_MAX;
  intersection_tmp.id = intersection->id;
  intersection_tmp.pos = intersection->pos;
  intersection_tmp.depth = intersection->depth + 1;
  return (ray_tracer_depth(world, ray, intersection_tmp));
  // intersection_tmp.type = '0';
  // ray.origin = intersection->pos;
  // ori = color_scalar(intersection->color, intersection->transparence_coef);
  // if (get_closest_intersection(world, ray, &intersection_tmp) == 1)
  // {
  //  res = intersection->color;
  //  intersection->color = intersection_tmp.color;
  //  intersection->id = intersection_tmp.id;
  //  intersection->t = DBL_MAX;
  //  intersection->depth += 1;
  // }
  //  else
  //    res = ori;
  //  return (res);
}

__host__ __device__ t_color 	handle_refraction(t_world world,
  t_ray ray, t_intersection *intersection)
{
  double tmp;
  double cosi, etai, etat, eta, k;
  t_vec3d n, dirtmp;
  t_intersection inter;
  t_color color;

  inter.t = DBL_MAX;
  inter.type = '0';
  inter.id = intersection->id;
	color = intersection->color;
  etat = intersection->refraction_coef;
  etai = 1; ///////////
  n = intersection->normal_v;
  if (vector_dot(ray.dir, n) < -1)
    cosi = -1;
  else if (cosi > 1)
    cosi = 1;
  if (cosi < 0)
    cosi = -cosi;
  else
  {
    tmp = etai;
    etai = etat;
    etat = tmp;
    n = vector_scalar(intersection->normal_v, -1);
  }
  eta = etai / etat;
  k = 1 - eta * eta * (1 - cosi * cosi);
  if (k < 0)
    return (color);
  else
  {
    ray.origin = intersection->pos;
    dirtmp = ray.dir;
    ray.dir = vector_add(vector_scalar(dirtmp, eta), vector_scalar(n, (eta * cosi - sqrt(k))));
    ray.dir = vector_normalize(ray.dir);
    if (get_closest_intersection(world, ray, &inter))
      return (inter.color);
  }
//  return (k < 0 ? 0 : eta * ray.dir + (eta * cosi - sqrt(k)) * n);
	return (color);
}
//   // double			calc[3];
// 	// double			ref_coef;
//   //
// 	// t_color			color;
// 	calc[0] = -vector_dot(intersection->normal_v, ray.dir);
// 	calc[1] = sqrt(1 - (ref_coef * ref_coef) * (1 - (calc[0] * calc[0])));
// 	if (calc[0] < 0)
// 		calc[2] = (ref_coef * calc[0] - calc[1]);
// 	else
// 		calc[2] = (ref_coef * calc[0] + calc[1]);
//  	ray.dir = vector_normalize(vector_add(
// 		vector_scalar(ray.dir, ref_coef),
// 		vector_scalar(intersection->normal_v, calc[2])));
// 	ray.origin = intersection->pos;
// 	if (get_closest_intersection(world, ray, intersection))
// 	{
// //		color = intersection->color;
// 	// 	color_scalar(&color, ref_coef);
// 	 	color = color_add(color, intersection->color);
//   }
// 	else
// 		color = (t_color){0, 0, 0};
//}

__host__ __device__ t_color handle_chess(t_ray ray,
  t_intersection *intersection)
{
	float			x;
	float			y;
	t_color		color;
  t_vec3d   pos;

  pos = vector_calculate(intersection->pos, intersection->normal_v);
  pos = vector_calculate(ray.origin, pos);
  if (intersection->type == 'p')
	{
		if (((int)((pos.x + 4500) * CHESS_PATTERN) ^
					(int)((pos.y + 4500) * CHESS_PATTERN) ^
					(int)((pos.z + 4500) * CHESS_PATTERN)) % 2 == 0)
		{
			color = intersection->chess;
			intersection->color = intersection->chess;
		}
	else
		color = intersection->color;
  }
	else
	{
		x = atan2(intersection->normal_v.z, intersection->normal_v.x) / M_PI + 1;
	  y = acos(intersection->normal_v.y) / M_PI;
		if ((fmodf(x * CHESS_PATTERN, 1) > 0.5) ^
				(fmodf(y * CHESS_PATTERN, 1) > 0.5) == 0)
		{
			color = intersection->chess;
			intersection->color =	intersection->chess;
		}
		else
			color = intersection->color;
	}

	return (color);
}
