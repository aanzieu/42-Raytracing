#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <float.h>

extern "C" {
  #include "../../../../includes/vectors.h"
  #include "../../../../includes/rt.h"
  #include "../../cudaheader/gpu_rt.h"
}

// __host__ __device__ inline t_color color_cpy(t_color res)
// {
//   return ((t_color){res.r, res.g, res.b});
// }

__host__ __device__ double    clamp(double min, double max, double value)
{
  if (value >= min && value <= max)
    return (value);
  if (value < min)
    return (min);
  return (max);
}

__host__ __device__ t_color 	handle_reflection(t_world world, t_ray ray,
  t_intersection *intersection)
{
  t_intersection  intersection_tmp;

  intersection_tmp.t = DBL_MAX;
  intersection_tmp.id = intersection->id;
  intersection_tmp.type = '0';
  intersection_tmp.depth = intersection->depth + 1;
  ray.origin = intersection->pos;
  ray.dir = vector_normalize(vector_substract(ray.dir,
    vector_scalar(intersection->normal_v,
    2 * vector_dot(ray.dir, intersection->normal_v))));
  return (ray_tracer_depth(world, ray, intersection_tmp));
}

__host__ __device__ t_color 	handle_transparence(t_world world,
  t_ray ray, t_intersection *intersection)
{
  t_intersection  intersection_tmp;

  intersection_tmp.t = DBL_MAX;
  intersection_tmp.id = intersection->id;
  intersection_tmp.pos = intersection->pos;
  intersection_tmp.type = '0';
  intersection_tmp.depth = intersection->depth + 1;
  ray.origin = intersection->pos;
  return (ray_tracer_depth(world, ray, intersection_tmp));
}

__host__ __device__ t_color 	handle_refraction(t_world world,
  t_ray ray, t_intersection *intersection)
{
  t_intersection  intersection_tmp;
  double          etai;
  double          cosi;
  double          eta;
  double          etat;
  double          k;
  t_vec3d         n;

  intersection_tmp.t = DBL_MAX;
  intersection_tmp.id = intersection->id;
  intersection_tmp.pos = intersection->pos;
  intersection_tmp.type = '0';
  intersection_tmp.depth = intersection->depth + 1;
  ray.origin = intersection->pos;
  cosi = clamp(-1, 1, vector_dot(ray.dir, intersection->normal_v));
  etai = 1.0;
  etat = intersection->refraction_coef;
  n = intersection->normal_v;
  if (cosi < 0)
    cosi = -cosi;
  else
  {
    swap_double(&etai, &etat);
    n = vector_scalar(n, -1);
  }
  eta = etai / etat;
  k = 1.0 - eta * eta * (1.0 - cosi * cosi);
  if (k < 0)
    return (intersection->color);
  ray.dir = vector_add(vector_scalar(ray.dir, eta),
        vector_scalar(n, (eta * cosi - sqrt(k))));
  return (ray_tracer_depth(world, ray, intersection_tmp));
}

__host__ __device__ t_color handle_chess(t_ray ray,
  t_intersection *intersection)
{
	float			x;
	float			y;
	t_color		color;
  t_vec3d   pos;

  pos = vector_calculate(intersection->pos, intersection->normal_v);
  pos = vector_calculate(ray.origin, pos);
  if (intersection->type == 'p')
	{
		if (((int)((pos.x + 4500) * CHESS_PATTERN) ^
					(int)((pos.y + 4500) * CHESS_PATTERN) ^
					(int)((pos.z + 4500) * CHESS_PATTERN)) % 2 == 0)
		{
			color = intersection->chess;
//			intersection->color = intersection->chess;
		}
	else
		color = intersection->color;
  }
	else
	{
		x = atan2(intersection->normal_v.z, intersection->normal_v.x) / M_PI + 1;
	  y = acos(intersection->normal_v.y) / M_PI;
		if ((fmodf(x * CHESS_PATTERN, 1) > 0.5) ^
				(fmodf(y * CHESS_PATTERN, 1) > 0.5) == 0)
		{
			color = intersection->chess;
	//		intersection->color =	intersection->chess;
		}
		else
			color = intersection->color;
	}
	return (color);
}
