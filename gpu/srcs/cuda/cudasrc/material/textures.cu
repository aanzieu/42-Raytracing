#include <hip/hip_runtime.h>
#include <float.h>

extern "C" {
  #include "../../../../includes/vectors.h"
  #include "../../../../includes/rt.h"
  #include "../../cudaheader/gpu_rt.h"
}

// __host__ __device__ t_color wood(t_world world, t_vec3d point, t_perlin perlin)
// {
//    double x = point.x * perlin.scale;
//    double y = point.y * perlin.scale;
//    double z = point.z * perlin.scale;
//
//    double grain = apply_noise(world, x, y, z) * 5;
//    grain = grain - (int)grain;
// 	 t_color color2, wood1, wood2;
// 	 wood1 = (t_color){155 / 255, 93 / 255, 73 / 255};
// 	 wood2 = (t_color){238 / 255, 170 / 255, 117 / 255};
// 	 color2 = color_add(color_scalar(wood1, grain), color_scalar(wood2, (1.0f - grain)));
//    return (color2);
// }
