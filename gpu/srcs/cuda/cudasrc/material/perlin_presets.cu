#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <float.h>

extern "C" {
  #include "../../../../includes/vectors.h"
  #include "../../../../includes/rt.h"
  #include "../../cudaheader/gpu_rt.h"
}

__host__ __device__ static t_color apply_marble(t_world world, t_vec3d pos,\
	t_color color, t_perlin perlin, int *p)
{
	double		x, y, z, coef;
	t_color		color2;
	int				level;

	coef = 0;
	level = 1;
	x = pos.x * perlin.scale * 100;
	y = pos.y * perlin.scale * 100;
	z = pos.z * perlin.scale * 100;
	while (level++ < 10)
	 	coef +=  (1.0f / level) * fabs(apply_noise(world,
		 	level * 0.05 * x, level * 0.15 * y, level * 0.05 * z, p));
	coef = 0.5f * sin((x + y) * 0.05f + coef) + 0.5f;
  color2 = (t_color){color.r / 2, color.g / 2, color.b / 2};
	return (color_add(color_scalar(color, coef),
			color_scalar(color2, (1.0f - coef))));
}

__host__ __device__ static t_color apply_wood(t_world world, t_vec3d pos,\
	t_color color, t_perlin perlin, int *p)
{
	double		x, y, z, grain;
	t_color		color2;

	x = pos.x * perlin.scale * 10;
	y = pos.y * perlin.scale * 10;
	z = pos.z * perlin.scale * 10;
	grain = apply_noise(world, x, y, z, p) * 5;
	grain = grain - (int)grain;
  color2 = (t_color){color.r / 2, color.g / 2, color.b / 2};
	return (color_add(color_scalar(color, grain),
		color_scalar(color2, (1.0f - grain))));
}

__host__ __device__ void 	apply_noise_dist(t_world world,\
				t_intersection *intersection, t_perlin perlin)
{
	if (perlin.pre_set == MARBLE)
		intersection->color = apply_marble(world, intersection->pos,\
			 intersection->color, perlin, world.p);
	else if (perlin.pre_set == WOOD)
		intersection->color = apply_wood(world, intersection->pos,\
			 intersection->color, perlin, world.p);
  if (perlin.is_set == 1)
		intersection->normal_v = normal_perturbation(world,
			intersection->normal_v, intersection->pos, perlin, world.p);
}
