#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <float.h>

extern "C" {
  #include "../../../../includes/vectors.h"
  #include "../../../../includes/rt.h"
  #include "../../cudaheader/gpu_rt.h"
}

__host__ __device__ t_color apply_marble(t_world world, t_vec3d pos,\
	t_color color, t_perlin perlin)
{
	double		x, y, z, coef;
	t_color		color2;
	int				level;

	coef = 0;
	level = 1;
	x = pos.x * perlin.scale * 100;
	y = pos.y * perlin.scale * 100;
	z = pos.z * perlin.scale * 100;
	while (level++ < 10)
	 	coef +=  (1.0f / level) * fabs(apply_noise(world,
		 	level * 0.05 * x, level * 0.15 * y, level * 0.05 * z));
	coef = 0.5f * sin((x + y) * 0.05f + coef) + 0.5f;
	color2 = (t_color){1 - color.r, 1 - color.g, 1 - color.b};
	return (color_add(color_scalar(color, coef),
			color_scalar(color2, (1.0f - coef))));
}

__host__ __device__ t_color apply_wood(t_world world, t_vec3d pos,\
	t_color color, t_perlin perlin)
{
	double		x, y, z, grain;
	t_color		color2;

	x = pos.x * perlin.scale * 10;
	y = pos.y * perlin.scale * 10;
	z = pos.z * perlin.scale * 10;
	grain = apply_noise(world, x, y, z) * 5;
	grain = grain - (int)grain;
	color2 = (t_color){1 - color.r, 1 - color.g, 1 - color.b};
	return (color_add(color_scalar(color, grain),
		color_scalar(color2, (1.0f - grain))));
}

__host__ __device__ void 	apply_noise_dist(t_world world,\
				t_intersection *intersection, t_perlin perlin)
{
	if (perlin.pre_set == MARBLE)
		intersection->color = apply_marble(world, intersection->pos,\
			 intersection->color, perlin);
	else if (perlin.pre_set == WOOD)
		intersection->color = apply_wood(world, intersection->pos,\
			 intersection->color, perlin);
 	// else if (perlin.pre_set == GLASS)
 	// 	intersection->color = apply_glass(world, intersection->pos,\
 	// 		intersection->color, perlin);
	if (perlin.is_set == 1)
		intersection->normal_v = normal_perturbation(world,
			intersection->normal_v, intersection->pos, perlin);
}
