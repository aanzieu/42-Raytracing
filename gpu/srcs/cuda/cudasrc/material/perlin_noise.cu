#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <float.h>

extern "C" {
  #include "../../../../includes/vectors.h"
  #include "../../../../includes/rt.h"
  #include "../../cudaheader/gpu_rt.h"
}

__host__ __device__ static double fade(double t)
{
   return (t * t * t * (t * (t * 6 - 15) + 10));
}

__host__ __device__ static double lerp(double t, double a, double b)
{
   return (a + t * (b - a));
}

__host__ __device__ static double grad(int hash, double x, double y, double z)
{
   int h = hash & 15;
   double u = h < 8 || h == 12 || h == 13 ? x : y;
	 double v = h < 4 || h == 12 || h == 13 ? y : z;
   return (((h & 1) == 0 ? u : -u) + ((h & 2) == 0 ? v : -v));
}

__host__ __device__ double apply_noise(t_world world, double x, double y,
	      double z)
{
   int X = (int)floor(x) & 255;
	 int Y = (int)floor(y) & 255;
	 int Z = (int)floor(z) & 255;
   x -= floor(x);
   y -= floor(y);
   z -= floor(z);
	 double u = fade(x);
	 double v = fade(y);
	 double w = fade(z);
	 int A = world.p[X] + Y;
	 int AA = world.p[A] + Z;
	 int AB = world.p[A + 1] + Z;
	 int B = world.p[X + 1] + Y;
	 int BA = world.p[B] + Z;
	 int BB = world.p[B + 1] + Z;
	 return (lerp(w, lerp(v, lerp(u, grad(world.p[AA], x, y, z),
			       grad(world.p[BA], x - 1, y, z)),
						 	lerp(u, grad(world.p[AB], x, y - 1, z),
			    	 		grad(world.p[BB], x - 1, y - 1, z))),
						 			lerp(v, lerp(u, grad(world.p[AA + 1], x, y, z - 1),
						 				grad(world.p[BA + 1], x - 1, y, z - 1)),
			               	lerp(u, grad(world.p[AB + 1], x, y - 1, z - 1),
			              		grad(world.p[BB + 1], x - 1, y - 1, z - 1)))));
}

__host__ __device__ t_vec3d normal_perturbation(t_world world, t_vec3d normal,
				t_vec3d intersection_pos, t_perlin perlin)
{
   t_vec3d noise;
	 t_vec3d normal_pert;
	 double x, y, z;
	//  double scale = 0.7;
	//  double amount = 2;

   x = intersection_pos.x / perlin.scale;
   y = intersection_pos.y / perlin.scale;
   z = intersection_pos.z / perlin.scale;
   noise.x = (float)(apply_noise(world, x, y, z));
   noise.y = (float)(apply_noise(world, y, z, x));
   noise.z = (float)(apply_noise(world, z, x, y));
	 normal_pert = vector_normalize(vector_add(normal,
		 	vector_scalar(noise, perlin.amount)));
   return (normal_pert);
}
