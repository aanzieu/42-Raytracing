#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   shader.c                                           :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: svilau <marvin@42.fr>                      +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2016/10/20 10:49:50 by svilau            #+#    #+#             */
/*   Updated: 2017/03/17 14:07:23 by svilau           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

extern "C" {
	#include "rt.h"
	#include "gpu_rt.h"
	#include <equation.h>
	#include <time.h>
}
#include <float.h>

__host__ __device__ t_color	direct_light(t_world world, t_color color,\
	t_light light, t_ray ray)
{
	t_vec3d	light_vector;
	t_color	direct_light;
	double 	angle;

	direct_light = new_color(10, 10, 10);
	light_vector = light.dir_v;
	angle = vector_dot(vector_scalar(world.camera.dir_v, -1), light_vector);
	direct_light = color_scalar(direct_light, angle);
	return (color_add(color, direct_light));
}

__host__ __device__ static double  get_distributed_shadow(t_world world,\
	t_light light, t_intersection collision)
{
	t_intersection	collision_tmp;
	t_light			box[9];
	t_ray				shadow;
	double			dist_intersection, coef;
	int 				i, j;

	i = 0;
	coef = 0;
	shadow.origin = collision.pos;
	while (i++ < 3)
	{
		j = 0;
		while (j++ < 3)
		{
			new_intersection(&collision_tmp);
			box[i].pos = (t_vec3d){light.pos.x + (0.05 * i), light.pos.y,
					light.pos.z + (0.05 * j)};
			shadow.dir = vector_calculate(collision.pos, box[i].pos);
			if (get_closest_intersection(world, shadow, &collision_tmp) == 1)
			{
				dist_intersection = vector_length(vector_calculate(collision.pos,
					collision_tmp.pos));
				if (dist_intersection < vector_length(shadow.dir))
					coef += collision_tmp.transparence_coef > 0 ?
						collision_tmp.transparence_coef : 1;
			}
		}
	}
	return (coef <= 0 ? 0 : 0.0000001 + (1 - (coef / 9)));
}

__host__ __device__ double		get_shadow(t_world world, t_light light,
	t_intersection collision)
{
	t_intersection	collision_tmp;
	t_ray				shadow;
	double			dist_light;
	double			dist_intersection;

	if (light.type == LIGHT_BOX)
		return (get_distributed_shadow(world, light, collision));
	collision_tmp.t = DBL_MAX;
	collision_tmp.type = '0';
	shadow.dir = vector_calculate(collision.pos, light.pos);
	shadow.origin = collision.pos;
	dist_light = vector_length(shadow.dir);
	collision_tmp.id = -1;
	if (get_closest_intersection(world, shadow, &collision_tmp) == 1)
	{
		dist_intersection = vector_length(vector_calculate(collision.pos,
												collision_tmp.pos));
		if (dist_intersection < dist_light)
			return (collision_tmp.transparence_coef > 0 ? collision_tmp.transparence_coef : -1);
	}
	return (0);
}

__host__ __device__ t_color	specular_light(t_world world, t_color color,\
	t_vec3d ray, t_intersection intersection, t_vec3d light_vector, t_light light)
{
	t_vec3d reflected_vector;
	t_vec3d raypos_tmp;
	double	specular_angle;

	raypos_tmp = vector_scalar(intersection.normal_v, 2 * vector_dot(ray, intersection.normal_v));
	reflected_vector = vector_substract(ray, raypos_tmp);
	reflected_vector = vector_normalize(reflected_vector);
	specular_angle = pow(vector_dot(light_vector, reflected_vector), 150);
	light.color = color_scalar(light.color, specular_angle);
	light.color = color_scalar(light.color, 0);
	//light.color = color_scalar(light.color, intersection.reflection_coef);
	light.color = color_scalar(light.color, light.intensity_coef);
	return (color_add(color, light.color));
}

__host__ __device__ t_vec3d	get_light_vector(t_world world, t_intersection intersection, t_light light)
{
	if (world.light_type == 1)
		return(vector_normalize(vector_calculate(intersection.pos, light.pos)));
	else
		return(light.dir_v);
}

__host__ __device__	t_color	get_light_at(t_world world, t_color color,\
	t_light light, t_intersection intersection, t_ray ray)
{
	t_vec3d		light_vector;
	t_color		tmp;
	double		angle, shadow_coef;

	tmp =  new_color(0, 0, 0);
	light_vector = get_light_vector(world, intersection, light);
	angle = vector_dot(intersection.normal_v, light_vector);
	shadow_coef = get_shadow(world, light, intersection);
	if (angle > 0 && shadow_coef >= 0)
	{
		tmp = color_add(color, intersection.color);
		tmp = color_scalar(tmp, angle);
		tmp = color_scalar(tmp, light.intensity_coef);
		if (shadow_coef > 0)
			tmp = color_scalar(tmp, shadow_coef);
		color = color_add(color, tmp);
		color = specular_light(world, color, ray.dir, intersection, light_vector, light);
	}
	return (color);
}
