#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   shader.c                                           :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: svilau <marvin@42.fr>                      +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2016/10/20 10:49:50 by svilau            #+#    #+#             */
/*   Updated: 2017/03/17 14:07:23 by svilau           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

extern "C" {
	#include "rt.h"
	#include "gpu_rt.h"
	#include <equation.h>
}
#include <float.h>

__host__ __device__ int		get_shadow(t_world world, t_light light,
													t_intersection collision)
{
	t_intersection	collision_tmp;
	t_ray			shadow;
	double			dist_light;
	double			dist_intersection;

	collision_tmp.t = DBL_MAX;
	collision_tmp.type = '0';
	shadow.dir = vector_calculate(collision.pos, light.pos);
	shadow.origin = collision.pos;
	dist_light = vector_length(shadow.dir);
	collision_tmp.id = -1;
	if (get_closest_intersection(world, shadow, &collision_tmp) == 1)
	{
		dist_intersection = vector_length(vector_calculate(collision.pos,
												collision_tmp.pos));
		if (dist_intersection < dist_light)
			return (1);
	}
	return (0);
}

__host__ __device__ t_color	specular_light(t_world world, t_color color, t_vec3d ray, t_intersection intersection,
													t_vec3d light_vector, t_light light)
{
	t_vec3d reflected_vector;
	t_vec3d raypos_tmp;
	double	specular_angle;

	raypos_tmp = vector_scalar(intersection.normal_v, 2 * vector_dot(ray, intersection.normal_v));
	reflected_vector = vector_substract(ray, raypos_tmp);
	reflected_vector = vector_normalize(reflected_vector);
	specular_angle = pow(vector_dot(light_vector, reflected_vector), 150);
	light.color = color_scalar(light.color, specular_angle);
	//light.color = color_scalar(light.color, intersection.reflection_coef);
	light.color = color_scalar(light.color, light.intensity_coef);
	return (color_add(color, light.color));
}

__host__ __device__ t_vec3d	get_light_vector(t_world world, t_intersection intersection, t_light light)
{
	if (world.light_type == 1)
		return(vector_normalize(vector_calculate(intersection.pos, light.pos)));
	else
		return(light.dir_v);
}

__host__ __device__	t_color	get_light_at(t_world world, t_color color, t_light light,
													t_intersection intersection, t_ray ray)
{
	t_vec3d		light_vector;
	t_color		tmp;
	double		angle;

	tmp =  new_color(0, 0, 0);
	light_vector = get_light_vector(world, intersection, light);
	angle = vector_dot(intersection.normal_v, light_vector);
	if (angle > 0 && get_shadow(world, light, intersection) == 0)
	{
		tmp = color_add(color, intersection.color);
		tmp = color_scalar(tmp, angle);
		tmp = color_scalar(tmp, light.intensity_coef);
		color = color_add(color, tmp);
		color = specular_light(world, color, ray.dir, intersection, light_vector, light);
	}
	return (color);
}

//		color = color_add(color_scalar(tmp, 0.5), color_scalar(color2, 0.5));
