#include "hip/hip_runtime.h"
extern "C" {
	#include <rt.h>
	#include <gpu_rt.h>
	#include <vectors.h>
	#include <equation.h>
}

#pragma hd_warning_disable

__host__ __device__ Uint32 	getpixel(SDL_Surface *surface, int x, int y)
{
  int		bpp;
  Uint8 *p;

	bpp = surface->format->BytesPerPixel;
	p = (Uint8 *)surface->pixels + y * surface->pitch + x * bpp;
  if (bpp == 1)
    return (*p);
  else if (bpp == 2)
    return (*(Uint16 *)p);
  else if (bpp == 3)
    if (SDL_BYTEORDER == SDL_BIG_ENDIAN)
      return (p[0] << 16 | p[1] << 8 | p[2]);
    else
      return (p[0] | p[1] << 8 | p[2] << 16);
  else if (bpp == 4)
    return (*(Uint32 *)p);
	return (0);
}

__host__ __device__ t_color load_texture_at(t_texture texture, t_ray ray,\
		t_intersection *intersection, t_world world)
{
	Uint32		pixel;
	Uint8			r, g, b;
	double 		u, v;
	t_color 	ret;

	u = (0.5 + atanf(intersection->normal_v.x / -(intersection->normal_v.z
		+ SURFACE_TOLERANCE)) / (2 * M_PI)) * texture.tex->w;
	v = (0.5 - asin(intersection->normal_v.y) / M_PI) * texture.tex->h;
	u = fmod(u, (double)(texture.tex->w - 1.0));
	v = fmod(v, (double)(texture.tex->h - 1.0));
	pixel = getpixel(texture.tex, u, v);
	SDL_GetRGB(pixel, texture.tex->format, &r, &g, &b);
	ret.r = (double)(r) / 255;
	ret.g = (double)(g) / 255;
	ret.b = (double)(b) / 255;
	return (ret);
}