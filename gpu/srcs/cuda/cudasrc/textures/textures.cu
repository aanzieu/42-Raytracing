#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <float.h>

extern "C" {
  #include "../../../../includes/vectors.h"
  #include "../../../../includes/rt.h"
  #include "../../cudaheader/gpu_rt.h"
}

__host__ __device__ t_color 	handle_reflection(t_world world, t_ray ray,
															t_intersection *intersection)
{
	int				i;
	int				depth;
	t_color		color = (t_color){0, 0, 0};

	i = 0;
	int j = 0;
	depth = intersection->reflection_coef < MAX_DEPTH ? intersection->reflection_coef : MAX_DEPTH;
	while(i < depth && j < depth)
	{
		intersection->t = DBL_MAX;
		intersection->type = '0';
		ray.origin = intersection->pos;
		ray.dir = vector_normalize(vector_substract(ray.dir,
			vector_scalar(intersection->normal_v,
				2 * vector_dot(ray.dir, intersection->normal_v))));
		get_closest_intersection(world, ray, intersection);
		if (intersection->type != '0')
		{
			color_add(&color, *intersection->color);
			if (intersection->reflection_coef == 0)
				i++;
		}
		else
		{
			color = (t_color){0, 0, 0};
			*intersection->color = (t_color){0, 0, 0};
			return (color);
		}
		i++;
		j++;
	}
	*intersection->color = color;
	return (color);
}

__host__ __device__ t_color 	handle_refraction_transparence(t_world world,
																			t_ray ray, t_intersection *intersection)
{
	double			calc[3];
	double			ref_coef;
	t_color			color;

	intersection->t = DBL_MAX;
	intersection->type = '0';
	color = *intersection->color;
 	ref_coef = intersection->refraction_coef;
	if (ref_coef > 1)
	{
		calc[0] = vector_dot(intersection->normal_v, ray.dir);
		calc[1] = sqrt(1 - (ref_coef * ref_coef) * (1 - (calc[0] * calc[0])));
		if (calc[0] > 0)
			calc[2] = (ref_coef * calc[0] - calc[1]);
		else
			calc[2] = (ref_coef * calc[0] + calc[1]);
	 	ray.dir = vector_normalize(vector_add(
			vector_scalar(ray.dir, ref_coef),
			vector_scalar(intersection->normal_v, calc[2])));
	}
	ray.origin = intersection->pos;
	if (get_closest_intersection(world, ray, intersection))
	{
		color = *intersection->color;
		// color_scalar(&color, ref_coef);
		// color_add(&color, *intersection->color);
	}
	else
		color = (t_color){0, 0, 0};
	*intersection->color = color;
	return (color);
}

__host__ __device__ t_color chess_effect(t_intersection *intersection)
{
	float			x;
	float			y;
	t_color		color;

  if (intersection->type == 'p')
	{
		if (((int)((intersection->pos.x + 450) * CHESS_PATTERN) ^
					(int)((intersection->pos.y + 450) * CHESS_PATTERN) ^
					(int)((intersection->pos.z + 450) * CHESS_PATTERN)) % 2 == 0)
		{
			color = *intersection->chess;
			intersection->color = intersection->chess;
		}
		else
			color = *intersection->color;
	}
	else
	{
		x = atan2(intersection->normal_v.z, intersection->normal_v.x) / M_PI + 1;
	  y = acos(intersection->normal_v.y) / M_PI;
		if ((fmodf(x * CHESS_PATTERN, 1) > 0.5) ^
				(fmodf(y * CHESS_PATTERN, 1) > 0.5) == 0)
		{
			color = *intersection->chess;
			intersection->color =	intersection->chess;
		}
		else
			color = *intersection->color;
	}

	return (color);
}
