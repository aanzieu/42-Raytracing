#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <float.h>

extern "C" {
  #include "../../../../includes/vectors.h"
  #include "../../../../includes/rt.h"
  #include "../../cudaheader/gpu_rt.h"
}

__host__ __device__ t_color 	handle_refraction_transparence(t_world world,
																			t_ray ray, t_intersection *intersection)
{
	double			calc[3];
	double			ref_coef;
	t_color			color;

	intersection->t = DBL_MAX;
	intersection->type = '0';
	color = *intersection->color;
 	ref_coef = intersection->refraction_coef;
	if (ref_coef > 1)
	{
		calc[0] = vector_dot(intersection->normal_v, ray.dir);
		calc[1] = sqrt(1 - (ref_coef * ref_coef) * (1 - (calc[0] * calc[0])));
		if (calc[0] < 0)
			calc[2] = (ref_coef * calc[0] - calc[1]);
		else
			calc[2] = (ref_coef * calc[0] + calc[1]);
	 	ray.dir = vector_normalize(vector_add(
			vector_scalar(ray.dir, ref_coef),
			vector_scalar(intersection->normal_v, calc[2])));
	}
	ray.origin = intersection->pos;
	if (get_closest_intersection(world, ray, intersection))
	{
		color = *intersection->color;
	// 	color_scalar(&color, ref_coef);
	// 	color_add(&color, *intersection->color);
	 }
	else
		color = (t_color){0, 0, 0};
	return (color);
}

__host__ __device__ t_color chess_effect(t_intersection *intersection)
{
	float			x;
	float			y;
	t_color		color;

  if (intersection->type == 'p')
	{
		if (((int)((intersection->pos.x + 450) * CHESS_PATTERN) ^
					(int)((intersection->pos.y + 450) * CHESS_PATTERN) ^
					(int)((intersection->pos.z + 450) * CHESS_PATTERN)) % 2 == 0)
		{
			color = *intersection->chess;
			intersection->color = intersection->chess;
		}
		else
			color = *intersection->color;
	}
	else
	{
		x = atan2(intersection->normal_v.z, intersection->normal_v.x) / M_PI + 1;
	  y = acos(intersection->normal_v.y) / M_PI;
		if ((fmodf(x * CHESS_PATTERN, 1) > 0.5) ^
				(fmodf(y * CHESS_PATTERN, 1) > 0.5) == 0)
		{
			color = *intersection->chess;
			intersection->color =	intersection->chess;
		}
		else
			color = *intersection->color;
	}

	return (color);
}
