#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <float.h>

extern "C" {
#include "../../../includes/display.h"
#include "../../../includes/vectors.h"
#include "../../../includes/rt.h"
#include "../cudaheader/gpu_rt.h"
}


#define CUDA_ERROR_CHECK
#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
#define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )

inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
	if ( hipSuccess != err )
	{
		fprintf( stderr, "cudaSafeCall() failed at %s:%i : %s\n",
				file, line, hipGetErrorString( err ) );
		exit(-1);
	}
#endif
	return;
}


void checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
	system("pause");
    exit(EXIT_FAILURE); 
  }
} 

//inline void __cudaCheckError( const char *file, const int line )
//{
//#ifdef CUDA_ERROR_CHECK
//	hipError_t err = hipGetLastError();
//	if (hipSuccess != err)
//	{
//		fprintf( stderr, "cudaCheckError() failed at %s:%i : %s\n",
//				file, line, hipGetErrorString( err ) );
//		exit( -1 );
//	}
//#endif
//	return;
//}

__host__ __device__ void 	color_scal(t_color *c1, double coef)
{
	c1->r *= coef;
	c1->g *= coef;
	c1->b *= coef;
}

__host__ __device__	void	color_sum(t_color *c1, t_color *c2)
{
	c1->r += c2->r;
	c1->g += c2->g;
	c1->b += c2->b;
}


__host__ __device__ void apply_recurse(t_world *world, t_ray *ray,
	t_intersection *intersection, t_color *color, int depth)
{
	if (depth > MAX_DEPTH) {
		color->r = 0; 
		color->g = 0;
		color->b = 0;
		return ;
	}
	else {
		if (intersection->transparence_coef > 0) {
		
			t_intersection intersection_tmp;
			t_ray			ray_tmp;

			ray_tmp.origin = intersection->pos;
			ray_tmp.dir = ray->dir;
			get_closest_intersection(*world, ray_tmp, &intersection_tmp);
			color_scal(color, 1 - intersection->transparence_coef);
			apply_recurse(world, &ray_tmp, &intersection_tmp, color, depth + 1);
		}
	}
}


__global__ void test_recursive_cuda(int *a, unsigned int constw, unsigned int consth, t_world *world, int depth)
{
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int index = row * constw + col;

	t_ray	ray;
	t_intersection intersection;

	new_intersection(&intersection);
	get_up_left(world);
	get_ray_direction(*world, &ray, col, row);

	t_color					color;
	int							i;

	i = 0;
	color = new_color(0, 0, 0);
	if (get_closest_intersection(*world, ray, &intersection)) {
			//if(world.keys.light_none == 1 && world.mode == 0)
			//	apply_recurse(world, &ray, &intersection, &color, depth);

			color = color_multiply(color, world->ambient.color);
			color = color_scalar(color, world->ambient.intensity);
			if(world->keys.light_none == 0)
				color = intersection.color;
		//	if(world.keys.select == 1)
		//		cartoon_effect(world, &color, world.lights[i], intersection, ray);
			while (i < world->lights_len && world->keys.light_none == 1)
			{
				color = get_light_at(*world, color, world->lights[i], intersection, ray);
		//		if (world.keys.pad_0 == 6)
		//			cartoon_effect(world, &color, world.lights[i], intersection, ray);
				i++;
			}
		}
//}
	a[index] = get_color(color);
}



//__global__ void test(int *a, unsigned int constw, unsigned int consth, t_world world)
//{
//	int col = blockIdx.x * blockDim.x + threadIdx.x;
//	int row = blockIdx.y * blockDim.y + threadIdx.y;
//	int index = row * constw + col;
//	a[index] = ray_tracer(world, col, row + world.offsets.y_min);
//}

extern "C" void render_cuda(int *a_h, unsigned int constw, unsigned int consth, t_world *world, int reset)
{
	int 		*a_d = 0;
	t_sphere	*spheres_d = NULL;
	t_plane		*planes_d = NULL;
	t_cylinder	*cylinders_d = NULL;
	t_cone		*cones_d = NULL;
	t_light		*lights_d = NULL;
	// static t_paraboloid	*paraboloids_d;
	
	dim3		threads_per_block(32, 32);
	dim3		grid_size(constw / threads_per_block.x, consth / threads_per_block.y);

	//*****   init size of map    ******

	size_t		size = 0;
	size = constw * consth * sizeof(int);
	hipMalloc(&a_d, size);

	int	depth = 1;
	//*****   mcudamalloc obj   ******

	hipMalloc(&spheres_d, sizeof(t_sphere) * world->spheres_len);
	hipMalloc(&planes_d, sizeof(t_plane) * world->planes_len);
	hipMalloc(&cylinders_d, sizeof(t_cylinder) * world->cylinders_len);
	hipMalloc(&cones_d, sizeof(t_cone) * world->cones_len);
	hipMalloc(&lights_d, sizeof(t_light) * world->lights_len);

	hipMemcpy(spheres_d, world->spheres, sizeof(t_sphere) * world->spheres_len, hipMemcpyHostToDevice);
//		world.spheres = spheres_d;
	hipMemcpy(planes_d, world->planes, sizeof(t_plane) * world->planes_len, hipMemcpyHostToDevice);
//		world.planes = planes_d;
	hipMemcpy(cylinders_d, world->cylinders, sizeof(t_cylinder) * world->cylinders_len, hipMemcpyHostToDevice);
//		world.cylinders = cylinders_d;
	hipMemcpy(cones_d, world->cones, sizeof(t_cone) * world->cones_len, hipMemcpyHostToDevice);
//		world.cones = cones_d;
	hipMemcpy(lights_d, world->lights, sizeof(t_light) * world->lights_len, hipMemcpyHostToDevice);
//		world.lights = lights_d;
	

	//size_t limits;
  	//hipDeviceSetLimit(hipLimitStackSize, 1024*sizeof(float));
  	//hipDeviceGetLimit(&limits, hipLimitStackSize);

//	checkCUDAError("pre-raytraceRay error");

	//test <<< grid_size, threads_per_block>>> (a_d, constw, consth, world);

	test_recursive_cuda <<< grid_size, threads_per_block>>> (a_d, constw, consth, world, depth);
//	checkCUDAError("raytraceRay error");

	hipMemcpy(a_h, a_d, size, hipMemcpyDeviceToHost);

 	//free up stuff, or else we'll leak memory like a madman	
//	if(spheres_d != NULL)
		hipFree(spheres_d);
//	 if(planes_d != NULL)
	 	hipFree(planes_d);
//	if(cones_d != NULL)
		hipFree(cones_d);
//	if(cylinders_d != NULL)
		hipFree(cylinders_d);
//	if(lights_d != NULL)
		hipFree(lights_d);
	//if(a_d != NULL)
		hipFree(a_d);

	hipDeviceSynchronize();
//	checkCUDAError("Kernel Error");
}