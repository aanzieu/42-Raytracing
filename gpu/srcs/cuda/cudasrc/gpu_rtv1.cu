#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <float.h>
//#include "./cuPrintf.cu"

extern "C" {
#include "../../../includes/display.h"
#include "../../../includes/vectors.h"
#include "../../../includes/rt.h"
#include "../cudaheader/gpu_rt.h"
}

#define CUDA_ERROR_CHECK
#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
#define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )

inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
	if ( hipSuccess != err )
	{
		fprintf( stderr, "cudaSafeCall() failed at %s:%i : %s\n",
				file, line, hipGetErrorString( err ) );
		exit(-1);
	}
#endif
	return;
}


void checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
	system("pause");
    exit(EXIT_FAILURE);
  }
}

// __global__ void sendImageToWorld(int* a_h, int x, int y, int *a_d)
// {
//
//   int xx = (blockIdx.x * blockDim.x) + threadIdx.x;
//   int yy = (blockIdx.y * blockDim.y) + threadIdx.y;
//   int index = xx + (yy * x);
//
//   if(xx <= x && yy <= y)
//     a_h[index] = a_d[index];
// }

__global__ void test(int *a, unsigned int constw, unsigned int consth, t_world world)
{
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int index = row * constw + col;
	int yy = world.aa * row + world.offsets.y_min; 					//thread->y_max * thread->world->aa + world.offsets.y_min);
	int xx = world.aa * col; //
	// if (col < constw && row < consth)
		a[index] = ray_tracer_gpu(world, xx, yy);// + world.offsets.y_min);
}



//__global__ void test(int *a, unsigned int constw, unsigned int consth, t_world world)
//{
//	int col = blockIdx.x * blockDim.x + threadIdx.x;
//	int row = blockIdx.y * blockDim.y + threadIdx.y;
//	int index = row * constw + col;
//	a[index] = ray_tracer(world, col, row + world.offsets.y_min);
//}

extern "C" void render_cuda(int *a_h, unsigned int constw, unsigned int consth, t_world world, int reset)
{
	int 					*a_d = 0;
	size_t		size = constw * consth * sizeof(int);
 	hipMalloc(&a_d, size);



	t_sphere				*spheres_d = NULL;
	t_plane					*planes_d = NULL;
	t_cylinder			*cylinders_d = NULL;
	t_cone				*cones_d = NULL;
	t_disk				*disks_d = NULL;
	t_torus					*torus_d = NULL;
	t_mobius				*mobius_d = NULL;
	t_cube				*cubes_d = NULL;
	t_triangle			*triangles_d = NULL;
	t_paraboloid			*paraboloids_d = NULL;
	t_hyperboloid		*hyperboloids_d = NULL;
	t_light					*lights_d = NULL;

	dim3			threads_per_block(32, 32);
	dim3			grid_size(constw / threads_per_block.x, consth / threads_per_block.y);

	// checkCUDAError("test4");
	// hipMalloc((void**)&a_d, (int)constw * (int)consth * sizeof(int));
 	// hipMemcpy(a_d, a_h, (int)constw * (int)consth * sizeof(int), hipMemcpyHostToDevice);
	// checkCUDAError("before malloc objs");
	
	
	hipMalloc(&spheres_d, sizeof(t_sphere) * world.spheres_len);
	hipMalloc(&planes_d, sizeof(t_plane) * world.planes_len);
	
	hipMalloc(&cylinders_d, sizeof(t_cylinder) * world.cylinders_len);
	hipMalloc(&cubes_d, sizeof(t_cube) * world.cubes_len);
	hipMalloc(&triangles_d, sizeof(t_triangle) * world.triangles_len);
	
	hipMalloc(&torus_d, sizeof(t_torus) * world.torus_len);
	
	hipMalloc(&mobius_d, sizeof(t_mobius) * world.mobius_len);
	hipMalloc(&paraboloids_d, sizeof(t_paraboloid) * world.paraboloids_len);
	hipMalloc(&hyperboloids_d, sizeof(t_hyperboloid) * world.hyperboloids_len);
	hipMalloc(&cones_d, sizeof(t_cone) * world.cones_len);
	hipMalloc(&disks_d, sizeof(t_disk) * world.disks_len);
	
	hipMalloc(&lights_d, sizeof(t_light) * world.lights_len);
	
	// checkCUDAError("after malloc objs -- before memcpy objs");
	
	hipMemcpy(spheres_d, world.spheres, sizeof(t_sphere) * world.spheres_len, hipMemcpyHostToDevice);
	hipMemcpy(planes_d, world.planes, sizeof(t_plane) * world.planes_len, hipMemcpyHostToDevice);
	
	hipMemcpy(cylinders_d, world.cylinders, sizeof(t_cylinder) * world.cylinders_len, hipMemcpyHostToDevice);
	hipMemcpy(cubes_d, world.cubes, sizeof(t_cube) * world.cubes_len, hipMemcpyHostToDevice);
	hipMemcpy(triangles_d, world.triangles, sizeof(t_triangle) * world.triangles_len, hipMemcpyHostToDevice);
	
	hipMemcpy(torus_d, world.torus, sizeof(t_torus) * world.torus_len, hipMemcpyHostToDevice);
	
	hipMemcpy(mobius_d, world.mobius, sizeof(t_mobius) * world.mobius_len, hipMemcpyHostToDevice);
	hipMemcpy(paraboloids_d, world.paraboloids, sizeof(t_paraboloid) * world.paraboloids_len, hipMemcpyHostToDevice);
	hipMemcpy(hyperboloids_d, world.hyperboloids, sizeof(t_hyperboloid) * world.hyperboloids_len, hipMemcpyHostToDevice);
	hipMemcpy(cones_d, world.cones, sizeof(t_cone) * world.cones_len, hipMemcpyHostToDevice);
	hipMemcpy(disks_d, world.disks, sizeof(t_disk) * world.disks_len, hipMemcpyHostToDevice);
	
	hipMemcpy(lights_d, world.lights, sizeof(t_light) * world.lights_len, hipMemcpyHostToDevice);
	
	// checkCUDAError("after memcpy -- before world obj cpy");

	world.planes = planes_d;
	world.spheres = spheres_d;
	world.cylinders = cylinders_d;
	world.torus = torus_d;
	world.mobius = mobius_d;
	world.cubes = cubes_d;
	world.triangles = triangles_d;
	world.cones = cones_d;
	world.hyperboloids = hyperboloids_d;
	world.paraboloids = paraboloids_d;
	world.disks = disks_d;
	world.lights = lights_d;
	// hipSetDevice(device);
	// hipDeviceSetLimit(cudaLimitDevRuntimeSyncDepth, 0);
	// checkCUDAError("before RT");
	
	test <<< grid_size, threads_per_block >>> (a_d, constw, consth, world);
	
	// checkCUDAError("after RT");
	// hipDeviceSynchronize();
	// checkCUDAError("end4");

	// checkCUDAError("before free");
	if(spheres_d != NULL)
		hipFree(spheres_d);
	if(planes_d != NULL)
		hipFree(planes_d);
	 if(cubes_d != NULL)
	 	hipFree(cubes_d);
	 if(cylinders_d != NULL)
	 	hipFree(cylinders_d);
	 if(cones_d != NULL)
	 	hipFree(cones_d);
	 if(paraboloids_d != NULL)
	 	hipFree(paraboloids_d);
	 if(hyperboloids_d != NULL)
	 	hipFree(hyperboloids_d);
	 if(triangles_d != NULL)
	 	hipFree(triangles_d);
	 if(disks_d != NULL)
	 	hipFree(disks_d);
	if(torus_d != NULL)
		hipFree(torus_d);
	 if(mobius_d != NULL)
	 	hipFree(mobius_d);
	if(lights_d != NULL)
		hipFree(lights_d);
	// checkCUDAError("after free");

	hipMemcpy(a_h, a_d, size, hipMemcpyDeviceToHost);

	if(a_d != NULL)
		hipFree(a_d);
	// checkCUDAError("end");
	// checkCUDAError("end");
}


//
// int nDevices;
//
// hipGetDeviceCount(&nDevices);
// for (int i = 0; i < nDevices; i++) {
// 	hipDeviceProp_t prop;
// 	hipGetDeviceProperties(&prop, i);
// 	printf("Device Number: %d\n", i);
// 	printf("  Device name: %s\n", prop.name);
// 	printf("  Major: %d\n", prop.major);
// 	printf("  Minor: %d\n", prop.minor);
// 	printf("  Memory Clock Rate (KHz): %d\n",
// 		   prop.memoryClockRate);
// 	printf("  Memory Bus Width (bits): %d\n",
// 		   prop.memoryBusWidth);
// 	printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
// 		   2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
// }
