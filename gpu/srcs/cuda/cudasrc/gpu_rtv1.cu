#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <float.h>
//#include "./cuPrintf.cu"

extern "C" {
#include "../../../includes/display.h"
#include "../../../includes/vectors.h"
#include "../../../includes/rt.h"
#include "../cudaheader/gpu_rt.h"
}

#define CUDA_ERROR_CHECK
#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
#define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )


inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
	if ( hipSuccess != err )
	{
		fprintf( stderr, "cudaSafeCall() failed at %s:%i : %s\n",
				file, line, hipGetErrorString( err ) );
		exit(-1);
	}
#endif
	return;
}

void checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if ( hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
	system("pause");
    exit(EXIT_FAILURE);
  }
}

__global__ void test(int *a, unsigned int constw, unsigned int consth, t_world world)
{
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int index = row * constw + col;
	int yy = world.aa * row + world.offsets.y_min;
	int xx = world.aa * col;
	a[index] = ray_tracer_gpu(world, xx, yy);
}

extern "C" void render_cuda(int *a_h, unsigned int constw, unsigned int consth, t_world world, int reset)
{
	int 					*a_d = 0;
	int 					*p;
	size_t				size = constw * consth * sizeof(int);

	dim3						threads_per_block(32, 32);
	dim3						grid_size(constw / threads_per_block.x, consth / threads_per_block.y);

	t_sphere				*spheres_d = NULL;
	t_plane					*planes_d = NULL;
	t_cylinder			*cylinders_d = NULL;
	t_cone					*cones_d = NULL;
	t_disk					*disks_d = NULL;
	t_torus					*torus_d = NULL;
	t_mobius				*mobius_d = NULL;
	t_cube					*cubes_d = NULL;
	t_h_cube				*h_cubes_d = NULL;
	t_triangle			*triangles_d = NULL;
	t_paraboloid		*paraboloids_d = NULL;
	t_hyperboloid		*hyperboloids_d = NULL;
	t_light					*lights_d = NULL;

	static int p_test[] = {
		151, 160, 137, 91, 90, 15, 131, 13, 201, 95, 96, 53, 194, 233, 7, 225, 140,
		36, 103, 30, 69, 142, 8, 99, 37, 240, 21, 10, 23, 190, 6, 148, 247, 120,
		234, 75, 0, 26, 197, 62, 94, 252, 219, 203, 117, 35, 11, 32, 57, 177, 33,
		88, 237, 149, 56, 87, 174, 20, 125, 136, 171, 168, 68, 175, 74, 165, 71,
		134, 139, 48, 27, 166, 77, 146, 158, 231, 83, 111, 229, 122, 60, 211, 133,
		230, 220, 105, 92, 41, 55, 46, 245, 40, 244, 102, 143, 54, 65, 25, 63, 161,
		1, 216, 80, 73, 209, 76, 132, 187, 208, 89, 18, 169, 200, 196, 135, 130,
		116, 188, 159, 86, 164, 100, 109, 198, 173, 186, 3, 64, 52, 217, 226, 250,
		124, 123, 5, 202, 38, 147, 118, 126, 255, 82, 85, 212, 207, 206, 59, 227,
		47, 16, 58, 17, 182, 189, 28, 42, 23, 183, 170, 213, 119, 248, 152, 2, 44,
		154, 163, 70, 221, 153, 101, 155, 167, 43, 172, 9, 129, 22, 39, 253, 19, 98,
		108, 110, 79, 113, 224, 232, 178, 185, 112, 104, 218, 246, 97, 228, 251, 34,
		242, 193, 238, 210, 144, 12, 191, 179, 162, 241, 81, 51, 145, 235, 249, 14,
		239, 107, 49, 192, 214,  31, 181, 199, 106, 157, 184,  84, 204, 176, 115,
		121, 50, 45, 127, 4, 150, 254, 138, 236, 205, 93, 222, 114, 67, 29, 24, 72,
		243, 141, 128, 195, 78, 66, 215, 61, 156, 180
	};
	hipMalloc(&p, sizeof(int) * 512);
	hipMemcpy(p, p_test, sizeof(int) * 256, hipMemcpyHostToDevice);
	hipMemcpy((p + 256), p_test, sizeof(int) * 256, hipMemcpyHostToDevice);
	world.p = p;

	hipMalloc(&a_d, size);
	hipMalloc(&spheres_d, sizeof(t_sphere) * world.spheres_len);
	hipMalloc(&planes_d, sizeof(t_plane) * world.planes_len);
	hipMalloc(&cylinders_d, sizeof(t_cylinder) * world.cylinders_len);
	hipMalloc(&cubes_d, sizeof(t_cube) * world.cubes_len);
	hipMalloc(&h_cubes_d, sizeof(t_h_cube) * world.h_cubes_len);
	hipMalloc(&triangles_d, sizeof(t_triangle) * world.triangles_len);
	hipMalloc(&torus_d, sizeof(t_torus) * world.torus_len);
	hipMalloc(&mobius_d, sizeof(t_mobius) * world.mobius_len);
	hipMalloc(&paraboloids_d, sizeof(t_paraboloid) * world.paraboloids_len);
	hipMalloc(&hyperboloids_d, sizeof(t_hyperboloid) * world.hyperboloids_len);
	hipMalloc(&cones_d, sizeof(t_cone) * world.cones_len);
	hipMalloc(&disks_d, sizeof(t_disk) * world.disks_len);
	hipMalloc(&lights_d, sizeof(t_light) * world.lights_len);

	hipMemcpy(spheres_d, world.spheres, sizeof(t_sphere) * world.spheres_len, hipMemcpyHostToDevice);
	hipMemcpy(planes_d, world.planes, sizeof(t_plane) * world.planes_len, hipMemcpyHostToDevice);
	hipMemcpy(cylinders_d, world.cylinders, sizeof(t_cylinder) * world.cylinders_len, hipMemcpyHostToDevice);
	hipMemcpy(cubes_d, world.cubes, sizeof(t_cube) * world.cubes_len, hipMemcpyHostToDevice);
	hipMemcpy(h_cubes_d, world.h_cubes, sizeof(t_h_cube) * world.h_cubes_len, hipMemcpyHostToDevice);
	hipMemcpy(triangles_d, world.triangles, sizeof(t_triangle) * world.triangles_len, hipMemcpyHostToDevice);
	hipMemcpy(torus_d, world.torus, sizeof(t_torus) * world.torus_len, hipMemcpyHostToDevice);
	hipMemcpy(mobius_d, world.mobius, sizeof(t_mobius) * world.mobius_len, hipMemcpyHostToDevice);
	hipMemcpy(paraboloids_d, world.paraboloids, sizeof(t_paraboloid) * world.paraboloids_len, hipMemcpyHostToDevice);
	hipMemcpy(hyperboloids_d, world.hyperboloids, sizeof(t_hyperboloid) * world.hyperboloids_len, hipMemcpyHostToDevice);
	hipMemcpy(cones_d, world.cones, sizeof(t_cone) * world.cones_len, hipMemcpyHostToDevice);
	hipMemcpy(disks_d, world.disks, sizeof(t_disk) * world.disks_len, hipMemcpyHostToDevice);
	hipMemcpy(lights_d, world.lights, sizeof(t_light) * world.lights_len, hipMemcpyHostToDevice);

	world.planes = planes_d;
	world.spheres = spheres_d;
	world.cylinders = cylinders_d;
	world.torus = torus_d;
	world.mobius = mobius_d;
	world.cubes = cubes_d;
	world.h_cubes = h_cubes_d;
	world.triangles = triangles_d;
	world.cones = cones_d;
	world.hyperboloids = hyperboloids_d;
	world.paraboloids = paraboloids_d;
	world.disks = disks_d;
	world.lights = lights_d;

	test <<< grid_size, threads_per_block >>> (a_d, constw, consth, world);
	printf("test2\n");
	hipMemcpy(a_h, a_d, size, hipMemcpyDeviceToHost);

	if (spheres_d != NULL)
		hipFree(spheres_d);
	if (planes_d != NULL)
		hipFree(planes_d);
	if (cubes_d != NULL)
		hipFree(cubes_d);
	if (h_cubes_d != NULL)
		hipFree(h_cubes_d);
	if (cylinders_d != NULL)
		hipFree(cylinders_d);
	if (cones_d != NULL)
		hipFree(cones_d);
	if (paraboloids_d != NULL)
		hipFree(paraboloids_d);
	if (hyperboloids_d != NULL)
		hipFree(hyperboloids_d);
	if (triangles_d != NULL)
		hipFree(triangles_d);
	if (disks_d != NULL)
		hipFree(disks_d);
	if (torus_d != NULL)
		hipFree(torus_d);
	if (mobius_d != NULL)
		hipFree(mobius_d);
	if (lights_d != NULL)
		hipFree(lights_d);
	if (a_d != NULL)
		hipFree(a_d);
	if (p != NULL)
		hipFree(p);
	printf("test3\n");
}
