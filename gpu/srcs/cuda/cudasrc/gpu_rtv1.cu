#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <float.h>

extern "C" {
#include "../../../includes/display.h"
#include "../../../includes/vectors.h"
#include "../../../includes/rt.h"
#include "../cudaheader/gpu_rt.h"
}


#define CUDA_ERROR_CHECK
#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
#define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )

inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
	if ( hipSuccess != err )
	{
		fprintf( stderr, "cudaSafeCall() failed at %s:%i : %s\n",
				file, line, hipGetErrorString( err ) );
		exit(-1);
	}
#endif
	return;
}

inline void __cudaCheckError( const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
	hipError_t err = hipGetLastError();
	if (hipSuccess != err)
	{
		fprintf( stderr, "cudaCheckError() failed at %s:%i : %s\n",
				file, line, hipGetErrorString( err ) );
		exit( -1 );
	}
#endif
	return;
}

__global__ void test(int *a, unsigned int constw, unsigned int consth, t_world world)
{
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int index = row * constw + col;
	a[index] = ray_tracer(world, col, row + world.offsets.y_min);
}

extern "C" void render_cuda(int *a_h, unsigned int constw, unsigned int consth, t_world world, int reset)
{
	int 		*a_d = 0;
	t_sphere	*spheres_d = NULL;
	t_plane		*planes_d = NULL;
	//	static t_cylinder	*cylinders_d;
	t_cone		*cones_d = NULL;
	t_light		*lights_d = NULL;
	//	static t_paraboloid	*paraboloids_d;
	size_t		size = 0;
	dim3		threads_per_block(32, 32);
	dim3		grid_size(constw / threads_per_block.x, consth / threads_per_block.y);

	size = constw * consth * sizeof(int);
	CudaSafeCall(hipMalloc(&a_d, size));
	CudaSafeCall(hipMalloc(&spheres_d, sizeof(t_sphere) * world.spheres_len));
	CudaSafeCall(hipMalloc(&planes_d, sizeof(t_plane) * world.planes_len));
	//	CudaSafeCall(hipMalloc(&cylinders_d, sizeof(t_cylinder) * world.cylinders_len));
	CudaSafeCall(hipMalloc(&cones_d, sizeof(t_cone) * world.cones_len));
	CudaSafeCall(hipMalloc(&lights_d, sizeof(t_light) * world.lights_len));
	//	CudaSafeCall(hipMalloc(&paraboloids_d, sizeof(t_paraboloid) * world.paraboloids_len));

	CudaSafeCall(hipMemcpy(spheres_d, world.spheres, sizeof(t_sphere) * world.spheres_len, hipMemcpyHostToDevice));
	world.spheres = spheres_d;
	CudaSafeCall(hipMemcpy(planes_d, world.planes, sizeof(t_plane) * world.planes_len, hipMemcpyHostToDevice));
	world.planes = planes_d;
		// CudaSafeCall(hipMemcpy(cylinders_d, world.cylinders, sizeof(t_cylinder) * world.cylinders_len, hipMemcpyHostToDevice));
		// world.cylinders = cylinders_d;
	CudaSafeCall(hipMemcpy(cones_d, world.cones, sizeof(t_cone) * world.cones_len, hipMemcpyHostToDevice));
	world.cones = cones_d;

	CudaSafeCall(hipMemcpy(lights_d, world.lights, sizeof(t_light) * world.lights_len, hipMemcpyHostToDevice));
	world.lights = lights_d;
		// CudaSafeCall(hipMemcpy(paraboloids_d, world.paraboloids, sizeof(t_paraboloid) * world.paraboloids_len, hipMemcpyHostToDevice));
		// world.paraboloids = paraboloids_d;
	test <<< grid_size, threads_per_block>>> (a_d, constw, consth, world);
			CudaCheckError();
	if(spheres_d != NULL)
		CudaSafeCall(hipFree(spheres_d));
	if(planes_d != NULL)
		CudaSafeCall(hipFree(planes_d));
	if(cones_d != NULL)
		CudaSafeCall(hipFree(cones_d));
	if(lights_d != NULL)
		CudaSafeCall(hipFree(lights_d));
	CudaSafeCall(hipMemcpy(a_h, a_d, size, hipMemcpyDeviceToHost));
	if(a_d != NULL)
		CudaSafeCall(hipFree(a_d));
}
