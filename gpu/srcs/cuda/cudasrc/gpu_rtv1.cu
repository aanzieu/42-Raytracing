#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <float.h>
extern "C" {
	#include "../../../includes/display.h"
	#include "../../../includes/vectors.h"
	#include <rt.h>
	#include "gpu_rt.h"
}

#include "gpu_rt.h"

#define CUDA_ERROR_CHECK

#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )

#define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )

inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaSafeCall() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
#endif

    return;
}

inline void __cudaCheckError( const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    hipError_t err = hipGetLastError();
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaCheckError() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }

    // // More careful checking. However, this will affect performance.
    // // Comment away if needed.
    // err = hipDeviceSynchronize();
    // if( hipSuccess != err )
    // {
    //     fprintf( stderr, "cudaCheckError() with sync failed at %s:%i : %s\n",
    //              file, line, hipGetErrorString( err ) );
    // //     exit( -1 );
    // }
#endif

    return;
}

__global__ void test(int *a, unsigned int constw, unsigned int consth, t_world world)
{	
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int index = row * constw + col;
	a[index] = ray_tracer(world, col, row);
}

extern "C" void render_cuda(int *a_h, unsigned int constw, unsigned int consth, t_world world, int reset)
{
	int				i;
	int 			j;
	static int 		*a_d;
	int 			y;
	int				x;
	static t_sphere	*spheres_d;
	static t_plane		*planes_d;
	static t_cylinder	*cylinders_d;
	static t_cone		*cones_d;
	static t_light		*lights_d;
	static t_paraboloid	*paraboloids_d;
	static size_t	size = 0;
	dim3		threads_per_block(32, 32);
	dim3		grid_size(constw / threads_per_block.x, consth / threads_per_block.y);


	if (reset == 0)
	{
		if (size == 0)
		{
			size = constw * consth * sizeof(int);
			CudaSafeCall(hipMalloc(&a_d, size));
			CudaSafeCall(hipMalloc(&spheres_d, sizeof(t_sphere) * world.spheres_len));
			CudaSafeCall(hipMalloc(&planes_d, sizeof(t_plane) * world.planes_len));
			CudaSafeCall(hipMalloc(&cylinders_d, sizeof(t_cylinder) * world.cylinders_len));
			CudaSafeCall(hipMalloc(&cones_d, sizeof(t_cone) * world.cones_len));
			CudaSafeCall(hipMalloc(&lights_d, sizeof(t_light) * world.lights_len));
			CudaSafeCall(hipMalloc(&paraboloids_d, sizeof(t_paraboloid) * world.paraboloids_len));
		}
		CudaSafeCall(hipMemcpy(spheres_d, world.spheres, sizeof(t_sphere) * world.spheres_len, hipMemcpyHostToDevice));
		world.spheres = spheres_d;
		CudaSafeCall(hipMemcpy(planes_d, world.planes, sizeof(t_plane) * world.planes_len, hipMemcpyHostToDevice));
		world.planes = planes_d;
		CudaSafeCall(hipMemcpy(cylinders_d, world.cylinders, sizeof(t_cylinder) * world.cylinders_len, hipMemcpyHostToDevice));
		world.cylinders = cylinders_d;
		CudaSafeCall(hipMemcpy(cones_d, world.cones, sizeof(t_cone) * world.cones_len, hipMemcpyHostToDevice));
		world.cones = cones_d;
		CudaSafeCall(hipMemcpy(lights_d, world.lights, sizeof(t_light) * world.lights_len, hipMemcpyHostToDevice));
		world.lights = lights_d;
		CudaSafeCall(hipMemcpy(paraboloids_d, world.paraboloids, sizeof(t_paraboloid) * world.paraboloids_len, hipMemcpyHostToDevice));
		world.paraboloids = paraboloids_d;
		test <<< grid_size, threads_per_block>>> (a_d, constw, consth, world);
		// printf("Frame rendered\n");
		CudaCheckError();
		CudaSafeCall(hipMemcpy(a_h, a_d, size, hipMemcpyDeviceToHost));
		i = 0;
		y = 0;
		while (i < WIN_HEIGHT)
		{
			j = 0;
			x = 0;
			while (j < WIN_WIDTH)
			{
				pixel_to_image(world.window.screen, j, i, a_h[y * constw + x]);
				j++;
				if (j % world.render_factor == 0)
					x++;
			}
			i++;
			if (i % world.render_factor == 0)
				y++;
		}
	}
	else
	{
		hipFree(spheres_d);
		hipFree(planes_d);
		hipFree(cylinders_d);
		hipFree(cones_d);
		hipFree(lights_d);
		hipFree(paraboloids_d);
		hipFree(a_d);
	}
}
