/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   ray_tracer.c                                       :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: svilau <marvin@42.fr>                      +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2016/10/20 10:49:50 by svilau            #+#    #+#             */
/*     Updated: 2017/06/29 14:26:47 by huweber          ###   ########.fr     */
/*                                                                            */
/* ************************************************************************** */

extern "C" {
	#include <rt.h>
	#include "gpu_rt.h"
	#include "object.h"
	#include "vectors.h"
}
#include "../../../frameworks/SDL2.framework/Headers/SDL.h"
#include "../../../includes/display.h"
#include <math.h>
#include <float.h>
#include <hip/hip_runtime.h>

__host__ __device__ void	get_ray_direction(t_world world, t_ray *ray,
	int x, int y)
{
	t_vec3d dir_point;

	dir_point = vector_substract(
								vector_add(world.viewplane_aa.up_left,
									vector_scalar(world.camera.right_v,
										world.viewplane_aa.x_indent * x)),
										vector_scalar(world.camera.up_v,
											world.viewplane_aa.y_indent * y));
	ray->dir_point = dir_point;
	ray->dir = vector_calculate(world.camera.pos, dir_point);
	ray->origin = world.camera.pos;
}

/*
** Get the top-left point in virtual space of the viewplane
*/

__host__ __device__ void	get_up_left(t_world *world)
{
	world->viewplane.up_left = vector_substract(
									vector_add(
										vector_add(
											world->camera.pos,
											world->camera.dir_v),
										vector_scalar(
											world->camera.up_v,
											world->viewplane.height / 2.0f)),
									vector_scalar(
										world->camera.right_v,
										world->viewplane.width / 2.0f));
	world->viewplane_aa.up_left = vector_substract(
								vector_add(
									vector_add(
										world->camera.pos,
										world->camera.dir_v),
									vector_scalar(
										world->camera.up_v,
										world->viewplane_aa.height / 2.0f)),
								vector_scalar(
									world->camera.right_v,
									world->viewplane_aa.width / 2.0f));
}

__host__ __device__ double		get_closest_intersection(t_world world, t_ray ray,
										t_intersection *intersection)
{
	t_intersection	intersection_tmp;

	intersection_tmp.t = DBL_MAX;
	intersection_tmp.type = '0';
	intersection_tmp.id = intersection->id;
	get_closest_sphere(world, ray, intersection, &intersection_tmp);
	get_closest_mobius(world, ray, intersection, &intersection_tmp);
	get_closest_plane(world, ray, intersection, &intersection_tmp);
	get_closest_disk(world, ray, intersection, &intersection_tmp);
	get_closest_cone(world, ray, intersection, &intersection_tmp);
	get_closest_cube(world, ray, intersection, &intersection_tmp);
	get_closest_torus(world, ray, intersection, &intersection_tmp);
	get_closest_cylinder(world, ray, intersection, &intersection_tmp);
	get_closest_paraboloid(world, ray, intersection, &intersection_tmp);
	get_closest_hyperboloid(world, ray, intersection, &intersection_tmp);
	get_closest_triangle(world, ray, intersection, &intersection_tmp);
	if (intersection->type == '0')
		return (0);
	else
		return (1);
}

__host__ __device__ void	cartoon_effect(t_world world, t_color *color,
	t_light lights, t_intersection intersection, t_ray ray)
{
	if(world.keys.select == 1 && intersection.id_save == world.id_save)
	{
		if(vector_dot(intersection.normal_v, ray.dir) > -0.3 && intersection.type != 'p'
			&& vector_dot(intersection.normal_v, ray.dir) < 0.0000001)
		{
			color->r = 1;
			color->g = 1;
			color->b = 1;
		}
	}
	else if (world.keys.pad_0 == 6)
	{
		if(vector_dot(intersection.normal_v, ray.dir) > -0.3 && intersection.type != 'p'
			&& vector_dot(intersection.normal_v, ray.dir) < 0.0000001)
		{
			color->r = 0;
			color->g = 0;
			color->b = 0;
		}
	}
}

__host__ __device__ t_color	direct_light(t_world world, t_color color,
	t_light light, t_ray ray)
{
	t_vec3d	light_vector;
	t_color	direct_light;
	double 	angle;

	direct_light = new_color(10, 10, 10);
	light_vector = light.dir_v;
	angle = vector_dot(vector_scalar(world.camera.dir_v, -1), light_vector);
	direct_light = color_scalar(direct_light, angle);
	return (color_add(color, direct_light));
}

__host__ __device__ t_color apply_materials(t_world world, t_ray ray,
	t_intersection intersection)
{
	t_color color;
//	double	clamp;

	color = (t_color){0, 0, 0};
	if (intersection.transparence_coef > 0)
		color = color_add(color_scalar(color, 1 - intersection.transparence_coef),
		color_scalar(handle_transparence(world, ray, &intersection),
		intersection.transparence_coef));
	if (intersection.reflection_coef > 0)
		color = color_add(color_scalar(color, 1 - intersection.reflection_coef),
		color_scalar(handle_reflection(world, ray, &intersection),
		intersection.reflection_coef));
	if (intersection.refraction_coef > 0)
		color = color_add(color, handle_refraction(world, ray, &intersection));
	if (intersection.chess.r >= 0)
		color = color_add(color, handle_chess(ray, &intersection));
	// if (i > 1)
	// 	color = (t_color){color.r / i, color.g / i, color.b / i};
	return (color);
}

__host__ __device__ t_color		ray_tracer_depth(t_world world, t_ray ray,
	t_intersection intersection)
{
	t_color					color;
	int							i;

	i = 0;
	color = new_color(0, 0, 0);
	if (intersection.depth > MAX_DEPTH)
		return (color);
	get_closest_intersection(world, ray, &intersection);

	if (intersection.type == '0')
		return ((t_color){0, 0, 0});
//		return (color_scalar((t_color){66, 173, 212}, 0.3f / (y + 0.0001f)));
//	if(world.keys.light_none == 1)
//		color = apply_materials(world, ray, intersection);

	color = color_multiply(color, world.ambient.color);
	color = color_scalar(color, world.ambient.intensity);
	if(world.keys.light_none == 0)
	{
		color = intersection.color;
	//	color = color_scalar(color, 1);
	}
	if(world.keys.select == 1)
		cartoon_effect(world, &color, world.lights[i], intersection, ray);
	while (i < world.lights_len && world.keys.light_none == 1)
	{
		color = get_light_at(world, color, world.lights[i], intersection, ray);
		if (world.keys.pad_0 == 6)
			cartoon_effect(world, &color, world.lights[i], intersection, ray);
		i++;
	}
	return (color);
}

__host__ __device__ void 	new_intersection(t_intersection *intersection)
{
	intersection->t = DBL_MAX;
	intersection->type = '0';
	intersection->id = -1;
	intersection->depth = 0;
	intersection->color = (t_color){-1, -1, -1};
//	intersection->color = (t_color){1, 1, 1};

	intersection->chess = (t_color){-1, -1, -1};
	intersection->refraction_coef = 0;
	intersection->reflection_coef = 0;
	intersection->transparence_coef = 0;
}


__host__ __device__ int		ray_tracer(t_world world, int x, int y)
{
	t_ray      			ray;
	t_color					color = (t_color){0, 0, 0};
	t_intersection	intersection; 
	
	
	//t_world->aa = 4;
	
	int 						i = 0;
	int							j = 0;
	int							x_aa = 0;
	int							y_aa = 0;

	while (i < world.aa)
	{
		j = 0;
		y_aa = y + i;
		while (j < world.aa)
		{
			x_aa = x + j;
			new_intersection(&intersection);
			get_up_left(&world);
			get_ray_direction(world, &ray, x_aa, y_aa);
			color = color_add(color, ray_tracer_depth(world, ray, intersection));
			j++;
		}
		i++;
	}
	color = color_divide(color, world.aa * world.aa);
	return (get_color(color));
	//return(RED);
}
