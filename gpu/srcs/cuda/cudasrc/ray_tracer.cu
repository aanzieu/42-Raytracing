/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   ray_tracer.c                                       :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: svilau <marvin@42.fr>                      +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2016/10/20 10:49:50 by svilau            #+#    #+#             */
/*     Updated: 2017/06/29 14:26:47 by huweber          ###   ########.fr     */
/*                                                                            */
/* ************************************************************************** */

extern "C" {
	#include <rt.h>
	#include "gpu_rt.h"
	#include "object.h"
	#include "vectors.h"
}
#include "../../../frameworks/SDL2.framework/Headers/SDL.h"
#include "../../../includes/display.h"
#include <math.h>
#include <float.h>
#include <hip/hip_runtime.h>

__host__ __device__ void	get_ray_direction(t_world world, t_ray *ray, int x, int y)
{
	t_vec3d dir_point;

	dir_point = vector_substract(
								vector_add(world.viewplane.up_left, vector_scalar(world.camera.right_v, world.viewplane.x_indent * x)),
								vector_scalar(world.camera.up_v, world.viewplane.y_indent * y));
	ray->dir_point = dir_point;
	ray->dir = vector_calculate(world.camera.pos, dir_point);
	ray->origin = world.camera.pos;
}

/*
** Get the top-left point in virtual space of the viewplane
*/

__host__ __device__ void	get_up_left(t_world *world)
{
	world->viewplane.up_left = vector_substract(
									vector_add(
										vector_add(
											world->camera.pos,
											world->camera.dir_v),
										vector_scalar(
											world->camera.up_v,
											world->viewplane.height / 2.0f)),
									vector_scalar(
										world->camera.right_v,
										world->viewplane.width / 2.0f));
}

__host__ __device__ double		get_closest_intersection(t_world world, t_ray ray,
										t_intersection *intersection)
{
	t_intersection	intersection_tmp;

	intersection_tmp.t = DBL_MAX;
	intersection_tmp.type = '0';
	intersection_tmp.id = intersection->id;
	get_closest_sphere(world, ray, intersection, &intersection_tmp);
	get_closest_plane(world, ray, intersection, &intersection_tmp);
	get_closest_disk(world, ray, intersection, &intersection_tmp);
	get_closest_cone(world, ray, intersection, &intersection_tmp);
	get_closest_cylinder(world, ray, intersection, &intersection_tmp);
	get_closest_paraboloid(world, ray, intersection, &intersection_tmp);
	get_closest_hyperboloid(world, ray, intersection, &intersection_tmp);
	if (intersection->type == '0')
		return (0);
	else
		return (1);
}

__host__ __device__ void	cartoon_effect(t_world world, t_color *color, t_light lights, t_intersection intersection, t_ray ray)
{
	if(vector_dot(intersection.normal_v, ray.dir) > -0.3 && intersection.type != 'p'
			&& vector_dot(intersection.normal_v, ray.dir) < 0.0000001)
	{
		color->r = 0;
		color->g = 0;
		color->b = 0;
	}
}

__host__ __device__ void	direct_light(t_world world, t_color *color, t_light light, t_ray ray)
{
	t_vec3d	light_vector;
	t_color	direct_light;
	double 	angle;

	direct_light = new_color(10, 10, 10);
	light_vector = light.dir_v;
	angle = vector_dot(vector_scalar(world.camera.dir_v, -1), light_vector);
	color_scalar(&direct_light, angle);
	color_add(color, direct_light);
}

__host__ __device__ t_color 	handle_reflection(t_world world, t_ray ray,
															t_intersection *intersection)
{
	int					i;
	int					depth;
  t_color			color;

  i = 0;
  color = new_color(0, 0, 0);
  depth = intersection->reflection_coef < MAX_DEPTH ?
		intersection->reflection_coef : MAX_DEPTH;
  while (i < depth)
  {
    intersection->t = DBL_MAX;
    intersection->type = '0';
    ray.origin = intersection->pos;
    ray.dir = vector_normalize(vector_substract(ray.dir,
            vector_scalar(intersection->normal_v,
                    2 * vector_dot(ray.dir, intersection->normal_v))));
    get_closest_intersection(world, ray, intersection);
    if (intersection->type != '0')
      color = *intersection->color;
    else
    {
      color = new_color(0, 0, 0);
      intersection->color = &color;
      return (color);
    }
    i++;
  }
  return (color);
}

__host__ __device__ int		ray_tracer(t_world world, int x, int y)
{
	t_ray      			ray;
	t_intersection	intersection;
	t_color					color;
	int							i;

	i = 0;
	color = new_color(0, 0, 0);
	intersection.t = DBL_MAX;
	intersection.type = '0';
	intersection.id = -1;

	get_up_left(&world);
	get_ray_direction(world, &ray, x, y);
	get_closest_intersection(world, ray, &intersection);

	if (intersection.type == '0')
		return (0);
	if(world.keys.light_none == 1){
	if (intersection.reflection_coef == 0 && intersection.refraction_coef == 0)
		color = *intersection.color;
	if (intersection.reflection_coef != 0)
		color_add(&color, handle_reflection(world, ray, &intersection));
	if (intersection.refraction_coef != 0)
		color_add(&color, handle_refraction_transparence(world, ray, &intersection));
	if (intersection.chess->r != -1)
		color_add(&color, chess_effect(&intersection));
	}
	color_multiply(&color, world.ambient.color);
	color_scalar(&color, world.ambient.intensity);
	if (world.keys.light_none == 0)
		color = *intersection.color;
	while (i < world.lights_len && world.keys.light_none == 1)
	{
		get_light_at(world, &color, world.lights[i], intersection, ray);
		if (world.keys.pad_9 == 1)
			cartoon_effect(world, &color, world.lights[i], intersection, ray);
		i++;
	}
	return (get_color(color));
}
