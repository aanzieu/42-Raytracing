/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   ray_tracer.c                                       :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: svilau <marvin@42.fr>                      +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2016/10/20 10:49:50 by svilau            #+#    #+#             */
/*     Updated: 2017/06/29 14:26:47 by huweber          ###   ########.fr     */
/*                                                                            */
/* ************************************************************************** */

extern "C" {
	#include <rt.h>
	#include "gpu_rt.h"
	#include "object.h"
	#include "vectors.h"
}
#include "../../../frameworks/SDL2.framework/Headers/SDL.h"
#include "../../../includes/display.h"
#include <math.h>
#include <float.h>
#include <hip/hip_runtime.h>

__host__ __device__ t_intersection new_inter(void)
{
	t_intersection new_i;

	new_i.id = -1;
	new_i.t = DBL_MAX;
	new_i.type = '0';
	new_i.depth = 0;
	new_i.transparence_coef = -1;
	new_i.refraction_coef = -1;
	new_i.reflection_coef = -1;
	new_i.chess = (t_color){-1, -1, -1};
	new_i.color = (t_color){0, 0, 0};
	new_i.pos = (t_vec3d){0, 0, 0};
	new_i.normal_v = (t_vec3d){0, 0, 0};
	return (new_i);
}

__host__ __device__ void	get_ray_direction(t_world world, t_ray *ray,
	int x, int y)
{
	t_vec3d dir_point;

	dir_point = vector_substract(
								vector_add(world.viewplane_aa.up_left,
									vector_scalar(world.camera.right_v,
										world.viewplane_aa.x_indent * x)),
								vector_scalar(world.camera.up_v, world.viewplane_aa.y_indent * y));
	ray->dir_point = dir_point;
	ray->dir = vector_calculate(world.camera.pos, dir_point);
	ray->origin = world.camera.pos;
}

/*
** Get the top-left point in virtual space of the viewplane
*/

__host__ __device__ void	get_up_left(t_world *world)
{
	world->viewplane.up_left = vector_substract(
									vector_add(
										vector_add(
											world->camera.pos,
											world->camera.dir_v),
										vector_scalar(
											world->camera.up_v,
											world->viewplane.height / 2.0f)),
									vector_scalar(
										world->camera.right_v,
										world->viewplane.width / 2.0f));
	world->viewplane_aa.up_left = vector_substract(

									vector_add(
										vector_add(
											world->camera.pos,
											world->camera.dir_v),
										vector_scalar(
											world->camera.up_v,
											world->viewplane_aa.height / 2.0f)),
									vector_scalar(
										world->camera.right_v,
										world->viewplane_aa.width / 2.0f));
	// printf("%f %f %f\n", world->viewplane.up_left.x, world->viewplane.up_left.y, world->viewplane.up_left.z);
}

__host__ __device__ double		get_closest_intersection(t_world world, t_ray ray,
										t_intersection *intersection)
{
	t_intersection	intersection_tmp;

	intersection_tmp.t = DBL_MAX;
	intersection_tmp.type = '0';
	intersection_tmp.id = intersection->id;
	get_closest_sphere(world, ray, intersection, &intersection_tmp);
	get_closest_mobius(world, ray, intersection, &intersection_tmp);
	get_closest_torus(world, ray, intersection, &intersection_tmp);
	get_closest_plane(world, ray, intersection, &intersection_tmp);

	get_closest_disk(world, ray, intersection, &intersection_tmp);
	get_closest_cone(world, ray, intersection, &intersection_tmp);
	get_closest_cube(world, ray, intersection, &intersection_tmp);
	get_closest_cylinder(world, ray, intersection, &intersection_tmp);
	get_closest_paraboloid(world, ray, intersection, &intersection_tmp);
	get_closest_hyperboloid(world, ray, intersection, &intersection_tmp);
	get_closest_triangle(world, ray, intersection, &intersection_tmp);
	if (intersection->type == '0')
		return (0);
	return (1);
}

__host__ __device__ void	cartoon_effect(t_world world, t_color *color,
	t_light lights, t_intersection intersection, t_ray ray)
{
	if(world.keys.select == 1 && intersection.id_save == world.id_save)
	{
		if(vector_dot(intersection.normal_v, ray.dir) > -0.3 && intersection.type != 'p'
			&& vector_dot(intersection.normal_v, ray.dir) < 0.0000001)
		{
			color->r = 1;
			color->g = 1;
			color->b = 1;
		}
	}
	else if (world.keys.pad_0 == 6)
	{
		if(vector_dot(intersection.normal_v, ray.dir) > -0.3 && intersection.type != 'p'
			&& vector_dot(intersection.normal_v, ray.dir) < 0.0000001)
		{
			color->r = 0;
			color->g = 0;
			color->b = 0;
		}
	}
}

__host__ __device__ t_color	direct_light(t_world world, t_color color,
	t_light light, t_ray ray)
{
	t_vec3d	light_vector;
	t_color	direct_light;
	double 	angle;

	direct_light = new_color(10, 10, 10);
	light_vector = light.dir_v;
	angle = vector_dot(vector_scalar(world.camera.dir_v, -1), light_vector);
	direct_light = color_scalar(direct_light, angle);
	return (color_add(color, direct_light));
}

__host__ __device__ t_color apply_materials_cpu(t_world world, t_ray ray,
	t_intersection intersection, t_color color)
{
	if (intersection.transparence_coef > 0)
		color = color_add(color_scalar(color, 1 - intersection.transparence_coef),
		color_scalar(handle_transparence_cpu(world, ray, &intersection),
		intersection.transparence_coef));
	if (intersection.reflection_coef > 0)
		color = color_add(color_scalar(color, 1 - intersection.reflection_coef),
		color_scalar(handle_reflection_cpu(world, ray, &intersection),
		intersection.reflection_coef));
	if (intersection.refraction_coef > 0)
		color = color_add(color, handle_refraction_cpu(world, ray, &intersection));
	if (intersection.chess.r >= 0)
		color = handle_chess(ray, intersection);
	return (color);
}
__host__ __device__ double get_indice(t_intersection inter)
{
	if (inter.reflection_coef > 0)
		return (inter.reflection_coef);
	if (inter.transparence_coef > 0)
		return (inter.transparence_coef);
	return (0);
	// else if (intersection.refraction_coef > 0)
}

__host__ __device__ t_color apply_materials_gpu(t_world world, t_ray ray,
	t_intersection intersection, t_color color)
{
	t_color tmp = color;
	int			flag = 0, i = 0;

	double indice = get_indice(intersection);
	while (flag == 0 && intersection.depth < MAX_DEPTH)
	{
		flag = 1;
		if (intersection.transparence_coef > 0)
			tmp = handle_transparence_gpu(world, &ray, &intersection, &flag);//, intersection.transparence_coef));
		if (intersection.reflection_coef > 0)
			tmp = handle_reflection_gpu(world, &ray, &intersection, &flag);//, intersection.reflection_coef));
		if (intersection.refraction_coef > 0)
			tmp = handle_refraction_gpu(world, &ray, &intersection, &flag);
		if (intersection.chess.r >= 0)
				tmp = handle_chess(ray, intersection);
		if (flag == 0)
		{
			tmp = color_multiply(tmp, world.ambient.color);
			tmp = color_scalar(tmp, world.ambient.intensity);
			i = 0;
			while (i < world.lights_len)
			{
				tmp = get_light_at(world, tmp, world.lights[i], intersection, ray);
				i++;
			}
		}
	}
	if (indice > 0 && indice < 1)
		return (color_scalar(tmp, indice));
	else
		return (tmp);
}

__host__ __device__ t_color		ray_tracer_depth_cpu(t_world world, t_ray ray,
				t_intersection intersection)
{
	t_color					color;
	int							i;

	i = 0;
	color = new_color(0, 0, 0);
	if (intersection.depth > MAX_DEPTH)
		return (color);
	get_closest_intersection(world, ray, &intersection);
	if (intersection.type == '0')
		return ((t_color){0, 0, 0});
	if(world.keys.light_none == 0)
		color = intersection.color;
	if(world.keys.light_none == 1 && world.mode == 0)
		color = apply_materials_cpu(world, ray, intersection, color);
	color = color_multiply(color, world.ambient.color);
	color = color_scalar(color, world.ambient.intensity);
	if(world.keys.select == 1)
		cartoon_effect(world, &color, world.lights[i], intersection, ray);
	while (i < world.lights_len && world.keys.light_none == 1)
	{
		color = get_light_at(world, color, world.lights[i], intersection, ray);
		if (world.keys.pad_9 == 1)
			cartoon_effect(world, &color, world.lights[i], intersection, ray);
		i++;
	}
	return (color);
}

__host__ __device__ t_color		ray_tracer_depth_gpu(t_world world, t_ray ray,
				t_intersection intersection)
{
	t_color					color;
	int							i;

	i = 0;
	color = new_color(0, 0, 0);
	get_closest_intersection(world, ray, &intersection);
	if (intersection.type == '0')
		return ((t_color){0, 0, 0});
//		return (color_scalar((t_color){66, 173, 212}, 0.3f / (y + 0.0001f)));
		color = apply_materials_gpu(world, ray, intersection, color);

	// else
	// 	return (intersection.color);
	color = intersection.color;
	if(world.keys.light_none == 1)
		color = apply_materials_gpu(world, ray, intersection, color);
	color = color_multiply(color, world.ambient.color);
	color = color_scalar(color, world.ambient.intensity);
	if(world.keys.light_none == 0)
	{
		color = intersection.color;
	//	color = color_scalar(color, 1);
	}
	if(world.keys.select == 1)
		cartoon_effect(world, &color, world.lights[i], intersection, ray);
	while (i < world.lights_len && world.keys.light_none == 1)
	{
		color = get_light_at(world, color, world.lights[i], intersection, ray);
		if (world.keys.pad_0 == 6)
			cartoon_effect(world, &color, world.lights[i], intersection, ray);
		i++;
	}
	return (color);
}

__host__ __device__ int		ray_tracer_cpu(t_world world, int x, int y)
{
	t_ray      			ray;
	t_color					color = (t_color){0, 0, 0};
	t_intersection	intersection; //t_world->aa = 4;
	int i = 0, j = 0, xx = 0, yy = 0;

	while (i < world.aa)
	{
		j = 0;
		yy = y + i;
		while (j < world.aa)
		{
			xx = x + j;
			intersection = new_inter();
			ray.dir = (t_vec3d){0, 0, 0};
			ray.origin = (t_vec3d){0, 0, 0};
			get_up_left(&world);
			get_ray_direction(world, &ray, xx, yy);
			color = color_add(color, ray_tracer_depth_cpu(world, ray, intersection));
			j++;
		}
		i++;
	}
	color = color_divide(color, world.aa * world.aa);
	return (get_color(color));
}

__host__ __device__ int		ray_tracer_gpu(t_world world, int x, int y)
{
	t_ray      			ray;
	t_color					color = (t_color){0, 0, 0};
	t_intersection	intersection; //t_world->aa = 4;
	int i = 0, j = 0, xx = 0, yy = 0;

	while (i < world.aa)
	{
		j = 0;
		yy = y + i;
		while (j < world.aa)
		{
			xx = x + j;
			intersection = new_inter();
			ray.dir = (t_vec3d){0, 0, 0};
			ray.origin = (t_vec3d){0, 0, 0};
			get_up_left(&world);
			get_ray_direction(world, &ray, xx, yy);
			color = color_add(color, ray_tracer_depth_gpu(world, ray, intersection));
			j++;
		}
		i++;
	}
	color = color_divide(color, world.aa * world.aa);
	return (get_color(color));
	//return(RED);
}
