#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   ray_tracer.c                                       :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: svilau <marvin@42.fr>                      +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2016/10/20 10:49:50 by svilau            #+#    #+#             */
/*     Updated: 2017/06/29 14:26:47 by huweber          ###   ########.fr     */
/*                                                                            */
/* ************************************************************************** */

extern "C" {
	#include <rt.h>
	#include "gpu_rt.h"
	#include "object.h"
	#include "vectors.h"
}
#include "../../../frameworks/SDL2.framework/Headers/SDL.h"
#include "../../../includes/display.h"
#include <math.h>
#include <float.h>
#include <hip/hip_runtime.h>

__host__ __device__ void	get_ray_direction(t_world world, t_ray *ray, int x, int y)
{
	t_vec3d dir_point;

	dir_point = vector_substract(
								vector_add(world.viewplane.up_left, vector_scalar(world.camera.right_v, world.viewplane.x_indent * x)),
								vector_scalar(world.camera.up_v, world.viewplane.y_indent * y));
	ray->dir_point = dir_point;
	ray->dir = vector_calculate(world.camera.pos, dir_point);
	ray->origin = world.camera.pos;
}

/*
** Get the top-left point in virtual space of the viewplane
*/

__host__ __device__ void	get_up_left(t_world *world)
{
	world->viewplane.up_left = vector_substract(
									vector_add(
										vector_add(
											world->camera.pos,
											world->camera.dir_v),
										vector_scalar(
											world->camera.up_v,
											world->viewplane.height / 2.0f)),
									vector_scalar(
										world->camera.right_v,
										world->viewplane.width / 2.0f));
}

__host__ __device__ double		get_closest_intersection(t_world world, t_ray ray,
										t_intersection *intersection)
{
	t_intersection	intersection_tmp;

	intersection_tmp.t = DBL_MAX;
	intersection_tmp.type = '0';
	intersection_tmp.id = intersection->id;
	get_closest_sphere(world, ray, intersection, &intersection_tmp);
	get_closest_plane(world, ray, intersection, &intersection_tmp);
	get_closest_disk(world, ray, intersection, &intersection_tmp);
	get_closest_cone(world, ray, intersection, &intersection_tmp);
	get_closest_cylinder(world, ray, intersection, &intersection_tmp);
	get_closest_paraboloid(world, ray, intersection, &intersection_tmp);
	get_closest_hyperboloid(world, ray, intersection, &intersection_tmp);
	if (intersection->type == '0')
		return (0);
	else
		return (1);
}

__host__ __device__ void	cartoon_effect(t_world world, t_color *color, t_light lights, t_intersection intersection, t_ray ray)
{
	if(vector_dot(intersection.normal_v, ray.dir) > -0.3 && intersection.type != 'p'
			&& vector_dot(intersection.normal_v, ray.dir) < 0.0000001)
	{
		color->r = 0;
		color->g = 0;
		color->b = 0;
	}
}

__host__ __device__ t_color 	handle_reflexion(t_world world, t_ray *ray, t_intersection *intersection, t_color color)
{
	int				i;
	int				depth;

	i = 0;
	depth = intersection->reflexion_coef < MAX_DEPTH ? intersection->reflexion_coef : MAX_DEPTH;
	while(i < depth)// && i < MAX_DEPTH)
	{
		intersection->t = DBL_MAX;
		ray->origin = intersection->pos;
		ray->dir = vector_normalize(vector_substract(ray->dir,
			vector_scalar(intersection->normal_v,
				2 * vector_dot(ray->dir, intersection->normal_v))));
		get_closest_intersection(world, *ray, intersection);
		if (intersection->type != '0')
			color_add(&color, *intersection->color);
		i++;
	}
	return (color);
}

__host__ __device__ t_color 	handle_refraxion_transparence(t_world world, t_ray *ray, t_intersection *intersection)
{
	double			calc[2];
	double		 	refracted_t;
	double			ref_coef;
	t_color			color;

	color = *intersection->color;
	intersection->t = DBL_MAX;
	intersection->type = '0';
 	ref_coef = intersection->refraxion_coef;
	if (ref_coef != 1)
	{
		calc[0] = vector_dot(intersection->normal_v, ray->dir);
		calc[1] = sqrt(1 - (ref_coef * ref_coef) * (1 - (calc[0] * calc[0])));
		if (calc[0] > 0)
			refracted_t = (ref_coef * calc[0] - calc[1]);
		else
			refracted_t = (ref_coef * calc[0] + calc[1]);
	 	ray->dir = vector_normalize(vector_add(
			vector_scalar(ray->dir, ref_coef),
			vector_scalar(intersection->normal_v, refracted_t)));
	}
	ray->origin = intersection->pos;
	if (get_closest_intersection(world, *ray, intersection))
	{
		color_scalar(&color, 0.33);
		color_add(&color, *intersection->color);
	}
	else
		color = (t_color){0, 0, 0};
	intersection->color = &color;
	return (color);
}

__host__ __device__ int		ray_tracer(t_world world, int x, int y)
{
	t_ray      			ray, raycpy;
	t_intersection	intersection;
	t_color					color;
	int							i;

	i = 0;
	color = (t_color){0, 0, 0};
	intersection.t = DBL_MAX;
	intersection.type = '0';
	intersection.id = -1;

	get_up_left(&world);
	get_ray_direction(world, &ray, x, y);
	raycpy.origin = vector_copy(ray.origin);
	raycpy.dir = vector_copy(ray.dir);
	get_closest_intersection(world, ray, &intersection);

	if (intersection.type == '0')
		return (0);
	if (intersection.reflexion_coef == 0 && intersection.refraxion_coef == 0)
		color = *intersection.color;
	if (intersection.reflexion_coef != 0)
		color = handle_reflexion(world, &raycpy, &intersection, color);
	if (intersection.refraxion_coef != 0)
		color = handle_refraxion_transparence(world, intersection.refraxion_coef == 1 ? &ray : &raycpy, &intersection);

	color_scalar(&color, world.ambient.intensity);
	color_multiply(&color, world.ambient.color);
	while (i < world.lights_len)
	{
		get_light_at(world, &color, world.lights[i], intersection, ray);
		if (world.keys.pad_9 == 1)
			cartoon_effect(world, &color, world.lights[i], intersection, ray);
		i++;
	}
	return (get_color(color));
}
