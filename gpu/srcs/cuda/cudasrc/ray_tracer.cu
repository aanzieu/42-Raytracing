/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   ray_tracer.c                                       :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: svilau <marvin@42.fr>                      +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2016/10/20 10:49:50 by svilau            #+#    #+#             */
/*     Updated: 2017/06/29 14:26:47 by huweber          ###   ########.fr     */
/*                                                                            */
/* ************************************************************************** */

extern "C" {
	#include <rt.h>
	#include "gpu_rt.h"
	#include "object.h"
	#include "vectors.h"
}
#include "../../../frameworks/SDL2.framework/Headers/SDL.h"
#include "../../../includes/display.h"
#include <math.h>
#include <float.h>
#include <hip/hip_runtime.h>

__host__ __device__ void	get_ray_direction(t_world world, t_ray *ray, int x, int y)
{
	t_vec3d dir_point;

	dir_point = vector_substract(
								vector_add(world.viewplane.up_left, vector_scalar(world.camera.right_v, world.viewplane.x_indent * x)),
								vector_scalar(world.camera.up_v, world.viewplane.y_indent * y));
	ray->dir_point = dir_point;
	ray->dir = vector_calculate(world.camera.pos, dir_point);
	ray->origin = world.camera.pos;
}

/*
** Get the top-left point in virtual space of the viewplane
*/

__host__ __device__ void	get_up_left(t_world *world)
{
	world->viewplane.up_left = vector_substract(
									vector_add(
										vector_add(
											world->camera.pos,
											world->camera.dir_v),
										vector_scalar(
											world->camera.up_v,
											world->viewplane.height / 2.0f)),
									vector_scalar(
										world->camera.right_v,
										world->viewplane.width / 2.0f));
}

__host__ __device__ double		get_closest_intersection(t_world world, t_ray ray,
										t_intersection *intersection)
{
	t_intersection	intersection_tmp;

	intersection_tmp.t = DBL_MAX;
	intersection_tmp.type = '0';

	get_closest_sphere(world, ray, intersection, &intersection_tmp);
	get_closest_plane(world, ray, intersection, &intersection_tmp);
	get_closest_disk(world, ray, intersection, &intersection_tmp);
	get_closest_cone(world, ray, intersection, &intersection_tmp);
	get_closest_cylinder(world, ray, intersection, &intersection_tmp);
	get_closest_paraboloid(world, ray, intersection, &intersection_tmp);
	get_closest_hyperboloid(world, ray, intersection, &intersection_tmp);
	if (intersection->type == '0')
		return (0);
	else
		return (1);
}

__host__ __device__ void	cartoon_effect(t_world world, t_color *color, t_light lights, t_intersection intersection, t_ray ray)
{
	if(vector_dot(intersection.normal_v, ray.dir) > -0.3 && intersection.type != 'p'
			&& vector_dot(intersection.normal_v, ray.dir) < 0.0000001)
	{
		color->r = 0;
		color->g = 0;
		color->b = 0;
	}
}

__host__ __device__ int		ray_tracer(t_world world, int x, int y)
{
	t_ray      			ray;//, ray_save;
	t_intersection	intersection;//, intersection_save;
	t_color					color = (t_color){0, 0, 0};
	t_vec3d					reflected;
	int							i, depth;

	i = 0;
	intersection.t = DBL_MAX;
	intersection.type = '0';
	get_up_left(&world);
	get_ray_direction(world, &ray, x, y);
	get_closest_intersection(world, ray, &intersection);
	if (intersection.type == '0')
		return (get_color(color));
	// intersection_save = intersection;
	// ray_save = ray;
	if (intersection.reflexion_coef == 0)
	{
		color_add(&color, *intersection.color);
	  color_multiply(&color, world.ambient.color);
	  color_scalar(&color, world.ambient.intensity);
	}
	else
	{
		depth = intersection.reflexion_coef;
		while(i < depth && i < MAX_DEPTH)
		{
			ray.origin = intersection.pos;//vector_add(intersection.pos, (t_vec3d){0.00000001, 0.00000001, 0.00000001});
			reflected = vector_scalar(intersection.normal_v, 2 * vector_dot(ray.dir, intersection.normal_v));
			ray.dir = vector_normalize(vector_substract(ray.dir, reflected));
			get_closest_intersection(world, ray, &intersection);
			if (intersection.type != '0')
				color_add(&color, *intersection.color);
			i++;
		}
		color_multiply(&color, world.ambient.color);
		color_scalar(&color, world.ambient.intensity);

	}
	i = 0;
	while (i < world.lights_len)
	{
			get_light_at(world, &color, world.lights[i], intersection, ray);
		if (world.keys.pad_9 == 1)
			cartoon_effect(world, &color, world.lights[i], intersection, ray);
		i++;
	}
	// (void)intersection_save;
	// (void)ray_save;
	return (get_color(color));
}
