#include "hip/hip_runtime.h"
/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   ray_tracer.c                                       :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: svilau <marvin@42.fr>                      +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2016/10/20 10:49:50 by svilau            #+#    #+#             */
/*     Updated: 2017/06/29 14:26:47 by huweber          ###   ########.fr     */
/*                                                                            */
/* ************************************************************************** */

extern "C" {
	#include <rt.h>
	#include "gpu_rt.h"
	#include "object.h"
	#include "vectors.h"
}
#include "../../../frameworks/SDL2.framework/Headers/SDL.h"
#include "../../../includes/display.h"
#include <math.h>
#include <float.h>
#include <hip/hip_runtime.h>

__host__ __device__ void	get_ray_direction(t_world world, t_ray *ray, int x, int y)
{
	t_vec3d dir_point;

	dir_point = vector_substract(
								vector_add(world.viewplane.up_left, vector_scalar(world.camera.right_v, world.viewplane.x_indent * x)),
								vector_scalar(world.camera.up_v, world.viewplane.y_indent * y));
	ray->dir_point = dir_point;
	ray->dir = vector_calculate(world.camera.pos, dir_point);
	ray->origin = world.camera.pos;
}

/*
** Get the top-left point in virtual space of the viewplane
*/

__host__ __device__ void	get_up_left(t_world *world)
{
	world->viewplane.up_left = vector_substract(
									vector_add(
										vector_add(
											world->camera.pos,
											world->camera.dir_v),
										vector_scalar(
											world->camera.up_v,
											world->viewplane.height / 2.0f)),
									vector_scalar(
										world->camera.right_v,
										world->viewplane.width / 2.0f));
}

__host__ __device__ double		get_closest_intersection(t_world world, t_ray ray,
										t_intersection *intersection)
{
	t_intersection	intersection_tmp;

	intersection_tmp.t = DBL_MAX;
	intersection_tmp.type = '0';
	intersection_tmp.id = intersection->id;
	get_closest_sphere(world, ray, intersection, &intersection_tmp);
	get_closest_plane(world, ray, intersection, &intersection_tmp);
	get_closest_disk(world, ray, intersection, &intersection_tmp);
	get_closest_cone(world, ray, intersection, &intersection_tmp);
	get_closest_cylinder(world, ray, intersection, &intersection_tmp);
	get_closest_paraboloid(world, ray, intersection, &intersection_tmp);
	get_closest_hyperboloid(world, ray, intersection, &intersection_tmp);
	if (intersection->type == '0')
		return (0);
	else
		return (1);
}

__host__ __device__ void	cartoon_effect(t_world world, t_color *color, t_light lights, t_intersection intersection, t_ray ray)
{
	if(vector_dot(intersection.normal_v, ray.dir) > -0.3 && intersection.type != 'p'
			&& vector_dot(intersection.normal_v, ray.dir) < 0.0000001)
	{
		color->r = 0;
		color->g = 0;
		color->b = 0;
	}
}

__host__ __device__ t_color 	handle_reflexion(t_world world, t_ray *ray, t_intersection *intersection)
{
	int				i;
	int				depth;
	t_color		color = (t_color){0, 0, 0};

	i = 0;
	int j = 0;
	depth = intersection->reflexion_coef < MAX_DEPTH ? intersection->reflexion_coef : MAX_DEPTH;
	while(i < depth && j < depth)
	{
		intersection->t = DBL_MAX;
		intersection->type = '0';
		ray->origin = intersection->pos;
		ray->dir = vector_normalize(vector_substract(ray->dir,
			vector_scalar(intersection->normal_v,
				2 * vector_dot(ray->dir, intersection->normal_v))));
		get_closest_intersection(world, *ray, intersection);
		if (intersection->type != '0')
		{
			color_add(&color, *intersection->color);
			if (intersection->reflexion_coef == 0)
				i++;
		}
		else
		{
			color = (t_color){0, 0, 0};
			*intersection->color = (t_color){0, 0, 0};
			return (color);
		}
		i++;
		j++;
	}
	return (color);
}

__host__ __device__ t_color 	handle_refraxion_transparence(t_world world, t_ray *ray, t_intersection *intersection)
{
	double			calc[2];
	double		 	refracted_t;
	double			ref_coef;
	t_color			color;

	intersection->t = DBL_MAX;
	intersection->type = '0';
	color = *intersection->color;
 	ref_coef = intersection->refraxion_coef;
	if (ref_coef > 1)
	{
		calc[0] = vector_dot(intersection->normal_v, ray->dir);
		calc[1] = sqrt(1 - (ref_coef * ref_coef) * (1 - (calc[0] * calc[0])));
		if (calc[0] > 0)
			refracted_t = (ref_coef * calc[0] - calc[1]);
		else
			refracted_t = (ref_coef * calc[0] + calc[1]);
	 	ray->dir = vector_normalize(vector_add(
			vector_scalar(ray->dir, ref_coef),
			vector_scalar(intersection->normal_v, refracted_t)));
	}
	ray->origin = intersection->pos;
	if (get_closest_intersection(world, *ray, intersection))
	{
		color = *intersection->color;
		// color_scalar(&color, ref_coef);
		// color_add(&color, *intersection->color);
	}
	else
		color = (t_color){0, 0, 0};
	intersection->color = &color;
	return (color);
}

__host__ __device__ t_color chess_effect(t_world world, t_ray *ray,
																					t_intersection *intersection)
{
	float			x;
	float			y;
	t_color		color;

  if (intersection->type == 'p')
	{
		if (((int)((intersection->pos.x + 450) * CHESS_PATTERN) ^
					(int)((intersection->pos.y + 450) * CHESS_PATTERN) ^
					(int)((intersection->pos.z + 450) * CHESS_PATTERN)) % 2 == 0)
		{
			color = *intersection->chess;
			intersection->color = intersection->chess;
		}
		else
			color = *intersection->color;
	}
	else
	{
		x = atan2(intersection->normal_v.z, intersection->normal_v.x) / M_PI + 1;
	  y = acos(intersection->normal_v.y) / M_PI;
		if ((fmodf(x * CHESS_PATTERN, 1) > 0.5) ^
				(fmodf(y * CHESS_PATTERN, 1) > 0.5) == 0)
		{
			color = *intersection->chess;
			intersection->color = intersection->chess;
		}
		else
			color = *intersection->color;
	}
	return (color);
}

__host__ __device__ int		ray_tracer(t_world world, int x, int y)
{
	t_ray      			ray, raycpy;
	t_intersection	intersection;
	t_color					color;
	int							i;

	i = 0;
	color = (t_color){0, 0, 0};
	intersection.t = DBL_MAX;
	intersection.type = '0';
	intersection.id = -1;

	get_up_left(&world);
	get_ray_direction(world, &ray, x, y);
	raycpy.origin = vector_copy(ray.origin);
	raycpy.dir = vector_copy(ray.dir);
	get_closest_intersection(world, ray, &intersection);
	if (intersection.type == '0')
		return (0);
	if (intersection.reflexion_coef == 0 && intersection.refraxion_coef == 0)
		color = *intersection.color;
	if (intersection.reflexion_coef != 0)
		color_add(&color, handle_reflexion(world, &raycpy, &intersection));
	if (intersection.refraxion_coef != 0)
		color_add(&color, handle_refraxion_transparence(world, &raycpy, &intersection));
	if (intersection.chess->r != -1)
		color_add(&color, chess_effect(world, &raycpy, &intersection));
	color_multiply(&color, world.ambient.color);
	color_scalar(&color, world.ambient.intensity);
	while (i < world.lights_len)
	{
		get_light_at(world, &color, world.lights[i], intersection, ray);
		if (world.keys.pad_9 == 1)
			cartoon_effect(world, &color, world.lights[i], intersection, ray);
		i++;
	}
	return (get_color(color));
}
