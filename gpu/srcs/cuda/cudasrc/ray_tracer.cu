/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   ray_tracer.c                                       :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: svilau <marvin@42.fr>                      +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2016/10/20 10:49:50 by svilau            #+#    #+#             */
/*   Updated: 2017/03/17 14:07:23 by svilau           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

extern "C" {
	#include "../../../includes/rtv1.h"
}
#include "../../../frameworks/SDL2.framework/Headers/SDL.h"
#include "../cudaheader/gpu_rt.cuh"
#include "../../../includes/display.h"
#include <math.h>
#include <float.h>
#include <hip/hip_runtime.h>

__host__ __device__ void	get_ray_direction(t_world world, t_ray *ray, int x, int y)
{
	t_vec3d dir_point;

	dir_point = vector_substract(
							vector_add(
								world.viewplane.up_left,
								vector_scalar(
									world.camera.right_v,
									world.viewplane.x_indent * x)),
							vector_scalar(
								world.camera.up_v,
								world.viewplane.y_indent * y));
	ray->dir = vector_normalize(
				vector_calculate(
					world.camera.pos,
					dir_point));
	ray->origin = world.camera.pos;
}

/*
** Get the top-left point in virtual space of the viewplane
*/

__host__ __device__ void	get_up_left(t_world *world)
{
	world->viewplane.up_left = vector_substract(
									vector_add(
										vector_add(
											world->camera.pos,
											world->camera.dir_v),
										vector_scalar(
											world->camera.up_v,
											world->viewplane.height / 2.0f)),
									vector_scalar(
										world->camera.right_v,
										world->viewplane.width / 2.0f));
}

__host__ __device__ double		get_closest_intersection(t_world world, t_ray ray,
										t_intersection *intersection)
{
	t_intersection	intersection_tmp;

	intersection_tmp.t = DBL_MAX;
	intersection_tmp.type = '0';
	
	get_closest_sphere(world, ray, intersection, &intersection_tmp);
	get_closest_plane(world, ray, intersection, &intersection_tmp);
	get_closest_cone(world, ray, intersection, &intersection_tmp);
	get_closest_cylinder(world, ray, intersection, &intersection_tmp);
	if (intersection_tmp.type == '0')
		return (0);
	else
		return (1);
}

__host__ __device__ int		ray_tracer(t_world world, int x, int y)
{
	t_ray			ray;
	t_intersection	intersection;

	intersection.t = DBL_MAX;
	intersection.type = '0';
	get_up_left(&world);
	get_ray_direction(world, &ray, x, y);
	get_closest_intersection(world, ray, &intersection);
	if (intersection.type != '0')
	{
		if (get_shadow(world, intersection, world.lights[0]) == 1)
			return (BLACK);
		else
			return (get_light_at(world.lights[0], intersection));
	}
		return (BLACK);
}
