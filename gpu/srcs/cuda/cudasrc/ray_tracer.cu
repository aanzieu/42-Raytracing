/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   ray_tracer.c                                       :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: svilau <marvin@42.fr>                      +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2016/10/20 10:49:50 by svilau            #+#    #+#             */
/*   Updated: 2017/03/17 14:07:23 by svilau           ###   ########.fr       */
/*                                                                            */
/* ************************************************************************** */

extern "C" {
	#include <rt.h>
	#include "gpu_rt.h"
	#include "object.h"
	#include "vectors.h"
}
#include "../../../frameworks/SDL2.framework/Headers/SDL.h"
#include "../../../includes/display.h"
#include <math.h>
#include <float.h>
#include <hip/hip_runtime.h>

__host__ __device__ void	get_ray_direction(t_world world, t_ray *ray, int x, int y)
{
	t_vec3d dir_point;

	dir_point = vector_substract(
							vector_add(
								world.viewplane.up_left,
								vector_scalar(
									world.camera.right_v,
									world.viewplane.x_indent * x)),
							vector_scalar(
								world.camera.up_v,
								world.viewplane.y_indent * y));
	ray->dir =	vector_calculate(
					world.camera.pos,
					dir_point);
	ray->origin = world.camera.pos;
}

/*
** Get the top-left point in virtual space of the viewplane
*/

__host__ __device__ void	get_up_left(t_world *world)
{
	world->viewplane.up_left = vector_substract(
									vector_add(
										vector_add(
											world->camera.pos,
											world->camera.dir_v),
										vector_scalar(
											world->camera.up_v,
											world->viewplane.height / 2.0f)),
									vector_scalar(
										world->camera.right_v,
										world->viewplane.width / 2.0f));
}

__host__ __device__ double		get_closest_intersection(t_world world, t_ray ray,
										t_intersection *intersection)
{
	t_intersection	intersection_tmp;

	intersection_tmp.t = DBL_MAX;
	intersection_tmp.type = '0';
	
	get_closest_sphere(world, ray, intersection, &intersection_tmp);
	get_closest_plane(world, ray, intersection, &intersection_tmp);
	get_closest_cone(world, ray, intersection, &intersection_tmp);
	get_closest_cylinder(world, ray, intersection, &intersection_tmp);
	get_closest_paraboloid(world, ray, intersection, &intersection_tmp);
	if (intersection->type == '0')
		return (0);
	else
		return (1);
}

__host__ __device__ void	cartoon_effect(t_world world, t_color *color, t_light lights, t_intersection intersection, t_ray ray)
{
	if(vector_dot(intersection.normal_v, ray.dir) > -0.3 && intersection.type != 'p'
			&& vector_dot(intersection.normal_v, ray.dir) < 0.0000001)
	{
		color->r = 0;
		color->g = 0;
		color->b = 0;
	}
}

__host__ __device__ int		ray_tracer(t_world world, int x, int y)
{
	t_ray			ray;
	t_intersection	intersection;
	t_color			color;
	int				i;

	world.cartoon = 0;

	i = 0;
	intersection.t = DBL_MAX;
	intersection.type = '0';
	color = new_color(0, 0, 0);
	get_up_left(&world);
	get_ray_direction(world, &ray, x, y);
	get_closest_intersection(world, ray, &intersection);
	if (intersection.type != '0' && intersection.t > 0.0000001)
	{
		color_add(&color, *intersection.color);
		color_multiply(&color, world.ambient.color);
		color_scalar(&color, world.ambient.intensity);
		while (i < world.lights_len)
		{
			get_light_at(world, &color, world.lights[i], intersection, ray);
			if(world.cartoon == 1)
				cartoon_effect(world, &color, world.lights[i], intersection, ray);
			i++;
		}
	}
	return (get_color(color));
}
