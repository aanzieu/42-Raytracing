extern "C" {
	#include <rt.h>
	#include "gpu_rt.h"
	#include "object.h"
	#include "vectors.h"
}
#include "../../../frameworks/SDL2.framework/Headers/SDL.h"
#include "../../../includes/display.h"
#include <math.h>
#include <float.h>
#include <hip/hip_runtime.h>

__host__ __device__ void new_intersection(t_intersection *new_i)
{
	new_i->id = -1;
	new_i->t = DBL_MAX;
	new_i->type = '0';
	new_i->depth = 0;
	new_i->transparence_coef = -1;
	new_i->refraction_coef = -1;
	new_i->reflection_coef = -1;
	new_i->chess = (t_color){-1, -1, -1};
	new_i->color = (t_color){0, 0, 0};
	new_i->pos = (t_vec3d){0, 0, 0};
	new_i->normal_v = (t_vec3d){0, 0, 0};
}

__host__ __device__ void	cartoon_effect(t_world world, t_color *color,
	t_light lights, t_intersection intersection, t_ray ray)
{
	if(world.keys.select == 1 && intersection.id_save == world.id_save)
	{
		if(vector_dot(intersection.normal_v, ray.dir) > -0.3 && intersection.type != 'p'
			&& vector_dot(intersection.normal_v, ray.dir) < 0.0000001)
		{
			color->r = 1;
			color->g = 1;
			color->b = 1;
		}
	}
	else if (world.keys.pad_0 == 6)
	{
		if(vector_dot(intersection.normal_v, ray.dir) > -0.3 && intersection.type != 'p'
			&& vector_dot(intersection.normal_v, ray.dir) < 0.0000001)
		{
			color->r = 0;
			color->g = 0;
			color->b = 0;
		}
	}
}
