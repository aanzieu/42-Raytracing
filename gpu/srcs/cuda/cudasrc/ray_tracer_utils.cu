#include "hip/hip_runtime.h"
extern "C" {
	#include <rt.h>
	#include "gpu_rt.h"
	#include "object.h"
	#include "vectors.h"
}
#include "../../../frameworks/SDL2.framework/Headers/SDL.h"
#include "../../../includes/display.h"
#include <math.h>
#include <float.h>
#include <hip/hip_runtime.h>

__host__ __device__ int get_light_box(t_light light,\
	t_intersection intersection, t_ray ray)
{
	t_cube					light_box;
	t_intersection 	intersection_tmp;

	if (light.type != LIGHT_BOX)
		return (0);
	new_intersection(&intersection_tmp);
	intersection_tmp.t = intersection.t;
	light_box.min = light.pos;
	light_box.max = (t_vec3d){light.pos.x + 0.85, light.pos.y + 0.12,\
		light.pos.z + 0.85};
	if (get_cube(light_box, ray, &intersection_tmp) == 1)
		if (intersection_tmp.t < intersection.t && \
				intersection_tmp.t > SURFACE_TOLERANCE)
		return (1);
	return (0);
}

__host__ __device__ void new_intersection(t_intersection *new_i)
{
	new_i->id = -1;
	new_i->t = DBL_MAX;
	new_i->type = '0';
	new_i->depth = 0;
	new_i->transparence_coef = -1;
	new_i->refraction_coef = -1;
	new_i->reflection_coef = -1;
	new_i->chess = (t_color){-1, -1, -1};
	new_i->color = (t_color){0, 0, 0};
	new_i->pos = (t_vec3d){0, 0, 0};
	new_i->normal_v = (t_vec3d){0, 0, 0};
}

__host__ __device__ void	cartoon_effect(t_world world, t_color *color,
	t_light lights, t_intersection intersection, t_ray ray)
{
	if(world.keys.select == 1 && intersection.id_save == world.id_save)
	{
		if(vector_dot(intersection.normal_v, ray.dir) > -0.3 && intersection.type != 'p'
			&& vector_dot(intersection.normal_v, ray.dir) < 0.0000001)
		{
			color->r = 1;
			color->g = 1;
			color->b = 1;
		}
	}
	else if (world.keys.pad_0 == 6)
	{
		if(vector_dot(intersection.normal_v, ray.dir) > -0.3 && intersection.type != 'p'
			&& vector_dot(intersection.normal_v, ray.dir) < 0.0000001)
		{
			color->r = 0;
			color->g = 0;
			color->b = 0;
		}
	}
}
