/* ************************************************************************** */
/*                                                                            */
/*                                                        :::      ::::::::   */
/*   ray_tracer.c                                       :+:      :+:    :+:   */
/*                                                    +:+ +:+         +:+     */
/*   By: svilau <marvin@42.fr>                      +#+  +:+       +#+        */
/*                                                +#+#+#+#+#+   +#+           */
/*   Created: 2016/10/20 10:49:50 by svilau            #+#    #+#             */
/*     Updated: 2017/06/29 14:26:47 by huweber          ###   ########.fr     */
/*                                                                            */
/* ************************************************************************** */

extern "C" {
	#include <rt.h>
	#include "gpu_rt.h"
	#include "object.h"
	#include "vectors.h"
}
#include "../../../frameworks/SDL2.framework/Headers/SDL.h"
#include "../../../includes/display.h"
#include <math.h>
#include <float.h>
#include <hip/hip_runtime.h>

__host__ __device__ void	get_ray_direction(t_world world, t_ray *ray,
	int x, int y)
{
	t_vec3d dir_point;

	dir_point = vector_substract(
								vector_add(world.viewplane_aa.up_left,
									vector_scalar(world.camera.right_v,
										world.viewplane_aa.x_indent * x)),
								vector_scalar(world.camera.up_v, world.viewplane_aa.y_indent * y));
	ray->dir_point = dir_point;
	ray->dir = vector_calculate(world.camera.pos, dir_point);
	ray->origin = world.camera.pos;
}

/*
** Get the top-left point in virtual space of the viewplane
*/

__host__ __device__ void	get_up_left(t_world *world)
{
	world->viewplane.up_left = vector_substract(
									vector_add(
										vector_add(
											world->camera.pos,
											world->camera.dir_v),
										vector_scalar(
											world->camera.up_v,
											world->viewplane.height / 2.0f)),
									vector_scalar(
										world->camera.right_v,
										world->viewplane.width / 2.0f));
	world->viewplane_aa.up_left = vector_substract(

									vector_add(
										vector_add(
											world->camera.pos,
											world->camera.dir_v),
										vector_scalar(
											world->camera.up_v,
											world->viewplane_aa.height / 2.0f)),
									vector_scalar(
										world->camera.right_v,
										world->viewplane_aa.width / 2.0f));
}

__host__ __device__ double		get_closest_intersection(t_world world, t_ray ray,
										t_intersection *intersection)
{
	t_intersection	intersection_tmp;
	intersection_tmp.t = DBL_MAX;
	intersection_tmp.type = '0';
	intersection_tmp.id = intersection->id;
	get_closest_sphere(world, ray, intersection, &intersection_tmp);
	// get_closest_mobius(world, ray, intersection, &intersection_tmp);
	// get_closest_torus(world, ray, intersection, &intersection_tmp);
	get_closest_plane(world, ray, intersection, &intersection_tmp);
	// get_closest_disk(world, ray, intersection, &intersection_tmp);
	// get_closest_cone(world, ray, intersection, &intersection_tmp);
	// get_closest_cube(world, ray, intersection, &intersection_tmp);
	// get_closest_hollow_cube(world, ray, intersection, &intersection_tmp);
	// get_closest_cylinder(world, ray, intersection, &intersection_tmp);
	// get_closest_paraboloid(world, ray, intersection, &intersection_tmp);
	// get_closest_hyperboloid(world, ray, intersection, &intersection_tmp);
	// get_closest_triangle(world, ray, intersection, &intersection_tmp);
	if (intersection->type == '0')
		return (0);
	return (1);
}

__host__ __device__ t_color apply_materials_cpu(t_world world, t_ray ray,
	t_intersection intersection, t_color color)
{
	if (intersection.transparence_coef > 0)
		color = color_add(color_scalar(color, 1 - intersection.transparence_coef),
		color_scalar(handle_transparence_cpu(world, ray, &intersection),
		intersection.transparence_coef));
	if (intersection.reflection_coef > 0)
		color = color_add(color_scalar(color, 1 - intersection.reflection_coef),
		color_scalar(handle_reflection_cpu(world, ray, &intersection),
		intersection.reflection_coef));
	if (intersection.refraction_coef > 0)
		color = color_add(color, handle_refraction_cpu(world, ray, &intersection));
	if (intersection.chess.r >= 0)
		color = color_add(color, handle_chess(ray, intersection));
	return (color);
}

__host__ __device__ t_color		ray_tracer_depth_cpu(t_world world, t_ray ray,\
				t_intersection intersection)
{
	t_color					color;
	int							i;

	i = 0;
	color = new_color(0, 0, 0);
	if (intersection.depth > MAX_DEPTH)
		return (color);
	get_closest_intersection(world, ray, &intersection);
	if (intersection.type == '0')
		return ((t_color){0, 0, 0});
	if(world.keys.light_none == 0)
		color = intersection.color;
	if(world.keys.light_none == 1 && world.mode == 0)
		color = apply_materials_cpu(world, ray, intersection, color);
	color = color_multiply(color, world.ambient.color);
	color = color_scalar(color, world.ambient.intensity);
	if(world.keys.select == 1)
		cartoon_effect(world, &color, world.lights[i], intersection, ray);
	while (i < world.lights_len && world.keys.light_none == 1)
	{
		if (get_light_box(world.lights[i], intersection, ray) == 1)
			color = color_scalar(color_divide(
				color_add((t_color){0.5, 0.5, 0.5}, world.lights[i].color), 2),
				0.5 + 0.5 / world.lights[i].intensity_coef);
		color = get_light_at(world, color, world.lights[i], intersection, ray);
		if (world.keys.pad_9 == 1)
			cartoon_effect(world, &color, world.lights[i], intersection, ray);
		i++;
	}
	return (color);
}

__host__ __device__ int		ray_tracer_cpu(t_world world, int x, int y)
{
	t_ray      			ray;
	t_color					color = (t_color){0, 0, 0};
	t_intersection	intersection;
	int i = 0, j = 0, xx = 0, yy = 0;

	while (i < world.aa)
	{
		j = 0;
		yy = y + i;
		while (j < world.aa)
		{
			xx = x + j;
			new_intersection(&intersection);
			ray.dir = (t_vec3d){0, 0, 0};
			ray.origin = (t_vec3d){0, 0, 0};
			get_up_left(&world);
			get_ray_direction(world, &ray, xx, yy);
			color = color_add(color, ray_tracer_depth_cpu(world, ray, intersection));
			j++;
		}
		i++;
	}
	color = color_divide(color, world.aa * world.aa);
	return (get_color(color));
}
